#include "hip/hip_runtime.h"
#include "MD_core.cuh"

#define BOX_TRAJ_COMMAND "box"
#define BOX_TRAJ_DEFAULT_FILENAME "mdbox.txt"
#define TRAJ_COMMAND "crd"
#define TRAJ_DEFAULT_FILENAME "mdcrd.dat"
#define RESTART_COMMAND "rst"
#define RESTART_DEFAULT_FILENAME "restart"

static __global__ void MD_Iteration_Leap_Frog
(const int atom_numbers, VECTOR *vel, VECTOR *crd, VECTOR *frc, VECTOR *acc, const float *inverse_mass, const float dt)
{
	int i = blockDim.x*blockIdx.x + threadIdx.x;
	if (i < atom_numbers)
	{
		acc[i].x = inverse_mass[i] * frc[i].x;
		acc[i].y = inverse_mass[i] * frc[i].y;
		acc[i].z = inverse_mass[i] * frc[i].z;

		vel[i].x = vel[i].x + dt*acc[i].x;
		vel[i].y = vel[i].y + dt*acc[i].y;
		vel[i].z = vel[i].z + dt*acc[i].z;

		crd[i].x = crd[i].x + dt*vel[i].x;
		crd[i].y = crd[i].y + dt*vel[i].y;
		crd[i].z = crd[i].z + dt*vel[i].z;

		frc[i].x = 0.;
		frc[i].y = 0.;
		frc[i].z = 0.;
	}
}

static __global__ void MD_Iteration_Leap_Frog_With_Max_Velocity
(const int atom_numbers, VECTOR *vel, VECTOR *crd, VECTOR *frc, VECTOR *acc, const float *inverse_mass, const float dt, const float max_velocity)
{
	int i = blockDim.x*blockIdx.x + threadIdx.x;
	if (i < atom_numbers)
	{
		VECTOR acc_i = inverse_mass[i] * frc[i];
		VECTOR vel_i = vel[i] + dt * acc_i;
		vel_i = Make_Vector_Not_Exceed_Value(vel_i, max_velocity);
		vel[i] = vel_i;
		crd[i] = crd[i] + dt * vel_i;
		frc[i] = { 0.0f, 0.0f, 0.0f };
	}
}

static __global__ void MD_Iteration_Gradient_Descent
(const int atom_numbers, VECTOR *crd, VECTOR *frc, const float learning_rate)
{
	int i = blockDim.x*blockIdx.x + threadIdx.x;
	if (i < atom_numbers)
	{
		crd[i].x = crd[i].x + learning_rate * frc[i].x;
		crd[i].y = crd[i].y + learning_rate * frc[i].y;
		crd[i].z = crd[i].z + learning_rate * frc[i].z;

		frc[i].x = 0.;
		frc[i].y = 0.;
		frc[i].z = 0.;
	}
}


static __global__ void MD_Iteration_Speed_Verlet_1(const int atom_numbers, const float half_dt, const float dt, const VECTOR *acc, VECTOR *vel, VECTOR *crd, VECTOR *frc)
{
	int i = blockDim.x*blockIdx.x + threadIdx.x;
	if (i < atom_numbers)
	{
		vel[i].x = vel[i].x + half_dt*acc[i].x;
		vel[i].y = vel[i].y + half_dt*acc[i].y;
		vel[i].z = vel[i].z + half_dt*acc[i].z;
		crd[i].x = crd[i].x + dt*vel[i].x;
		crd[i].y = crd[i].y + dt*vel[i].y;
		crd[i].z = crd[i].z + dt*vel[i].z;
		frc[i].x = 0.;
		frc[i].y = 0.;
		frc[i].z = 0.;
	}
}

static __global__ void MD_Iteration_Speed_Verlet_2(const int atom_numbers, const float half_dt, const float *inverse_mass, const VECTOR *frc, VECTOR *vel, VECTOR *acc)
{
	int i = blockDim.x*blockIdx.x + threadIdx.x;
	if (i < atom_numbers)
	{
		acc[i].x = inverse_mass[i] * frc[i].x;
		acc[i].y = inverse_mass[i] * frc[i].y;
		acc[i].z = inverse_mass[i] * frc[i].z;
		vel[i].x = vel[i].x + half_dt*acc[i].x;
		vel[i].y = vel[i].y + half_dt*acc[i].y;
		vel[i].z = vel[i].z + half_dt*acc[i].z;
	}
}

static __global__ void MD_Iteration_Speed_Verlet_2_With_Max_Velocity(const int atom_numbers, const float half_dt, const float *inverse_mass, const VECTOR *frc, VECTOR *vel, VECTOR *acc, const float max_velocity)
{
	int i = blockDim.x*blockIdx.x + threadIdx.x;
	if (i < atom_numbers)
	{
		VECTOR acc_i = inverse_mass[i] * frc[i];
		VECTOR vel_i = vel[i] + half_dt * acc_i;

		vel[i] = Make_Vector_Not_Exceed_Value(vel_i, max_velocity);
		acc[i] = acc_i;
	}
}


static __global__ void Get_Center_Of_Mass(const int residue_numbers, const int *start, const int *end,
	const VECTOR *crd, const float *atom_mass, const float *residue_mass_inverse, VECTOR *center_of_mass)
{
	for (int residue_i = blockDim.x*blockIdx.x + threadIdx.x; residue_i < residue_numbers; residue_i += gridDim.x * blockDim.x)
	{
		VECTOR com_lin = { 0.0f, 0.0f, 0.0f };
		for (int atom_i = start[residue_i]; atom_i < end[residue_i]; atom_i += 1)
		{
			com_lin = com_lin + atom_mass[atom_i] * crd[atom_i];
		}
		center_of_mass[residue_i] = residue_mass_inverse[residue_i] * com_lin;
	}
}

static __global__ void Map_Center_Of_Mass(const int residue_numbers, const int *start, const int *end,
	const float scaler, const VECTOR *center_of_mass, const VECTOR box_length, const VECTOR *no_wrap_crd, VECTOR *crd)
{
	VECTOR trans_vec;
	VECTOR com;
	for (int residue_i = blockDim.x*blockIdx.x + threadIdx.x; residue_i < residue_numbers; residue_i += gridDim.x * blockDim.x)
	{
		com = center_of_mass[residue_i];

		trans_vec.x = com.x - floorf(com.x / box_length.x) * box_length.x;
		trans_vec.y = com.y - floorf(com.y / box_length.y) * box_length.y;
		trans_vec.z = com.z - floorf(com.z / box_length.z) * box_length.z;
		trans_vec = scaler * trans_vec - com;

		for (int atom_i = start[residue_i] + threadIdx.y; atom_i < end[residue_i]; atom_i += blockDim.y)
		{
			crd[atom_i] = no_wrap_crd[atom_i] + trans_vec;
		}
	}
}

static __global__ void Add_Sum_List(int n, float *atom_virial, float *sum_virial)
{
	float temp = 0;
	for (int i = threadIdx.x; i < n; i = i + blockDim.x)
	{
		temp = temp + atom_virial[i];
	}
	atomicAdd(sum_virial, temp);
}

static __global__ void Calculate_Pressure_Cuda(const float V_inverse, const float *ek, const float *virial, float *pressure)
{
	pressure[0] = (ek[0] * 2 + virial[0]) * 0.33333333333333f * V_inverse;
}




static __global__ void MD_Temperature
(const int residue_numbers, const int *start, const int *end, float *ek,
const VECTOR *atom_vel, const float *atom_mass)
{
	int residue_i = blockDim.x*blockIdx.x + threadIdx.x;
	if (residue_i < residue_numbers)
	{
		VECTOR momentum = { 0., 0., 0. };
		float res_mass = 0.; //待提出，只需要初始时计算一遍
		int s = start[residue_i];
		int e = end[residue_i];
		float mass_lin;
		for (int atom_i = s; atom_i < e; atom_i = atom_i + 1)
		{
			mass_lin = atom_mass[atom_i];

			momentum.x = momentum.x + mass_lin*atom_vel[atom_i].x;
			momentum.y = momentum.y + mass_lin*atom_vel[atom_i].y;
			momentum.z = momentum.z + mass_lin*atom_vel[atom_i].z;
			res_mass = res_mass + mass_lin;
		}
		ek[residue_i] = 0.5*(momentum.x*momentum.x + momentum.y*momentum.y + momentum.z*momentum.z) / res_mass * 2. / 3. / CONSTANT_kB / residue_numbers;
	}
}

static __global__ void MD_Residue_Ek
(const int residue_numbers,const int *start,const int *end,float *ek,
const VECTOR *atom_vel,const float *atom_mass)
{
	int residue_i = blockDim.x*blockIdx.x + threadIdx.x;
	if (residue_i < residue_numbers)
	{
		VECTOR momentum = {0.,0.,0.};
		float res_mass = 0.; //待提出，只需要初始时计算一遍
		int s = start[residue_i];
		int e = end[residue_i];
		float mass_lin;
		for (int atom_i = s; atom_i < e; atom_i = atom_i + 1)
		{
			mass_lin = atom_mass[atom_i];

			momentum.x = momentum.x + mass_lin*atom_vel[atom_i].x;
			momentum.y = momentum.y + mass_lin*atom_vel[atom_i].y;
			momentum.z = momentum.z + mass_lin*atom_vel[atom_i].z;
			res_mass = res_mass + mass_lin;
		}
		ek[residue_i] = 0.5*(momentum.x*momentum.x + momentum.y*momentum.y + momentum.z*momentum.z) / res_mass;
	}
}

static __global__ void MD_Atom_Ek
(const int atom_numbers, float *ek, const VECTOR *atom_vel, const float *atom_mass)
{
	int atom_i = blockDim.x*blockIdx.x + threadIdx.x;
	if (atom_i < atom_numbers)
	{
		VECTOR v = atom_vel[atom_i];
		ek[atom_i] = 0.5 * v * v * atom_mass[atom_i];
	}
}

void MD_INFORMATION::system_information::Initial(CONTROLLER *controller, MD_INFORMATION *md_info)
{
	this->md_info = md_info;
	steps = 0;
	step_limit = 1000;
	if (controller[0].Command_Exist("step_limit"))
	{
		step_limit = atoi(controller[0].Command("step_limit"));
	}

	target_temperature = 300.0f;
	if (md_info->mode >= md_info->NVT && controller[0].Command_Exist("target_temperature"))
	{
		target_temperature = atof(controller[0].Command("target_temperature"));
	}

	target_pressure = 1;
	if (md_info->mode == md_info->NPT &&  controller[0].Command_Exist("target_pressure"))
		target_pressure = atof(controller[0].Command("target_pressure"));
	target_pressure *= CONSTANT_PRES_CONVERTION_INVERSE;

	controller->Step_Print_Initial("step", "%d");
	controller->Step_Print_Initial("time", "%.3lf");
	controller->Step_Print_Initial("temperature", "%.2f");
	controller->Step_Print_Initial("potential", "%.2f");
	Cuda_Malloc_Safely((void**)&this->d_virial, sizeof(float));
	Cuda_Malloc_Safely((void**)&this->d_pressure, sizeof(float));
	Cuda_Malloc_Safely((void**)&this->d_temperature, sizeof(float));
	Cuda_Malloc_Safely((void**)&this->d_potential, sizeof(float));
	Cuda_Malloc_Safely((void**)&this->d_sum_of_atom_ek, sizeof(float));
}

void MD_INFORMATION::non_bond_information::Initial(CONTROLLER *controller, MD_INFORMATION *md_info)
{
	if (controller[0].Command_Exist("skin"))
	{
		skin = atof(controller[0].Command("skin"));
	}
	else
	{
		skin = 2.0;
	}
	controller->printf("    skin set to %.2f Angstram\n", skin);

	if (controller[0].Command_Exist("cutoff"))
	{
		cutoff = atof(controller[0].Command("cutoff"));
	}
	else
	{
		cutoff = 10.0;
	}
	controller->printf("    cutoff set to %.2f Angstram\n", cutoff);
	/*===========================
	读取排除表相关信息
	============================*/
	if (controller[0].Command_Exist("exclude_in_file"))
	{
		FILE *fp = NULL;
		controller->printf("    Start reading excluded list:\n");
		Open_File_Safely(&fp, controller[0].Command("exclude_in_file"), "r");
		
		int atom_numbers = 0;
		fscanf(fp, "%d %d", &atom_numbers, &excluded_atom_numbers);
		if (md_info->atom_numbers > 0 && md_info->atom_numbers != atom_numbers)
		{
			controller->printf("        Error: atom_numbers is not equal: %d %d\n", md_info->atom_numbers, atom_numbers);
			getchar();
			exit(1);
		}
		else if (md_info->atom_numbers == 0)
		{
			md_info->atom_numbers = atom_numbers;
		}
		controller->printf("        excluded list total length is %d\n", excluded_atom_numbers);

		Cuda_Malloc_Safely((void**)&d_excluded_list_start, sizeof(int)*atom_numbers);
		Cuda_Malloc_Safely((void**)&d_excluded_numbers, sizeof(int)*atom_numbers);
		Cuda_Malloc_Safely((void**)&d_excluded_list, sizeof(int)*excluded_atom_numbers);

		Malloc_Safely((void**)&h_excluded_list_start, sizeof(int)*atom_numbers);
		Malloc_Safely((void**)&h_excluded_numbers, sizeof(int)*atom_numbers);
		Malloc_Safely((void**)&h_excluded_list, sizeof(int)*excluded_atom_numbers);
		int count = 0;
		for (int i = 0; i < atom_numbers; i++)
		{
			fscanf(fp, "%d", &h_excluded_numbers[i]);
			h_excluded_list_start[i] = count;
			for (int j = 0; j < h_excluded_numbers[i]; j++)
			{
				fscanf(fp, "%d", &h_excluded_list[count]);
				count++;
			}
		}
		hipMemcpy(d_excluded_list_start, h_excluded_list_start, sizeof(int)*atom_numbers, hipMemcpyHostToDevice);
		hipMemcpy(d_excluded_numbers, h_excluded_numbers, sizeof(int)*atom_numbers, hipMemcpyHostToDevice);
		hipMemcpy(d_excluded_list, h_excluded_list, sizeof(int)*excluded_atom_numbers, hipMemcpyHostToDevice);
		controller->printf("    End reading excluded list\n\n");
		fclose(fp);
	}
	else if (controller[0].Command_Exist("amber_parm7"))
	{
		/*===========================
		从parm中读取排除表相关信息
		============================*/
		FILE *parm = NULL;
		Open_File_Safely(&parm, controller[0].Command("amber_parm7"), "r");
		controller->printf("    Start reading excluded list from AMBER file:\n");
		while (true)
		{
			char temps[CHAR_LENGTH_MAX];
			char temp_first_str[CHAR_LENGTH_MAX];
			char temp_second_str[CHAR_LENGTH_MAX];
			if (!fgets(temps, CHAR_LENGTH_MAX, parm))
			{
				break;
			}
			if (sscanf(temps, "%s %s", temp_first_str, temp_second_str) != 2)
			{
				continue;
			}
			if (strcmp(temp_first_str, "%FLAG") == 0
				&& strcmp(temp_second_str, "POINTERS") == 0)
			{
				fgets(temps, CHAR_LENGTH_MAX, parm);

				int atom_numbers = 0;
				fscanf(parm, "%d\n", &atom_numbers);
				if (md_info->atom_numbers > 0 && md_info->atom_numbers != atom_numbers)
				{
					controller->printf("        Error: atom_numbers is not equal: %d %d\n", md_info->atom_numbers, atom_numbers);
					getchar();
					exit(1);
				}
				else if (md_info->atom_numbers == 0)
				{
					md_info->atom_numbers = atom_numbers;
				}
				Cuda_Malloc_Safely((void**)&d_excluded_list_start, sizeof(int)*atom_numbers);
				Cuda_Malloc_Safely((void**)&d_excluded_numbers, sizeof(int)*atom_numbers);

				Malloc_Safely((void**)&h_excluded_list_start, sizeof(int)*atom_numbers);
				Malloc_Safely((void**)&h_excluded_numbers, sizeof(int)*atom_numbers);
				for (int i = 0; i < 9; i = i + 1)
				{
					fscanf(parm, "%d\n", &excluded_atom_numbers);
				}
				fscanf(parm, "%d\n", &excluded_atom_numbers);
				controller->printf("        excluded list total length is %d\n", excluded_atom_numbers);

				Cuda_Malloc_Safely((void**)&d_excluded_list, sizeof(int)*excluded_atom_numbers);
				Malloc_Safely((void**)&h_excluded_list, sizeof(int)*excluded_atom_numbers);
			}

			//read atom_excluded_number for every atom
			if (strcmp(temp_first_str, "%FLAG") == 0
				&& strcmp(temp_second_str, "NUMBER_EXCLUDED_ATOMS") == 0)
			{
				fgets(temps, CHAR_LENGTH_MAX, parm);
				for (int i = 0; i<md_info->atom_numbers; i = i + 1)
				{
					fscanf(parm, "%d\n", &h_excluded_numbers[i]);
				}
			}
			//read every atom's excluded atom list
			if (strcmp(temp_first_str, "%FLAG") == 0
				&& strcmp(temp_second_str, "EXCLUDED_ATOMS_LIST") == 0)
			{
				int count = 0;
				int none_count = 0;
				int lin = 0;
				fgets(temps, CHAR_LENGTH_MAX, parm);
				for (int i = 0; i<md_info->atom_numbers; i = i + 1)
				{
					h_excluded_list_start[i] = count;
					for (int j = 0; j<h_excluded_numbers[i]; j = j + 1)
					{
						fscanf(parm, "%d\n", &lin);
						if (lin == 0)
						{
							h_excluded_numbers[i] = 0;
							break;
						}
						else
						{
							h_excluded_list[count] = lin - 1;
							count = count + 1;
						}
					}
					if (h_excluded_numbers[i] > 0)
						thrust::sort(&h_excluded_list[h_excluded_list_start[i]], &h_excluded_list[h_excluded_list_start[i]] + h_excluded_numbers[i]);
				}
			}
		}

		hipMemcpy(d_excluded_list_start, h_excluded_list_start, sizeof(int)*md_info->atom_numbers, hipMemcpyHostToDevice);
		hipMemcpy(d_excluded_numbers, h_excluded_numbers, sizeof(int)*md_info->atom_numbers, hipMemcpyHostToDevice);
		hipMemcpy(d_excluded_list, h_excluded_list, sizeof(int)*excluded_atom_numbers, hipMemcpyHostToDevice);
		controller->printf("    End reading excluded list from AMBER file\n\n");
		fclose(parm);
	}
	else
	{
		int atom_numbers = md_info->atom_numbers;
		excluded_atom_numbers = 0;
		controller->printf("    Set all atom exclude no atoms as default\n"); 

		Cuda_Malloc_Safely((void**)&d_excluded_list_start, sizeof(int)*atom_numbers);
		Cuda_Malloc_Safely((void**)&d_excluded_numbers, sizeof(int)*atom_numbers);
		Cuda_Malloc_Safely((void**)&d_excluded_list, sizeof(int)*excluded_atom_numbers);

		Malloc_Safely((void**)&h_excluded_list_start, sizeof(int)*atom_numbers);
		Malloc_Safely((void**)&h_excluded_numbers, sizeof(int)*atom_numbers);
		Malloc_Safely((void**)&h_excluded_list, sizeof(int)*excluded_atom_numbers);


		int count = 0;
		for (int i = 0; i < atom_numbers; i++)
		{
			h_excluded_numbers[i] = 0;
			h_excluded_list_start[i] = count;
			for (int j = 0; j < h_excluded_numbers[i]; j++)
			{
				h_excluded_list[count] = 0;
				count++;
			}
		}
		hipMemcpy(d_excluded_list_start, h_excluded_list_start, sizeof(int)*atom_numbers, hipMemcpyHostToDevice);
		hipMemcpy(d_excluded_numbers, h_excluded_numbers, sizeof(int)*atom_numbers, hipMemcpyHostToDevice);
		hipMemcpy(d_excluded_list, h_excluded_list, sizeof(int)*excluded_atom_numbers, hipMemcpyHostToDevice);
	}

}

void MD_INFORMATION::periodic_box_condition_information::Initial(CONTROLLER *controller, VECTOR box_length)
{
	crd_to_uint_crd_cof = CONSTANT_UINT_MAX_FLOAT / box_length;
	quarter_crd_to_uint_crd_cof = 0.25 * crd_to_uint_crd_cof;
	uint_dr_to_dr_cof = 1.0f / crd_to_uint_crd_cof;
}

void MD_INFORMATION::Read_Mode(CONTROLLER *controller)
{
	if (controller[0].Command_Exist("mode"))
	{
		if (is_str_equal(controller[0].Command("mode"), "NVT"))
		{
			controller->printf("    Mode set to NVT\n");
			mode = 1;
		}
		else if (is_str_equal(controller[0].Command("mode"), "NPT"))
		{
			controller->printf("    Mode set to NPT\n");
			mode = 2;
		}
		else if (is_str_equal(controller[0].Command("mode"), "Minimization"))
		{
			controller->printf("    Mode set to Energy Minimization\n");
			mode = -1;
		}
		else if (is_str_equal(controller[0].Command("mode"), "NVE"))
		{
			controller->printf("    Mode set to NVE\n");
			mode = 0;
		}
		else
		{
			controller->printf("    Warning: Mode '%s' not match. Set to NVE as default\n", controller[0].Command("mode"));
			mode = 0;
		}
	}
	else
	{
		controller->printf("    Mode set to NVE as default\n");
		mode = 0;
	}
}

void MD_INFORMATION::Read_dt(CONTROLLER *controller)
{
	if (controller[0].Command_Exist("dt"))
	{
		controller->printf("    dt set to %f ps\n", atof(controller[0].Command("dt")));
		dt = atof(controller[0].Command("dt")) * CONSTANT_TIME_CONVERTION;
		sscanf(controller[0].Command("dt"), "%lf", &sys.dt_in_ps);
	}
	else
	{
		dt = 0.001 * CONSTANT_TIME_CONVERTION;
		sys.dt_in_ps = 0.001;
		controller->printf("    dt set to %f ps\n", 0.001);
	}
}

void MD_INFORMATION::trajectory_output::Initial(CONTROLLER *controller, MD_INFORMATION *md_info)
{
	this->md_info = md_info;
	current_crd_synchronized_step = 0;
	is_molecule_map_output = 0;
	if (controller[0].Command_Exist("molecule_map_output"))
	{
		is_molecule_map_output = atoi(controller[0].Command("molecule_map_output"));
	}
	write_trajectory_interval = 1000;
	if (controller[0].Command_Exist("write_information_interval"))
	{
		write_trajectory_interval = atoi(controller[0].Command("write_information_interval"));
	}
	write_restart_file_interval = write_trajectory_interval;
	if (controller[0].Command_Exist("write_restart_file_interval"))
	{
		write_restart_file_interval = atoi(controller[0].Command("write_restart_file_interval"));
	}
	if (controller->Command_Exist(TRAJ_COMMAND))
	{
		Open_File_Safely(&crd_traj, controller->Command(TRAJ_COMMAND), "wb");
	}
	else
	{
		Open_File_Safely(&crd_traj, TRAJ_DEFAULT_FILENAME, "wb");
	}
	if (controller->Command_Exist(BOX_TRAJ_COMMAND))
	{
		Open_File_Safely(&box_traj, controller->Command(BOX_TRAJ_COMMAND), "w");
	}
	else
	{
		Open_File_Safely(&box_traj, BOX_TRAJ_DEFAULT_FILENAME, "w");
	}
	if (controller->Command_Exist(RESTART_COMMAND))
	{
		strcpy(restart_name, controller->Command(RESTART_COMMAND));
	}
	else
	{
		strcpy(restart_name, RESTART_DEFAULT_FILENAME);
	}
}

void MD_INFORMATION::NVE_iteration::Initial(CONTROLLER *controller, MD_INFORMATION *md_info)
{
	this->md_info = md_info;
	max_velocity = -1;
	if (controller[0].Command_Exist("nve_velocity_max"))
	{
		max_velocity = atof(controller[0].Command("nve_velocity_max"));
	}
}
void MD_INFORMATION::residue_information::Read_AMBER_Parm7(const char *file_name, CONTROLLER controller)
{
	FILE *parm = NULL;
	Open_File_Safely(&parm, file_name, "r");
	controller.printf("    Start reading residue informataion from AMBER parm7:\n");
	
	while (true)
	{
		char temps[CHAR_LENGTH_MAX];
		char temp_first_str[CHAR_LENGTH_MAX];
		char temp_second_str[CHAR_LENGTH_MAX];
		if (fgets(temps, CHAR_LENGTH_MAX, parm) == NULL)
		{
			break;
		}
		if (sscanf(temps, "%s %s", temp_first_str, temp_second_str) != 2)
		{
			continue;
		}
		//read in atomnumber atomljtypenumber
		if (strcmp(temp_first_str, "%FLAG") == 0
			&& strcmp(temp_second_str, "POINTERS") == 0)
		{
			fgets(temps, CHAR_LENGTH_MAX, parm);

			int atom_numbers = 0;
			fscanf(parm, "%d", &atom_numbers);
			if (md_info->atom_numbers > 0 && md_info->atom_numbers != atom_numbers)
			{
				controller.printf("        Error: atom_numbers is not equal: %d %d\n", md_info->atom_numbers, atom_numbers);
				getchar();
				exit(1);
			}
			else if (md_info->atom_numbers == 0)
			{
				md_info->atom_numbers = atom_numbers;
			}
			for (int i = 0; i < 10; i = i + 1)
			{
				int lin;
				fscanf(parm, "%d\n", &lin);
			}
			fscanf(parm, "%d\n", &this->residue_numbers);//NRES
			controller.printf("        residue_numbers is %d\n", this->residue_numbers);

			Malloc_Safely((void**)&h_mass, sizeof(float)*this->residue_numbers);
			Malloc_Safely((void**)&h_mass_inverse, sizeof(float)*this->residue_numbers);
			Malloc_Safely((void**)&h_res_start, sizeof(int)*this->residue_numbers);
			Malloc_Safely((void**)&h_res_end, sizeof(int)*this->residue_numbers);
			Malloc_Safely((void**)&h_momentum, sizeof(float)*this->residue_numbers);
			Malloc_Safely((void**)&h_center_of_mass, sizeof(VECTOR)*this->residue_numbers);
			Malloc_Safely((void**)&h_sigma_of_res_ek, sizeof(float));

			Cuda_Malloc_Safely((void**)&d_mass, sizeof(float)*this->residue_numbers);
			Cuda_Malloc_Safely((void**)&d_mass_inverse, sizeof(float)*this->residue_numbers);
			Cuda_Malloc_Safely((void**)&d_res_start, sizeof(int)*this->residue_numbers);
			Cuda_Malloc_Safely((void**)&d_res_end, sizeof(int)*this->residue_numbers);
			Cuda_Malloc_Safely((void**)&d_momentum, sizeof(float)*this->residue_numbers);
			Cuda_Malloc_Safely((void**)&d_center_of_mass, sizeof(VECTOR)*this->residue_numbers);
			Cuda_Malloc_Safely((void**)&res_ek_energy, sizeof(float)*this->residue_numbers); 
			Cuda_Malloc_Safely((void**)&sigma_of_res_ek, sizeof(float));
		}//FLAG POINTERS

		//residue range read
		if (strcmp(temp_first_str, "%FLAG") == 0
			&& strcmp(temp_second_str, "RESIDUE_POINTER") == 0)
		{
			fgets(temps, CHAR_LENGTH_MAX, parm);
			//注意读进来的数的编号要减1
			int *lin_serial;
			Malloc_Safely((void**)&lin_serial, sizeof(int)* this->residue_numbers);
			for (int i = 0; i<this->residue_numbers; i = i + 1)
			{
				fscanf(parm, "%d\n", &lin_serial[i]);
			}
			for (int i = 0; i<this->residue_numbers - 1; i = i + 1)
			{
				h_res_start[i] = lin_serial[i] - 1;
				h_res_end[i] = lin_serial[i + 1] - 1;
			}
			h_res_start[this->residue_numbers - 1] = lin_serial[this->residue_numbers - 1] - 1;
			h_res_end[this->residue_numbers - 1] = md_info->atom_numbers + 1 - 1;

			free(lin_serial);
		}
	}//while cycle

	hipMemcpy(this->d_res_start, h_res_start, sizeof(int)*this->residue_numbers, hipMemcpyHostToDevice);
	hipMemcpy(this->d_res_end, h_res_end, sizeof(int)*this->residue_numbers, hipMemcpyHostToDevice);

	controller.printf("    End reading residue informataion from AMBER parm7\n");

	fclose(parm);
}

void MD_INFORMATION::residue_information::Initial(CONTROLLER *controller, MD_INFORMATION *md_info)
{
	this->md_info = md_info;
	if (!(controller[0].Command_Exist("residue_in_file")))
	{
		if (controller[0].Command_Exist("amber_parm7"))
		{
			Read_AMBER_Parm7(controller[0].Command("amber_parm7"), controller[0]);
			is_initialzed = 1;
		}
		//对于没有residue输入的模拟，默认每个粒子作为一个residue
		else
		{
			residue_numbers = md_info->atom_numbers;
			controller->printf("    Set default residue list:\n");
			controller->printf("        residue_numbers is %d\n", residue_numbers);
			Malloc_Safely((void**)&h_mass, sizeof(float)*this->residue_numbers);
			Malloc_Safely((void**)&h_mass_inverse, sizeof(float)*this->residue_numbers);
			Malloc_Safely((void**)&h_res_start, sizeof(int)*this->residue_numbers);
			Malloc_Safely((void**)&h_res_end, sizeof(int)*this->residue_numbers);
			Malloc_Safely((void**)&h_momentum, sizeof(float)*this->residue_numbers);
			Malloc_Safely((void**)&h_center_of_mass, sizeof(VECTOR)*this->residue_numbers);
			Malloc_Safely((void**)&h_sigma_of_res_ek, sizeof(float));

			Cuda_Malloc_Safely((void**)&d_mass, sizeof(float)*this->residue_numbers);
			Cuda_Malloc_Safely((void**)&d_mass_inverse, sizeof(float)*this->residue_numbers);
			Cuda_Malloc_Safely((void**)&d_res_start, sizeof(int)*this->residue_numbers);
			Cuda_Malloc_Safely((void**)&d_res_end, sizeof(int)*this->residue_numbers);
			Cuda_Malloc_Safely((void**)&d_momentum, sizeof(float)*this->residue_numbers);
			Cuda_Malloc_Safely((void**)&d_center_of_mass, sizeof(VECTOR)*this->residue_numbers);
			Cuda_Malloc_Safely((void**)&res_ek_energy, sizeof(float)*this->residue_numbers);
			Cuda_Malloc_Safely((void**)&sigma_of_res_ek, sizeof(float));
			int count = 0;
			int temp=1;//每个粒子作为一个residue
			for (int i = 0; i < residue_numbers; i++)
			{
				h_res_start[i] = count;
				count += temp;
				h_res_end[i] = count;
			}
			hipMemcpy(d_res_start, h_res_start, sizeof(int)* residue_numbers, hipMemcpyHostToDevice);
			hipMemcpy(d_res_end, h_res_end, sizeof(int)* residue_numbers, hipMemcpyHostToDevice);
			controller->printf("    End reading residue list\n\n");
			is_initialzed = 1;
		}
	}
	else
	{
		FILE *fp = NULL;
		controller->printf("    Start reading residue list:\n");
		Open_File_Safely(&fp, controller[0].Command("residue_in_file"), "r");
		int atom_numbers = 0;
		fscanf(fp, "%d %d", &atom_numbers, &residue_numbers);
		if (md_info->atom_numbers > 0 && md_info->atom_numbers != atom_numbers)
		{
			controller->printf("        Error: atom_numbers is not equal: %d %d\n", md_info->atom_numbers, atom_numbers);
			getchar();
			exit(1);
		}
		else if (md_info->atom_numbers == 0)
		{
			md_info->atom_numbers = atom_numbers;
		}
		controller->printf("        residue_numbers is %d\n", residue_numbers);
		Malloc_Safely((void**)&h_mass, sizeof(float)*this->residue_numbers);
		Malloc_Safely((void**)&h_mass_inverse, sizeof(float)*this->residue_numbers);
		Malloc_Safely((void**)&h_res_start, sizeof(int)*this->residue_numbers);
		Malloc_Safely((void**)&h_res_end, sizeof(int)*this->residue_numbers);
		Malloc_Safely((void**)&h_momentum, sizeof(float)*this->residue_numbers);
		Malloc_Safely((void**)&h_center_of_mass, sizeof(VECTOR)*this->residue_numbers);
		Malloc_Safely((void**)&h_sigma_of_res_ek, sizeof(float));

		Cuda_Malloc_Safely((void**)&d_mass, sizeof(float)*this->residue_numbers);
		Cuda_Malloc_Safely((void**)&d_mass_inverse, sizeof(float)*this->residue_numbers);
		Cuda_Malloc_Safely((void**)&d_res_start, sizeof(int)*this->residue_numbers);
		Cuda_Malloc_Safely((void**)&d_res_end, sizeof(int)*this->residue_numbers);
		Cuda_Malloc_Safely((void**)&d_momentum, sizeof(float)*this->residue_numbers);
		Cuda_Malloc_Safely((void**)&d_center_of_mass, sizeof(VECTOR)*this->residue_numbers);
		Cuda_Malloc_Safely((void**)&res_ek_energy, sizeof(float)*this->residue_numbers);
		Cuda_Malloc_Safely((void**)&sigma_of_res_ek, sizeof(float));

		int count = 0;
		int temp;
		for (int i = 0; i < residue_numbers; i++)
		{
			h_res_start[i] = count;
			fscanf(fp, "%d", &temp);
			count += temp;
			h_res_end[i] = count;
		}
		hipMemcpy(d_res_start, h_res_start, sizeof(int)* residue_numbers, hipMemcpyHostToDevice);
		hipMemcpy(d_res_end, h_res_end, sizeof(int)* residue_numbers, hipMemcpyHostToDevice);
		controller->printf("    End reading residue list\n\n");
		fclose(fp);
		is_initialzed = 1;
	}
	if (is_initialzed)
	{
		if (md_info->h_mass != NULL)
		{
			for (int i = 0; i < residue_numbers; i++)
			{
				float temp_mass = 0;
				for (int j = h_res_start[i]; j < h_res_end[i]; j++)
				{
					temp_mass += md_info->h_mass[j];
				}
				this->h_mass[i] = temp_mass;
				if (temp_mass == 0)
					this->h_mass_inverse[i] = 0;
				else
					this->h_mass_inverse[i] = 1.0 / temp_mass;
			}
			hipMemcpy(d_mass_inverse, h_mass_inverse, sizeof(float)* residue_numbers, hipMemcpyHostToDevice);
			hipMemcpy(d_mass, h_mass, sizeof(float)* residue_numbers, hipMemcpyHostToDevice);
		}
		else
		{
			controller->printf("    Error: atom mass should be initialized before residue mass\n");
			getchar();
			exit(1);
		}
	}
}

void MD_INFORMATION::Read_Coordinate_And_Velocity(CONTROLLER *controller)
{
	sys.start_time = 0.0;
	if (controller[0].Command_Exist("coordinate_in_file"))
	{
		Read_Coordinate_In_File(controller[0].Command("coordinate_in_file"), controller[0]);
		if (controller[0].Command_Exist("velocity_in_file"))
		{
			FILE *fp = NULL;
			controller->printf("    Start reading velocity_in_file:\n");
			Open_File_Safely(&fp, controller[0].Command("velocity_in_file"), "r");
			
			int atom_numbers = 0;
			char lin[CHAR_LENGTH_MAX];
			fgets(lin, CHAR_LENGTH_MAX, fp);
			int scanf_ret = sscanf(lin, "%d", &atom_numbers);
			if (this->atom_numbers > 0 && this->atom_numbers != atom_numbers)
			{
				controller->printf("        Error: atom_numbers is not equal: %d %d\n", this->atom_numbers, atom_numbers);
				getchar();
				exit(1);
			}
			Malloc_Safely((void**)&velocity, sizeof(VECTOR)*this->atom_numbers);
			Cuda_Malloc_Safely((void**)&vel, sizeof(VECTOR)*this->atom_numbers);
			for (int i = 0; i < atom_numbers; i++)
			{
				fscanf(fp, "%f %f %f", &velocity[i].x, &velocity[i].y, &velocity[i].z);
			}
			hipMemcpy(vel, velocity, sizeof(VECTOR)* atom_numbers, hipMemcpyHostToDevice);
			controller->printf("    End reading velocity_in_file\n\n");
			fclose(fp);
		}
		else
		{
			controller->printf("    Velocity is set to zero as default\n");
			Malloc_Safely((void**)&velocity, sizeof(VECTOR)*this->atom_numbers);
			Cuda_Malloc_Safely((void**)&vel, sizeof(VECTOR)*this->atom_numbers);
			for (int i = 0; i < atom_numbers; i++)
			{
				velocity[i].x = 0;
				velocity[i].y = 0;
				velocity[i].z = 0;
			}
			hipMemcpy(vel, velocity, sizeof(VECTOR)* atom_numbers, hipMemcpyHostToDevice);
		}
	}
	else if (controller[0].Command_Exist("amber_rst7"))
	{
		output.amber_irest = 1;
		if (controller[0].Command_Exist("amber_irest"))
			output.amber_irest = atoi(controller[0].Command("amber_irest"));
		Read_Rst7(controller[0].Command("amber_rst7"), output.amber_irest, controller[0]);
	}
	else
	{
		printf("MD basic information needed. Specify the coordinate in file.\n");
		getchar();
		exit(1);
	}
}

void MD_INFORMATION::Read_Mass(CONTROLLER *controller)
{
	if (controller[0].Command_Exist("mass_in_file"))
	{
		FILE *fp = NULL;
		controller->printf("    Start reading mass:\n");
		Open_File_Safely(&fp, controller[0].Command("mass_in_file"), "r");
		int atom_numbers = 0;
		char lin[CHAR_LENGTH_MAX];
		fgets(lin, CHAR_LENGTH_MAX, fp);
		int scanf_ret = sscanf(lin, "%d", &atom_numbers);
		if (this->atom_numbers > 0 && this->atom_numbers != atom_numbers)
		{
			controller->printf("        Error: atom_numbers is not equal: %d %d\n", this->atom_numbers, atom_numbers);
			getchar();
			exit(1);
		}
		else if (this->atom_numbers == 0)
		{
			this->atom_numbers = atom_numbers;
		}
		Malloc_Safely((void**)&h_mass, sizeof(float)* atom_numbers);
		Malloc_Safely((void**)&h_mass_inverse, sizeof(float)* atom_numbers);
		Cuda_Malloc_Safely((void**)&d_mass, sizeof(float)* atom_numbers);
		Cuda_Malloc_Safely((void**)&d_mass_inverse, sizeof(float)* atom_numbers);
		sys.total_mass = 0;
		for (int i = 0; i < atom_numbers; i++)
		{
			fscanf(fp, "%f", &h_mass[i]);
			sys.total_mass += h_mass[i];
			if (h_mass[i] == 0)
				h_mass_inverse[i] = 0;
			else
				h_mass_inverse[i] = 1.0 / h_mass[i];

		}
		controller->printf("    End reading mass\n\n");
		fclose(fp);
	}
	else if (controller[0].Command_Exist("amber_parm7"))
	{
		FILE *parm = NULL;
		Open_File_Safely(&parm, controller[0].Command("amber_parm7"), "r");
		controller[0].printf("    Start reading mass from AMBER parm7:\n");
		while (true)
		{
			char temps[CHAR_LENGTH_MAX];
			char temp_first_str[CHAR_LENGTH_MAX];
			char temp_second_str[CHAR_LENGTH_MAX];
			if (fgets(temps, CHAR_LENGTH_MAX, parm) == NULL)
			{
				break;
			}
			if (sscanf(temps, "%s %s", temp_first_str, temp_second_str) != 2)
			{
				continue;
			}
			//read in atomnumber atomljtypenumber
			if (strcmp(temp_first_str, "%FLAG") == 0
				&& strcmp(temp_second_str, "POINTERS") == 0)
			{
				fgets(temps, CHAR_LENGTH_MAX, parm);

				int atom_numbers = 0;
				fscanf(parm, "%d", &atom_numbers);
				if (this->atom_numbers > 0 && this->atom_numbers != atom_numbers)
				{
					controller->printf("        Error: atom_numbers is not equal: %d %d\n", this->atom_numbers, atom_numbers);
					getchar();
					exit(1);
				}
				else if (this->atom_numbers == 0)
				{
					this->atom_numbers = atom_numbers;
				}
				Malloc_Safely((void**)&h_mass, sizeof(float)* atom_numbers);
				Malloc_Safely((void**)&h_mass_inverse, sizeof(float)* atom_numbers);
				Cuda_Malloc_Safely((void**)&d_mass, sizeof(float)* atom_numbers);
				Cuda_Malloc_Safely((void**)&d_mass_inverse, sizeof(float)* atom_numbers);
			}
			if (strcmp(temp_first_str, "%FLAG") == 0
				&& strcmp(temp_second_str, "MASS") == 0)
			{
				fgets(temps, CHAR_LENGTH_MAX, parm);
				double lin;
				sys.total_mass = 0;
				for (int i = 0; i < this->atom_numbers; i = i + 1)
				{
					fscanf(parm, "%lf\n", &lin);
					this->h_mass[i] = (float)lin;
					if (h_mass[i] == 0)
						h_mass_inverse[i] = 0;
					else
						h_mass_inverse[i] = 1.0f / h_mass[i];
					sys.total_mass += h_mass[i];
				}
			}
		}
		controller[0].printf("    End reading mass from AMBER parm7\n\n");
		fclose(parm);
	}
	else if (atom_numbers > 0)
	{
		controller[0].printf("    mass is set to 20 as default\n");
		sys.total_mass = 0;
		Malloc_Safely((void**)&h_mass, sizeof(float)* atom_numbers);
		Malloc_Safely((void**)&h_mass_inverse, sizeof(float)* atom_numbers);
		Cuda_Malloc_Safely((void**)&d_mass, sizeof(float)* atom_numbers);
		Cuda_Malloc_Safely((void**)&d_mass_inverse, sizeof(float)* atom_numbers);
		for (int i = 0; i < atom_numbers; i++)
		{
			h_mass[i] = 20;
			h_mass_inverse[i] = 1.0 / h_mass[i];
			sys.total_mass += h_mass[i];
		}
	}
	else
	{
		controller[0].printf("    Error: failed to initialze mass, because no atom_numbers found\n");
		getchar();
		exit(1);
	}
	if (atom_numbers > 0)
	{
		hipMemcpy(d_mass, h_mass, sizeof(float)* atom_numbers, hipMemcpyHostToDevice);
		hipMemcpy(d_mass_inverse, h_mass_inverse, sizeof(float)* atom_numbers, hipMemcpyHostToDevice);
	}
}

void MD_INFORMATION::Read_Charge(CONTROLLER *controller)
{
	if (controller[0].Command_Exist("charge_in_file"))
	{
		FILE *fp = NULL;
		controller->printf("    Start reading charge:\n");
		Open_File_Safely(&fp, controller[0].Command("charge_in_file"), "r");
		int atom_numbers = 0;
		char lin[CHAR_LENGTH_MAX];
		fgets(lin, CHAR_LENGTH_MAX, fp);
		int scanf_ret = sscanf(lin, "%d", &atom_numbers);
		if (this->atom_numbers > 0 && this->atom_numbers != atom_numbers)
		{
			controller->printf("        Error: atom_numbers is not equal: %d %d\n", this->atom_numbers, atom_numbers);
			getchar();
			exit(1);
		}
		else if (this->atom_numbers == 0)
		{
			this->atom_numbers = atom_numbers;
		}
		Malloc_Safely((void**)&h_charge, sizeof(float)* atom_numbers);
		Cuda_Malloc_Safely((void**)&d_charge, sizeof(float)* atom_numbers);
		for (int i = 0; i < atom_numbers; i++)
		{
			fscanf(fp, "%f", &h_charge[i]);
		}
		controller->printf("    End reading charge\n\n");
		fclose(fp);
	}
	else if (controller[0].Command_Exist("amber_parm7"))
	{
		FILE *parm = NULL;
		Open_File_Safely(&parm, controller[0].Command("amber_parm7"), "r");
		controller[0].printf("    Start reading charge from AMBER parm7:\n");
		while (true)
		{
			char temps[CHAR_LENGTH_MAX];
			char temp_first_str[CHAR_LENGTH_MAX];
			char temp_second_str[CHAR_LENGTH_MAX];
			if (fgets(temps, CHAR_LENGTH_MAX, parm) == NULL)
			{
				break;
			}
			if (sscanf(temps, "%s %s", temp_first_str, temp_second_str) != 2)
			{
				continue;
			}
			//read in atomnumber atomljtypenumber
			if (strcmp(temp_first_str, "%FLAG") == 0
				&& strcmp(temp_second_str, "POINTERS") == 0)
			{
				fgets(temps, CHAR_LENGTH_MAX, parm);

				int atom_numbers = 0;
				fscanf(parm, "%d", &atom_numbers);
				if (this->atom_numbers > 0 && this->atom_numbers != atom_numbers)
				{
					controller->printf("        Error: atom_numbers is not equal: %d %d\n", this->atom_numbers, atom_numbers);
					getchar();
					exit(1);
				}
				else if (this->atom_numbers == 0)
				{
					this->atom_numbers = atom_numbers;
				}
				Malloc_Safely((void**)&h_charge, sizeof(float)* atom_numbers);
				Cuda_Malloc_Safely((void**)&d_charge, sizeof(float)* atom_numbers);
			}
			if (strcmp(temp_first_str, "%FLAG") == 0
				&& strcmp(temp_second_str, "CHARGE") == 0)
			{
				fgets(temps, CHAR_LENGTH_MAX, parm);
				double lin;
				for (int i = 0; i < this->atom_numbers; i = i + 1)
				{
					fscanf(parm, "%f", &h_charge[i]);
				}
			}
		}
		controller[0].printf("    End reading charge from AMBER parm7\n\n");
		fclose(parm);
	}
	else if (atom_numbers > 0)
	{
		controller[0].printf("    charge is set to 0 as default\n");
		Malloc_Safely((void**)&h_charge, sizeof(float)* atom_numbers);
		Cuda_Malloc_Safely((void**)&d_charge, sizeof(float)* atom_numbers);
		for (int i = 0; i < atom_numbers; i++)
		{
			h_charge[i] = 0;
		}
	}
	else
	{
		controller[0].printf("    Error: failed to initialze charge, because no atom_numbers found\n");
		getchar();
		exit(1);
	}
	if (atom_numbers > 0)
	{
		hipMemcpy(d_charge, h_charge, sizeof(float)* atom_numbers, hipMemcpyHostToDevice);
	}
}

//MD_INFORMATION成员函数
void MD_INFORMATION::Initial(CONTROLLER *controller)
{
	controller->printf("START INITIALZING MD CORE:\n");
	atom_numbers = 0; //初始化，使得能够进行所有原子数目是否相等的判断
	
	strcpy(md_name, controller[0].Command("md_name"));
	Read_Mode(controller);
	Read_dt(controller);

	Read_Coordinate_And_Velocity(controller);
	
	
	Read_Mass(controller);
	Read_Charge(controller);
    
	sys.Initial(controller, this);  //!需要先初始化坐标和速度
	nb.Initial(controller, this);
	
	output.Initial(controller, this);

	nve.Initial(controller, this);
	
	res.Initial(controller, this);

	pbc.Initial(controller, sys.box_length);
	
	Atom_Information_Initial();

	is_initialized = 1;
	controller->printf("    structure last modify date is %d\n", last_modify_date);
	controller->printf("END INITIALZING MD CORE\n\n");
}

void MD_INFORMATION::Atom_Information_Initial()
{
	Malloc_Safely((void**)&this->force, sizeof(VECTOR)*this->atom_numbers);
	Malloc_Safely((void**)&this->h_atom_energy, sizeof(float)* atom_numbers);
	Malloc_Safely((void**)&this->h_atom_virial, sizeof(double)* atom_numbers);
	Cuda_Malloc_Safely((void**)&this->acc, sizeof(VECTOR)*this->atom_numbers);
	Cuda_Malloc_Safely((void**)&this->frc, sizeof(VECTOR)*this->atom_numbers);
	Cuda_Malloc_Safely((void **)&this->uint_crd, sizeof(UNSIGNED_INT_VECTOR)*this->atom_numbers);
	Cuda_Malloc_Safely((void**)&this->d_atom_energy, sizeof(float)* atom_numbers);
	Cuda_Malloc_Safely((void**)&this->d_atom_virial, sizeof(float)* atom_numbers);
	Cuda_Malloc_Safely((void**)&this->d_atom_ek, sizeof(float)* atom_numbers);
	Reset_List << <ceilf((float)3.*this->atom_numbers / 32), 32 >> >
		(3 * this->atom_numbers, (float*)this->acc, 0.);
	Reset_List << <ceilf((float)3.*this->atom_numbers / 32), 32 >> >
		(3 * this->atom_numbers, (float*)this->frc, 0.);
	sys.freedom = 3 * atom_numbers; //最大自由度，后面减
}

void MD_INFORMATION::Read_Coordinate_In_File(const char* file_name, CONTROLLER controller)
{
	FILE *fp =NULL;
	controller.printf("    Start reading coordinate_in_file:\n");
	Open_File_Safely(&fp, file_name, "r");
	char lin[CHAR_LENGTH_MAX];
	fgets(lin, CHAR_LENGTH_MAX, fp);
	int atom_numbers = 0;
	int scanf_ret = sscanf(lin, "%d %lf", &atom_numbers, &sys.start_time);
	if (this->atom_numbers > 0 && this->atom_numbers != atom_numbers)
	{
		controller.printf("        Error: atom_numbers is not equal: %d %d\n", this->atom_numbers, atom_numbers);
		getchar();
		exit(1);
	}
	else if (this->atom_numbers == 0)
	{
		this->atom_numbers = atom_numbers;
	}
	if (scanf_ret == 0)
	{
		controller.printf("        Error: Atom_numbers not found.\n");
		getchar();
		exit(1);
	}
	else  if (scanf_ret == 1)
	{
		sys.start_time = 0;
	}

	controller.printf("        atom_numbers is %d\n", this->atom_numbers);
	controller.printf("        system start_time is %lf\n", this->sys.start_time);
	Malloc_Safely((void**)&coordinate, sizeof(VECTOR)*this->atom_numbers);
	Cuda_Malloc_Safely((void**)&crd, sizeof(VECTOR)*this->atom_numbers);

	for (int i = 0; i < atom_numbers; i++)
	{
		fscanf(fp, "%f %f %f", &coordinate[i].x, &coordinate[i].y, &coordinate[i].z);
	}
	fscanf(fp, "%f %f %f", &sys.box_length.x, &sys.box_length.y, &sys.box_length.z);
	controller.printf("        box_length is\n            x: %f\n            y: %f\n            z: %f\n", sys.box_length.x, sys.box_length.y, sys.box_length.z);
	hipMemcpy(crd, coordinate, sizeof(VECTOR)* atom_numbers, hipMemcpyHostToDevice);
	controller.printf("    End reading coordinate_in_file\n\n");
	fclose(fp);
}
void MD_INFORMATION::Read_Rst7(const char* file_name, int irest, CONTROLLER controller)
{
	FILE *fin = NULL;
	Open_File_Safely(&fin, file_name, "r");
	controller.printf("    Start reading AMBER rst7:\n");
	char lin[CHAR_LENGTH_MAX];
	int atom_numbers = 0;
	fgets(lin, CHAR_LENGTH_MAX, fin);
	fgets(lin, CHAR_LENGTH_MAX, fin);
	int has_vel = 0;
	int scanf_ret = sscanf(lin, "%d %lf", &atom_numbers, &sys.start_time);
	if (this->atom_numbers > 0 && this->atom_numbers != atom_numbers)
	{
		controller.printf("        Error: atom_numbers is not equal: %d %d\n", this->atom_numbers, atom_numbers);
		getchar();
		exit(1);
	}
	else if (this->atom_numbers == 0)
	{
		this->atom_numbers = atom_numbers;
	}
	if (scanf_ret == 0)
	{
		controller.printf("        Error: Atom_numbers not found.\n");
		getchar();
		exit(1);
	}
	else  if (scanf_ret == 2)
	{
		has_vel = 1;
	}
	else
	{
		sys.start_time = 0;
	}

	Malloc_Safely((void**)&coordinate, sizeof(VECTOR)*this->atom_numbers);
	Cuda_Malloc_Safely((void**)&crd, sizeof(VECTOR)*this->atom_numbers);
	Malloc_Safely((void**)&velocity, sizeof(VECTOR)*this->atom_numbers);
	Cuda_Malloc_Safely((void**)&vel, sizeof(VECTOR)*this->atom_numbers);

	controller.printf("        atom_numbers is %d\n", this->atom_numbers);
	controller.printf("        system start time is %lf\n", this->sys.start_time);

	if (has_vel == 0 || irest == 0)
	{
		controller.printf("        All velocity will be set to 0\n");
	}


	for (int i = 0; i < this->atom_numbers; i = i + 1)
	{
		fscanf(fin, "%f %f %f",
			&this->coordinate[i].x,
			&this->coordinate[i].y,
			&this->coordinate[i].z);
	}
	if (has_vel)
	{
		for (int i = 0; i < this->atom_numbers; i = i + 1)
		{
			fscanf(fin, "%f %f %f",
				&this->velocity[i].x,
				&this->velocity[i].y,
				&this->velocity[i].z);
		}
	}
	if (irest == 0 || !has_vel)
	{
		for (int i = 0; i < this->atom_numbers; i = i + 1)
		{
			this->velocity[i].x = 0.0;
			this->velocity[i].y = 0.0;
			this->velocity[i].z = 0.0;
		}
	}
	fscanf(fin, "%f %f %f", &this->sys.box_length.x, &this->sys.box_length.y, &this->sys.box_length.z);
	controller.printf("        system size is %f %f %f\n", this->sys.box_length.x, this->sys.box_length.y, this->sys.box_length.z);
	hipMemcpy(this->crd, this->coordinate, sizeof(VECTOR)*this->atom_numbers, hipMemcpyHostToDevice);
	hipMemcpy(this->vel, this->velocity, sizeof(VECTOR)*this->atom_numbers, hipMemcpyHostToDevice);
	//in some bad rst7, the coordinates will be extremly bad, so need a full box map
	for (int i = 0; i < 10; i = i + 1)
	{
		Crd_Periodic_Map << <ceilf((float)this->atom_numbers / 32), 32 >> >
			(this->atom_numbers, this->crd, this->sys.box_length);
	}
	fclose(fin);
	controller.printf("    End reading AMBER rst7\n\n");
	
}

void MD_INFORMATION::trajectory_output::Append_Crd_Traj_File(FILE *fp)
{
	if (md_info->is_initialized)
	{
		md_info->Crd_Vel_Device_To_Host();
		if (fp == NULL)
		{
			fp = crd_traj;
		}
		fwrite(&md_info->coordinate[0].x, sizeof(VECTOR), md_info->atom_numbers, fp);
	}
}

void MD_INFORMATION::trajectory_output::Append_Box_Traj_File(FILE *fp)
{
	if (md_info->is_initialized)
	{
		if (fp == NULL)
		{
			fp = box_traj;
		}
		fprintf(fp, "%f %f %f %.0f %.0f %.0f\n", md_info->sys.box_length.x, md_info->sys.box_length.y, md_info->sys.box_length.z, 90.0f, 90.0f, 90.0f);
	}
}

void MD_INFORMATION::trajectory_output::Export_Restart_File(const char *rst7_name)
{
	if (!md_info->is_initialized)
		return;

	char filename[CHAR_LENGTH_MAX];
	if (rst7_name == NULL)
		strcpy(filename, restart_name);
	else
		strcpy(filename, rst7_name);
	md_info->Crd_Vel_Device_To_Host();
	if (amber_irest >= 0)
	{
		const char *sys_name = md_info->md_name;
		FILE *lin = NULL;
		Open_File_Safely(&lin, filename, "w");
		fprintf(lin, "%s\n", sys_name);
		fprintf(lin, "%8d %.3lf\n", md_info->atom_numbers, md_info->sys.Get_Current_Time());
		int s = 0;
		for (int i = 0; i < md_info->atom_numbers; i = i + 1)
		{
			fprintf(lin, "%12.7f%12.7f%12.7f", md_info->coordinate[i].x, md_info->coordinate[i].y, md_info->coordinate[i].z);
			s = s + 1;
			if (s == 2)
			{
				s = 0;
				fprintf(lin, "\n");
			}
		}
		if (s == 1)
		{
			s = 0;
			fprintf(lin, "\n");
		}
		for (int i = 0; i < md_info->atom_numbers; i = i + 1)
		{
			fprintf(lin, "%12.7f%12.7f%12.7f", md_info->velocity[i].x, md_info->velocity[i].y, md_info->velocity[i].z);
			s = s + 1;
			if (s == 2)
			{
				s = 0;
				fprintf(lin, "\n");
			}
		}
		if (s == 1)
		{
			s = 0;
			fprintf(lin, "\n");
		}
		fprintf(lin, "%12.7f%12.7f%12.7f", (float)md_info->sys.box_length.x, (float)md_info->sys.box_length.y, (float)md_info->sys.box_length.z);
		fprintf(lin, "%12.7f%12.7f%12.7f", (float)90., (float)90., (float)90.);
		fclose(lin);
	}
	else
	{
		FILE *lin = NULL;
		FILE *lin2 = NULL;
		char buffer[CHAR_LENGTH_MAX];
		sprintf(buffer, "%s_%s.txt", filename, "coordinate");
		Open_File_Safely(&lin, buffer, "w");
		sprintf(buffer, "%s_%s.txt", filename, "velocity");
		Open_File_Safely(&lin2, buffer, "w");
		fprintf(lin, "%d %.3lf\n", md_info->atom_numbers, md_info->sys.Get_Current_Time());
		fprintf(lin2, "%d %.3lf\n", md_info->atom_numbers, md_info->sys.Get_Current_Time());
		for (int i = 0; i < md_info->atom_numbers; i++)
		{
			fprintf(lin, "%12.7f %12.7f %12.7f\n", md_info->coordinate[i].x, md_info->coordinate[i].y, md_info->coordinate[i].z);
			fprintf(lin2, "%12.7f %12.7f %12.7f\n", md_info->velocity[i].x, md_info->velocity[i].y, md_info->velocity[i].z);
		}
		fprintf(lin, "%12.7f %12.7f %12.7f %12.7f %12.7f %12.7f", md_info->sys.box_length.x, md_info->sys.box_length.y, md_info->sys.box_length.z, 90.0f, 90.0f, 90.0f);
		fclose(lin);
		fclose(lin2);
	}
}


void MD_INFORMATION::Update_Volume(double factor)
{
	double f_inv = 1.0 / factor;

	sys.box_length = factor * sys.box_length;
	pbc.crd_to_uint_crd_cof = CONSTANT_UINT_MAX_FLOAT / sys.box_length;
	pbc.quarter_crd_to_uint_crd_cof = 0.25 * pbc.crd_to_uint_crd_cof;
	pbc.uint_dr_to_dr_cof = 1.0f / pbc.crd_to_uint_crd_cof;
	MD_Information_Crd_To_Uint_Crd();
}


float MD_INFORMATION::system_information::Get_Density()
{
	density = total_mass * 1e24f / 6.023e23f / Get_Volume();
	return density;
}

double MD_INFORMATION::system_information::Get_Current_Time()
{
	current_time = start_time + (double)dt_in_ps * steps;
	return current_time;
}

float MD_INFORMATION::system_information::Get_Volume()
{
	volume = box_length.x * box_length.y * box_length.z;
	return volume;
}

void MD_INFORMATION::MD_Information_Crd_To_Uint_Crd()
{
	Crd_To_Uint_Crd << <ceilf((float)this->atom_numbers / 128), 128 >> >
		(this->atom_numbers, pbc.quarter_crd_to_uint_crd_cof, crd, uint_crd);
}


void MD_INFORMATION::NVE_iteration::Leap_Frog()
{
	if (max_velocity <= 0)
	{
		MD_Iteration_Leap_Frog << <ceilf((float)md_info->atom_numbers / 128), 128 >> >
			(md_info->atom_numbers, md_info->vel, md_info->crd, md_info->frc, md_info->acc, md_info->d_mass_inverse, md_info->dt);
	}
	else
	{
		MD_Iteration_Leap_Frog_With_Max_Velocity << <ceilf((float)md_info->atom_numbers / 128), 128 >> >
			(md_info->atom_numbers, md_info->vel, md_info->crd, md_info->frc, md_info->acc, md_info->d_mass_inverse, md_info->dt, max_velocity);
	}
}

void MD_INFORMATION::MD_Information_Gradient_Descent()
{
	MD_Iteration_Gradient_Descent << <ceilf((float)this->atom_numbers / 128), 128 >> >
		(this->atom_numbers, this->crd, this->frc, dt * dt);
	Reset_List << <ceilf((float)this->atom_numbers * 3 / 128), 128 >> > (atom_numbers * 3, (float*)vel, 0);
}

void MD_INFORMATION::NVE_iteration::Velocity_Verlet_1()
{
	MD_Iteration_Speed_Verlet_1 << <ceilf((float)md_info->atom_numbers / 32), 32 >> >
		(md_info->atom_numbers, 0.5*md_info->dt, md_info->dt, md_info->acc, md_info->vel, md_info->crd, md_info->frc);
}

void MD_INFORMATION::NVE_iteration::Velocity_Verlet_2()
{
	if (max_velocity <= 0)
	{
		MD_Iteration_Speed_Verlet_2 << <ceilf((float)md_info->atom_numbers / 32), 32 >> >
			(md_info->atom_numbers, 0.5*md_info->dt, md_info->d_mass_inverse, md_info->frc, md_info->vel, md_info->acc);
	}
	else
	{
		MD_Iteration_Speed_Verlet_2_With_Max_Velocity << <ceilf((float)md_info->atom_numbers / 32), 32 >> >
			(md_info->atom_numbers, 0.5*md_info->dt, md_info->d_mass_inverse, md_info->frc, md_info->vel, md_info->acc, max_velocity);
	}
}


float MD_INFORMATION::system_information::Get_Total_Atom_Ek(int is_download)
{
	MD_Atom_Ek << <ceilf((float)md_info->atom_numbers / 32.), 32 >> >(md_info->atom_numbers, md_info->d_atom_ek, md_info->vel, md_info->d_mass);
	Sum_Of_List << <1, 1024 >> >(md_info->atom_numbers, md_info->d_atom_ek, d_sum_of_atom_ek);
	if (is_download)
	{
		hipMemcpy(&h_sum_of_atom_ek, d_sum_of_atom_ek, sizeof(float), hipMemcpyDeviceToHost);
		return h_sum_of_atom_ek;
	}
	else
	{
		return 0;
	}	
}

float MD_INFORMATION::system_information::Get_Atom_Temperature()
{
	h_temperature = Get_Total_Atom_Ek() * 2. / CONSTANT_kB / freedom;
	return h_temperature;
}

float MD_INFORMATION::residue_information::Get_Total_Residue_Ek(int is_download)
{
	MD_Residue_Ek << <ceilf((float)residue_numbers / 32.), 32 >> >
		(residue_numbers, d_res_start, d_res_end, res_ek_energy,
		md_info->vel, md_info->d_mass);
	Sum_Of_List << <1, 1024 >> >(residue_numbers, res_ek_energy, sigma_of_res_ek);
	if (is_download)
	{
		hipMemcpy(h_sigma_of_res_ek, sigma_of_res_ek, sizeof(float), hipMemcpyDeviceToHost);
		return h_sigma_of_res_ek[0];
	}
	else
	{
		return 0;
	}
}

float MD_INFORMATION::residue_information::Get_Residue_Temperature()
{
	h_temperature = Get_Total_Residue_Ek() * 2. / CONSTANT_kB / residue_numbers / 3;
	return h_temperature;
}

void MD_INFORMATION::residue_information::Residue_Crd_Map(VECTOR *no_wrap_crd, float scaler)
{
	Get_Center_Of_Mass << <20, 32 >> >(residue_numbers, d_res_start, d_res_end, no_wrap_crd, md_info->d_mass, d_mass_inverse, d_center_of_mass);
	Map_Center_Of_Mass << <20, { 32, 4 } >> >(residue_numbers, d_res_start, d_res_end, scaler, d_center_of_mass, md_info->sys.box_length, no_wrap_crd, md_info->crd);
}


void MD_INFORMATION::MD_Reset_Atom_Energy_And_Virial()
{
	need_potential = 0;
	Reset_List << <ceilf((float)atom_numbers / 1024.0f), 1024 >> >(atom_numbers, d_atom_energy, 0.0f);
	Reset_List << <1, 1 >> >(1, sys.d_potential, 0.0f);

	need_pressure = 0;
	Reset_List << <ceilf((float)atom_numbers / 1024.0f), 1024 >> >(atom_numbers, d_atom_virial, 0.0f);
	Reset_List << <1, 1 >> >(1, sys.d_virial, 0.0f);

	Reset_List << <ceilf((float)3 * atom_numbers / 1024.0f), 1024 >> >(3 * atom_numbers, (float*)frc, 0.0f);
}


void MD_INFORMATION::Calculate_Pressure_And_Potential_If_Needed(int is_download)
{
	if (need_pressure > 0)
	{
		sys.Get_Pressure(is_download);
	}
	if (need_potential > 0)
	{
		sys.Get_Potential(is_download);
	}
}

float MD_INFORMATION::system_information::Get_Pressure(int is_download)
{
	//计算动能
	MD_Atom_Ek << <ceilf((float)md_info->atom_numbers / 32.), 32 >> >(md_info->atom_numbers, md_info->d_atom_ek, md_info->vel, md_info->d_mass);
	Sum_Of_List << <1, 1024 >> >(md_info->atom_numbers, md_info->d_atom_ek, d_sum_of_atom_ek);

	//计算维里
	Add_Sum_List << <1, 1024 >> >(md_info->atom_numbers, md_info->d_atom_virial, d_virial);

	//合并起来
	Calculate_Pressure_Cuda << <1, 1 >> >(1.0 / Get_Volume(), d_sum_of_atom_ek, d_virial, d_pressure);

	if (is_download)
	{
		hipMemcpy(&h_pressure, d_pressure, sizeof(float), hipMemcpyDeviceToHost);
		return h_pressure;
	}
	else
	{
		return 0;
	}
}

float MD_INFORMATION::system_information::Get_Potential(int is_download)
{

	Add_Sum_List << <1, 1024 >> >(md_info->atom_numbers, md_info->d_atom_energy, d_potential);

	if (is_download)
	{
		hipMemcpy(&h_potential, d_potential, sizeof(float), hipMemcpyDeviceToHost);
		return h_potential;
	}
	else
	{
		return 0;
	}
}

void MD_INFORMATION::MD_Information_Frc_Device_To_Host()
{
	hipMemcpy(this->force, this->frc, sizeof(VECTOR)*this->atom_numbers, hipMemcpyDeviceToHost);
}

void MD_INFORMATION::MD_Information_Frc_Host_To_Device()
{
	hipMemcpy(this->frc, this->force, sizeof(VECTOR)*this->atom_numbers, hipMemcpyHostToDevice);
}

void MD_INFORMATION::Crd_Vel_Device_To_Host(int Do_Translation, int forced)
{
	if (output.current_crd_synchronized_step != sys.steps || forced)
	{	
		output.current_crd_synchronized_step = sys.steps;
		if (Do_Translation)
		{	
			hipMemcpy(coordinate, crd, sizeof(VECTOR) * atom_numbers, hipMemcpyDeviceToHost);
		}
		else
		{
			hipMemcpy(this->coordinate, this->crd, sizeof(VECTOR)*this->atom_numbers, hipMemcpyDeviceToHost);
		}
		hipMemcpy(this->velocity, this->vel, sizeof(VECTOR)*this->atom_numbers, hipMemcpyDeviceToHost);
	}
}

void MD_INFORMATION::Clear()
{

}
