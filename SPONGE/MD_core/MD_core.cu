#include "hip/hip_runtime.h"
#include "MD_core.cuh"

#define BOX_TRAJ_COMMAND "box"
#define BOX_TRAJ_DEFAULT_FILENAME "mdbox.txt"
#define TRAJ_COMMAND "crd"
#define TRAJ_DEFAULT_FILENAME "mdcrd.dat"
#define RESTART_COMMAND "rst"
#define RESTART_DEFAULT_FILENAME "restart"
//20210827用于输出速度和力
#define FRC_TRAJ_COMMAND "frc"
#define VEL_TRAJ_COMMAND "vel"

static __global__ void MD_Iteration_Leap_Frog
(const int atom_numbers, VECTOR *vel, VECTOR *crd, VECTOR *frc, VECTOR *acc, const float *inverse_mass, const float dt)
{
	int i = blockDim.x*blockIdx.x + threadIdx.x;
	if (i < atom_numbers)
	{
		acc[i].x = inverse_mass[i] * frc[i].x;
		acc[i].y = inverse_mass[i] * frc[i].y;
		acc[i].z = inverse_mass[i] * frc[i].z;

		vel[i].x = vel[i].x + dt*acc[i].x;
		vel[i].y = vel[i].y + dt*acc[i].y;
		vel[i].z = vel[i].z + dt*acc[i].z;

		crd[i].x = crd[i].x + dt*vel[i].x;
		crd[i].y = crd[i].y + dt*vel[i].y;
		crd[i].z = crd[i].z + dt*vel[i].z;

	}
}

static __global__ void MD_Iteration_Leap_Frog_With_Max_Velocity
(const int atom_numbers, VECTOR *vel, VECTOR *crd, VECTOR *frc, VECTOR *acc, const float *inverse_mass, const float dt, const float max_velocity)
{
	int i = blockDim.x*blockIdx.x + threadIdx.x;
	if (i < atom_numbers)
	{
		VECTOR acc_i = inverse_mass[i] * frc[i];
		VECTOR vel_i = vel[i] + dt * acc_i;
		vel_i = Make_Vector_Not_Exceed_Value(vel_i, max_velocity);
		vel[i] = vel_i;
		crd[i] = crd[i] + dt * vel_i;

	}
}

static __global__ void MD_Iteration_Gradient_Descent
(const int atom_numbers, VECTOR *crd, VECTOR *frc, const float *mass_inverse, const float dt, VECTOR *vel, const float momentum_keep)
{
	int i = blockDim.x*blockIdx.x + threadIdx.x;
	if (i < atom_numbers)
	{
		vel[i] = momentum_keep * vel[i] + dt * mass_inverse[i] * frc[i];
		crd[i] = dt * vel[i];
	}
}

static __global__ void MD_Iteration_Gradient_Descent_With_Max_Move
(const int atom_numbers, VECTOR *crd, VECTOR *frc, const float *mass_inverse, const float dt, VECTOR *vel, const float momentum_keep, float max_move)
{
	int i = blockDim.x*blockIdx.x + threadIdx.x;
	if (i < atom_numbers)
	{
		vel[i] = momentum_keep * vel[i] + dt * mass_inverse[i] * frc[i];
		VECTOR move = dt * vel[i];
		Make_Vector_Not_Exceed_Value(move, max_move);
		crd[i] = crd[i] + move;
	}
}

static __global__ void MD_Iteration_Speed_Verlet_1(const int atom_numbers, const float half_dt, const float dt, const VECTOR *acc, VECTOR *vel, VECTOR *crd, VECTOR *frc)
{
	int i = blockDim.x*blockIdx.x + threadIdx.x;
	if (i < atom_numbers)
	{
		vel[i].x = vel[i].x + half_dt*acc[i].x;
		vel[i].y = vel[i].y + half_dt*acc[i].y;
		vel[i].z = vel[i].z + half_dt*acc[i].z;
		crd[i].x = crd[i].x + dt*vel[i].x;
		crd[i].y = crd[i].y + dt*vel[i].y;
		crd[i].z = crd[i].z + dt*vel[i].z;
	}
}

static __global__ void MD_Iteration_Speed_Verlet_2(const int atom_numbers, const float half_dt, const float *inverse_mass, const VECTOR *frc, VECTOR *vel, VECTOR *acc)
{
	int i = blockDim.x*blockIdx.x + threadIdx.x;
	if (i < atom_numbers)
	{
		acc[i].x = inverse_mass[i] * frc[i].x;
		acc[i].y = inverse_mass[i] * frc[i].y;
		acc[i].z = inverse_mass[i] * frc[i].z;
		vel[i].x = vel[i].x + half_dt*acc[i].x;
		vel[i].y = vel[i].y + half_dt*acc[i].y;
		vel[i].z = vel[i].z + half_dt*acc[i].z;
	}
}

static __global__ void MD_Iteration_Speed_Verlet_2_With_Max_Velocity(const int atom_numbers, const float half_dt, const float *inverse_mass, const VECTOR *frc, VECTOR *vel, VECTOR *acc, const float max_velocity)
{
	int i = blockDim.x*blockIdx.x + threadIdx.x;
	if (i < atom_numbers)
	{
		VECTOR acc_i = inverse_mass[i] * frc[i];
		VECTOR vel_i = vel[i] + half_dt * acc_i;

		vel[i] = Make_Vector_Not_Exceed_Value(vel_i, max_velocity);
		acc[i] = acc_i;
	}
}


static __global__ void Get_Center_Of_Mass(const int residue_numbers, const int *start, const int *end,
	const VECTOR *crd, const float *atom_mass, const float *residue_mass_inverse, VECTOR *center_of_mass)
{
	for (int residue_i = blockDim.x*blockIdx.x + threadIdx.x; residue_i < residue_numbers; residue_i += gridDim.x * blockDim.x)
	{
		VECTOR com_lin = { 0.0f, 0.0f, 0.0f };
		for (int atom_i = start[residue_i]; atom_i < end[residue_i]; atom_i += 1)
		{
			com_lin = com_lin + atom_mass[atom_i] * crd[atom_i];
		}
		center_of_mass[residue_i] = residue_mass_inverse[residue_i] * com_lin;
	}
}

static __global__ void Map_Center_Of_Mass(const int residue_numbers, const int *start, const int *end,
	const float scaler, const VECTOR *center_of_mass, const VECTOR box_length, const VECTOR *no_wrap_crd, VECTOR *crd)
{
	VECTOR trans_vec;
	VECTOR com;
	for (int residue_i = blockDim.x*blockIdx.x + threadIdx.x; residue_i < residue_numbers; residue_i += gridDim.x * blockDim.x)
	{
		com = center_of_mass[residue_i];

		trans_vec.x = com.x - floorf(com.x / box_length.x) * box_length.x;
		trans_vec.y = com.y - floorf(com.y / box_length.y) * box_length.y;
		trans_vec.z = com.z - floorf(com.z / box_length.z) * box_length.z;
		trans_vec = scaler * trans_vec - com;

		for (int atom_i = start[residue_i] + threadIdx.y; atom_i < end[residue_i]; atom_i += blockDim.y)
		{
			crd[atom_i] = no_wrap_crd[atom_i] + trans_vec;
		}
	}
}

static __global__ void Add_Sum_List(int n, float *atom_virial, float *sum_virial)
{
	float temp = 0;
	for (int i = threadIdx.x; i < n; i = i + blockDim.x)
	{
		temp = temp + atom_virial[i];
	}
	atomicAdd(sum_virial, temp);
}

static __global__ void Calculate_Pressure_Cuda(const float V_inverse, const float *ek, const float *virial, float *pressure)
{
	pressure[0] = (ek[0] * 2 + virial[0]) * 0.33333333333333f * V_inverse;
}



/*
static __global__ void MD_Temperature
(const int residue_numbers, const int *start, const int *end, float *ek,
const VECTOR *atom_vel, const float *atom_mass)
{
	int residue_i = blockDim.x*blockIdx.x + threadIdx.x;
	if (residue_i < residue_numbers)
	{
		VECTOR momentum = { 0., 0., 0. };
		float res_mass = 0.; //待提出，只需要初始时计算一遍
		int s = start[residue_i];
		int e = end[residue_i];
		float mass_lin;
		for (int atom_i = s; atom_i < e; atom_i = atom_i + 1)
		{
			mass_lin = atom_mass[atom_i];

			momentum.x = momentum.x + mass_lin*atom_vel[atom_i].x;
			momentum.y = momentum.y + mass_lin*atom_vel[atom_i].y;
			momentum.z = momentum.z + mass_lin*atom_vel[atom_i].z;
			res_mass = res_mass + mass_lin;
		}
		ek[residue_i] = 0.5*(momentum.x*momentum.x + momentum.y*momentum.y + momentum.z*momentum.z) / res_mass * 2. / 3. / CONSTANT_kB / residue_numbers;
	}
}
*/
static __global__ void MD_Residue_Ek
(const int residue_numbers,const int *start,const int *end,float *ek,
const VECTOR *atom_vel,const float *atom_mass)
{
	int residue_i = blockDim.x*blockIdx.x + threadIdx.x;
	if (residue_i < residue_numbers)
	{
		VECTOR momentum = {0.,0.,0.};
		float res_mass = 0.; //待提出，只需要初始时计算一遍
		int s = start[residue_i];
		int e = end[residue_i];
		float mass_lin;
		for (int atom_i = s; atom_i < e; atom_i = atom_i + 1)
		{
			mass_lin = atom_mass[atom_i];

			momentum.x = momentum.x + mass_lin*atom_vel[atom_i].x;
			momentum.y = momentum.y + mass_lin*atom_vel[atom_i].y;
			momentum.z = momentum.z + mass_lin*atom_vel[atom_i].z;
			res_mass = res_mass + mass_lin;
		}
		ek[residue_i] = 0.5*(momentum.x*momentum.x + momentum.y*momentum.y + momentum.z*momentum.z) / res_mass;
	}
}

static __global__ void MD_Atom_Ek
(const int atom_numbers, float *ek, const VECTOR *atom_vel, const float *atom_mass)
{
	int atom_i = blockDim.x*blockIdx.x + threadIdx.x;
	if (atom_i < atom_numbers)
	{
		VECTOR v = atom_vel[atom_i];
		ek[atom_i] = 0.5 * v * v * atom_mass[atom_i];
	}
}

void MD_INFORMATION::system_information::Initial(CONTROLLER *controller, MD_INFORMATION *md_info)
{
	this->md_info = md_info;
	steps = 0;
	step_limit = 1000;
	if (controller[0].Command_Exist("step_limit"))
	{
		step_limit = atoi(controller[0].Command("step_limit"));
	}

	target_temperature = 300.0f;
	if (md_info->mode >= md_info->NVT && controller[0].Command_Exist("target_temperature"))
	{
		target_temperature = atof(controller[0].Command("target_temperature"));
	}

	target_pressure = 1;
	if (md_info->mode == md_info->NPT &&  controller[0].Command_Exist("target_pressure"))
		target_pressure = atof(controller[0].Command("target_pressure"));
	target_pressure *= CONSTANT_PRES_CONVERTION_INVERSE;

	controller->Step_Print_Initial("step", "%d");
	controller->Step_Print_Initial("time", "%.3lf");
	controller->Step_Print_Initial("temperature", "%.2f");
	controller->Step_Print_Initial("potential", "%.2f");
	Cuda_Malloc_Safely((void**)&this->d_virial, sizeof(float));
	Cuda_Malloc_Safely((void**)&this->d_pressure, sizeof(float));
	Cuda_Malloc_Safely((void**)&this->d_temperature, sizeof(float));
	Cuda_Malloc_Safely((void**)&this->d_potential, sizeof(float));
	Cuda_Malloc_Safely((void**)&this->d_sum_of_atom_ek, sizeof(float));
}

void MD_INFORMATION::non_bond_information::Initial(CONTROLLER *controller, MD_INFORMATION *md_info)
{
	if (controller[0].Command_Exist("skin"))
	{
		skin = atof(controller[0].Command("skin"));
	}
	else
	{
		skin = 2.0;
	}
	controller->printf("    skin set to %.2f Angstram\n", skin);

	if (controller[0].Command_Exist("cutoff"))
	{
		cutoff = atof(controller[0].Command("cutoff"));
	}
	else
	{
		cutoff = 10.0;
	}
	controller->printf("    cutoff set to %.2f Angstram\n", cutoff);
	/*===========================
	读取排除表相关信息
	============================*/
	if (controller[0].Command_Exist("exclude_in_file"))
	{
		FILE *fp = NULL;
		controller->printf("    Start reading excluded list:\n");
		Open_File_Safely(&fp, controller[0].Command("exclude_in_file"), "r");
		
		int atom_numbers = 0;
		int scanf_ret = fscanf(fp, "%d %d", &atom_numbers, &excluded_atom_numbers);
		if (md_info->atom_numbers > 0 && md_info->atom_numbers != atom_numbers)
		{
			controller->printf("        Error: atom_numbers is not equal: %d %d\n", md_info->atom_numbers, atom_numbers);
			getchar();
			exit(1);
		}
		else if (md_info->atom_numbers == 0)
		{
			md_info->atom_numbers = atom_numbers;
		}
		controller->printf("        excluded list total length is %d\n", excluded_atom_numbers);

		Cuda_Malloc_Safely((void**)&d_excluded_list_start, sizeof(int)*atom_numbers);
		Cuda_Malloc_Safely((void**)&d_excluded_numbers, sizeof(int)*atom_numbers);
		Cuda_Malloc_Safely((void**)&d_excluded_list, sizeof(int)*excluded_atom_numbers);

		Malloc_Safely((void**)&h_excluded_list_start, sizeof(int)*atom_numbers);
		Malloc_Safely((void**)&h_excluded_numbers, sizeof(int)*atom_numbers);
		Malloc_Safely((void**)&h_excluded_list, sizeof(int)*excluded_atom_numbers);
		int count = 0;
		for (int i = 0; i < atom_numbers; i++)
		{
			scanf_ret = fscanf(fp, "%d", &h_excluded_numbers[i]);
			h_excluded_list_start[i] = count;
			for (int j = 0; j < h_excluded_numbers[i]; j++)
			{
				scanf_ret = fscanf(fp, "%d", &h_excluded_list[count]);
				count++;
			}
		}
		hipMemcpy(d_excluded_list_start, h_excluded_list_start, sizeof(int)*atom_numbers, hipMemcpyHostToDevice);
		hipMemcpy(d_excluded_numbers, h_excluded_numbers, sizeof(int)*atom_numbers, hipMemcpyHostToDevice);
		hipMemcpy(d_excluded_list, h_excluded_list, sizeof(int)*excluded_atom_numbers, hipMemcpyHostToDevice);
		controller->printf("    End reading excluded list\n\n");
		fclose(fp);
	}
	else if (controller[0].Command_Exist("amber_parm7"))
	{
		/*===========================
		从parm中读取排除表相关信息
		============================*/
		FILE *parm = NULL;
		Open_File_Safely(&parm, controller[0].Command("amber_parm7"), "r");
		controller->printf("    Start reading excluded list from AMBER file:\n");
		while (true)
		{
			char temps[CHAR_LENGTH_MAX];
			char temp_first_str[CHAR_LENGTH_MAX];
			char temp_second_str[CHAR_LENGTH_MAX];
			if (!fgets(temps, CHAR_LENGTH_MAX, parm))
			{
				break;
			}
			if (sscanf(temps, "%s %s", temp_first_str, temp_second_str) != 2)
			{
				continue;
			}
			if (strcmp(temp_first_str, "%FLAG") == 0
				&& strcmp(temp_second_str, "POINTERS") == 0)
			{
				char *get_ret = fgets(temps, CHAR_LENGTH_MAX, parm);

				int atom_numbers = 0;
				int scanf_ret = fscanf(parm, "%d\n", &atom_numbers);
				if (md_info->atom_numbers > 0 && md_info->atom_numbers != atom_numbers)
				{
					controller->printf("        Error: atom_numbers is not equal: %d %d\n", md_info->atom_numbers, atom_numbers);
					getchar();
					exit(1);
				}
				else if (md_info->atom_numbers == 0)
				{
					md_info->atom_numbers = atom_numbers;
				}
				Cuda_Malloc_Safely((void**)&d_excluded_list_start, sizeof(int)*atom_numbers);
				Cuda_Malloc_Safely((void**)&d_excluded_numbers, sizeof(int)*atom_numbers);

				Malloc_Safely((void**)&h_excluded_list_start, sizeof(int)*atom_numbers);
				Malloc_Safely((void**)&h_excluded_numbers, sizeof(int)*atom_numbers);
				for (int i = 0; i < 9; i = i + 1)
				{
					scanf_ret = fscanf(parm, "%d\n", &excluded_atom_numbers);
				}
				scanf_ret = fscanf(parm, "%d\n", &excluded_atom_numbers);
				controller->printf("        excluded list total length is %d\n", excluded_atom_numbers);

				Cuda_Malloc_Safely((void**)&d_excluded_list, sizeof(int)*excluded_atom_numbers);
				Malloc_Safely((void**)&h_excluded_list, sizeof(int)*excluded_atom_numbers);
			}

			//read atom_excluded_number for every atom
			if (strcmp(temp_first_str, "%FLAG") == 0
				&& strcmp(temp_second_str, "NUMBER_EXCLUDED_ATOMS") == 0)
			{
				char *get_ret = fgets(temps, CHAR_LENGTH_MAX, parm);
				for (int i = 0; i<md_info->atom_numbers; i = i + 1)
				{
					int scanf_ret = fscanf(parm, "%d\n", &h_excluded_numbers[i]);
				}
			}
			//read every atom's excluded atom list
			if (strcmp(temp_first_str, "%FLAG") == 0
				&& strcmp(temp_second_str, "EXCLUDED_ATOMS_LIST") == 0)
			{
				int count = 0;
				int lin = 0;
				char *get_ret = fgets(temps, CHAR_LENGTH_MAX, parm);
				for (int i = 0; i<md_info->atom_numbers; i = i + 1)
				{
					h_excluded_list_start[i] = count;
					for (int j = 0; j<h_excluded_numbers[i]; j = j + 1)
					{
						int scanf_ret = fscanf(parm, "%d\n", &lin);
						if (lin == 0)
						{
							h_excluded_numbers[i] = 0;
							break;
						}
						else
						{
							h_excluded_list[count] = lin - 1;
							count = count + 1;
						}
					}
					if (h_excluded_numbers[i] > 0)
						thrust::sort(&h_excluded_list[h_excluded_list_start[i]], &h_excluded_list[h_excluded_list_start[i]] + h_excluded_numbers[i]);
				}
			}
		}

		hipMemcpy(d_excluded_list_start, h_excluded_list_start, sizeof(int)*md_info->atom_numbers, hipMemcpyHostToDevice);
		hipMemcpy(d_excluded_numbers, h_excluded_numbers, sizeof(int)*md_info->atom_numbers, hipMemcpyHostToDevice);
		hipMemcpy(d_excluded_list, h_excluded_list, sizeof(int)*excluded_atom_numbers, hipMemcpyHostToDevice);
		controller->printf("    End reading excluded list from AMBER file\n\n");
		fclose(parm);
	}
	else
	{
		int atom_numbers = md_info->atom_numbers;
		excluded_atom_numbers = 0;
		controller->printf("    Set all atom exclude no atoms as default\n"); 

		Cuda_Malloc_Safely((void**)&d_excluded_list_start, sizeof(int)*atom_numbers);
		Cuda_Malloc_Safely((void**)&d_excluded_numbers, sizeof(int)*atom_numbers);
		Cuda_Malloc_Safely((void**)&d_excluded_list, sizeof(int)*excluded_atom_numbers);

		Malloc_Safely((void**)&h_excluded_list_start, sizeof(int)*atom_numbers);
		Malloc_Safely((void**)&h_excluded_numbers, sizeof(int)*atom_numbers);
		Malloc_Safely((void**)&h_excluded_list, sizeof(int)*excluded_atom_numbers);


		int count = 0;
		for (int i = 0; i < atom_numbers; i++)
		{
			h_excluded_numbers[i] = 0;
			h_excluded_list_start[i] = count;
			for (int j = 0; j < h_excluded_numbers[i]; j++)
			{
				h_excluded_list[count] = 0;
				count++;
			}
		}
		hipMemcpy(d_excluded_list_start, h_excluded_list_start, sizeof(int)*atom_numbers, hipMemcpyHostToDevice);
		hipMemcpy(d_excluded_numbers, h_excluded_numbers, sizeof(int)*atom_numbers, hipMemcpyHostToDevice);
		hipMemcpy(d_excluded_list, h_excluded_list, sizeof(int)*excluded_atom_numbers, hipMemcpyHostToDevice);
	}

}

void MD_INFORMATION::periodic_box_condition_information::Initial(CONTROLLER *controller, VECTOR box_length)
{
	crd_to_uint_crd_cof = CONSTANT_UINT_MAX_FLOAT / box_length;
	quarter_crd_to_uint_crd_cof = 0.25 * crd_to_uint_crd_cof;
	uint_dr_to_dr_cof = 1.0f / crd_to_uint_crd_cof;
}

void MD_INFORMATION::Read_Mode(CONTROLLER *controller)
{
	if (controller[0].Command_Exist("mode"))
	{
		if (is_str_equal(controller[0].Command("mode"), "NVT"))
		{
			controller->printf("    Mode set to NVT\n");
			mode = 1;
		}
		else if (is_str_equal(controller[0].Command("mode"), "NPT"))
		{
			controller->printf("    Mode set to NPT\n");
			mode = 2;
		}
		else if (is_str_equal(controller[0].Command("mode"), "Minimization"))
		{
			controller->printf("    Mode set to Energy Minimization\n");
			mode = -1;
		}
		else if (is_str_equal(controller[0].Command("mode"), "NVE"))
		{
			controller->printf("    Mode set to NVE\n");
			mode = 0;
		}
		else
		{
			controller->printf("    Warning: Mode '%s' not match. Set to NVE as default\n", controller[0].Command("mode"));
			mode = 0;
		}
	}
	else
	{
		controller->printf("    Mode set to NVE as default\n");
		mode = 0;
	}
}

void MD_INFORMATION::Read_dt(CONTROLLER *controller)
{
	if (controller[0].Command_Exist("dt"))
	{
		controller->printf("    dt set to %f ps\n", atof(controller[0].Command("dt")));
		dt = atof(controller[0].Command("dt")) * CONSTANT_TIME_CONVERTION;
		sscanf(controller[0].Command("dt"), "%lf", &sys.dt_in_ps);
	}
	else
	{
		if (mode != MINIMIZATION)
			dt = 0.001;
		else
			dt = 1e-8;
		sys.dt_in_ps = 0.001;
		controller->printf("    dt set to %f ps\n", dt);
		dt *= CONSTANT_TIME_CONVERTION;
	}
	if (mode == MINIMIZATION)
	{
		sys.dt_in_ps = 0;
	}
}

void MD_INFORMATION::trajectory_output::Initial(CONTROLLER *controller, MD_INFORMATION *md_info)
{
	this->md_info = md_info;
	current_crd_synchronized_step = 0;
	is_molecule_map_output = 0;
	if (controller[0].Command_Exist("molecule_map_output"))
	{
		is_molecule_map_output = atoi(controller[0].Command("molecule_map_output"));
	}
	write_trajectory_interval = 1000;
	if (controller[0].Command_Exist("write_information_interval"))
	{
		write_trajectory_interval = atoi(controller[0].Command("write_information_interval"));
	}
	write_restart_file_interval = write_trajectory_interval;
	if (controller[0].Command_Exist("write_restart_file_interval"))
	{
		write_restart_file_interval = atoi(controller[0].Command("write_restart_file_interval"));
	}
	if (controller->Command_Exist(TRAJ_COMMAND))
	{
		Open_File_Safely(&crd_traj, controller->Command(TRAJ_COMMAND), "wb");
	}
	else
	{
		Open_File_Safely(&crd_traj, TRAJ_DEFAULT_FILENAME, "wb");
	}
	if (controller->Command_Exist(BOX_TRAJ_COMMAND))
	{
		Open_File_Safely(&box_traj, controller->Command(BOX_TRAJ_COMMAND), "w");
	}
	else
	{
		Open_File_Safely(&box_traj, BOX_TRAJ_DEFAULT_FILENAME, "w");
	}
	if (controller->Command_Exist(RESTART_COMMAND))
	{
		strcpy(restart_name, controller->Command(RESTART_COMMAND));
	}
	else
	{
		strcpy(restart_name, RESTART_DEFAULT_FILENAME);
	}
	//20210827用于输出速度和力
	if (controller->Command_Exist(FRC_TRAJ_COMMAND))
	{
		is_frc_traj = 1;
		Open_File_Safely(&frc_traj, controller->Command(FRC_TRAJ_COMMAND), "wb");
	}
	if (controller->Command_Exist(VEL_TRAJ_COMMAND))
	{
		is_vel_traj = 1;
		Open_File_Safely(&vel_traj, controller->Command(VEL_TRAJ_COMMAND), "wb");
	}
}

void MD_INFORMATION::NVE_iteration::Initial(CONTROLLER *controller, MD_INFORMATION *md_info)
{
	this->md_info = md_info;
	max_velocity = -1;
	if (controller[0].Command_Exist("nve_velocity_max"))
	{
		max_velocity = atof(controller[0].Command("nve_velocity_max"));
	}
}
void MD_INFORMATION::residue_information::Read_AMBER_Parm7(const char *file_name, CONTROLLER controller)
{
	FILE *parm = NULL;
	Open_File_Safely(&parm, file_name, "r");
	controller.printf("    Start reading residue informataion from AMBER parm7:\n");
	
	while (true)
	{
		char temps[CHAR_LENGTH_MAX];
		char temp_first_str[CHAR_LENGTH_MAX];
		char temp_second_str[CHAR_LENGTH_MAX];
		if (fgets(temps, CHAR_LENGTH_MAX, parm) == NULL)
		{
			break;
		}
		if (sscanf(temps, "%s %s", temp_first_str, temp_second_str) != 2)
		{
			continue;
		}
		//read in atomnumber atomljtypenumber
		if (strcmp(temp_first_str, "%FLAG") == 0
			&& strcmp(temp_second_str, "POINTERS") == 0)
		{
			char *get_ret = fgets(temps, CHAR_LENGTH_MAX, parm);

			int atom_numbers = 0;
			int scanf_ret = fscanf(parm, "%d", &atom_numbers);
			if (md_info->atom_numbers > 0 && md_info->atom_numbers != atom_numbers)
			{
				controller.printf("        Error: atom_numbers is not equal: %d %d\n", md_info->atom_numbers, atom_numbers);
				getchar();
				exit(1);
			}
			else if (md_info->atom_numbers == 0)
			{
				md_info->atom_numbers = atom_numbers;
			}
			for (int i = 0; i < 10; i = i + 1)
			{
				int lin;
				scanf_ret = fscanf(parm, "%d\n", &lin);
			}
			scanf_ret = fscanf(parm, "%d\n", &this->residue_numbers);//NRES
			controller.printf("        residue_numbers is %d\n", this->residue_numbers);

			Malloc_Safely((void**)&h_mass, sizeof(float)*this->residue_numbers);
			Malloc_Safely((void**)&h_mass_inverse, sizeof(float)*this->residue_numbers);
			Malloc_Safely((void**)&h_res_start, sizeof(int)*this->residue_numbers);
			Malloc_Safely((void**)&h_res_end, sizeof(int)*this->residue_numbers);
			Malloc_Safely((void**)&h_momentum, sizeof(float)*this->residue_numbers);
			Malloc_Safely((void**)&h_center_of_mass, sizeof(VECTOR)*this->residue_numbers);
			Malloc_Safely((void**)&h_sigma_of_res_ek, sizeof(float));

			Cuda_Malloc_Safely((void**)&d_mass, sizeof(float)*this->residue_numbers);
			Cuda_Malloc_Safely((void**)&d_mass_inverse, sizeof(float)*this->residue_numbers);
			Cuda_Malloc_Safely((void**)&d_res_start, sizeof(int)*this->residue_numbers);
			Cuda_Malloc_Safely((void**)&d_res_end, sizeof(int)*this->residue_numbers);
			Cuda_Malloc_Safely((void**)&d_momentum, sizeof(float)*this->residue_numbers);
			Cuda_Malloc_Safely((void**)&d_center_of_mass, sizeof(VECTOR)*this->residue_numbers);
			Cuda_Malloc_Safely((void**)&res_ek_energy, sizeof(float)*this->residue_numbers); 
			Cuda_Malloc_Safely((void**)&sigma_of_res_ek, sizeof(float));
		}//FLAG POINTERS

		//residue range read
		if (strcmp(temp_first_str, "%FLAG") == 0
			&& strcmp(temp_second_str, "RESIDUE_POINTER") == 0)
		{
			char *get_ret = fgets(temps, CHAR_LENGTH_MAX, parm);
			//注意读进来的数的编号要减1
			int *lin_serial;
			Malloc_Safely((void**)&lin_serial, sizeof(int)* this->residue_numbers);
			for (int i = 0; i<this->residue_numbers; i = i + 1)
			{
				int scanf_ret = fscanf(parm, "%d\n", &lin_serial[i]);
			}
			for (int i = 0; i<this->residue_numbers - 1; i = i + 1)
			{
				h_res_start[i] = lin_serial[i] - 1;
				h_res_end[i] = lin_serial[i + 1] - 1;
			}
			h_res_start[this->residue_numbers - 1] = lin_serial[this->residue_numbers - 1] - 1;
			h_res_end[this->residue_numbers - 1] = md_info->atom_numbers + 1 - 1;

			free(lin_serial);
		}
	}//while cycle

	hipMemcpy(this->d_res_start, h_res_start, sizeof(int)*this->residue_numbers, hipMemcpyHostToDevice);
	hipMemcpy(this->d_res_end, h_res_end, sizeof(int)*this->residue_numbers, hipMemcpyHostToDevice);

	controller.printf("    End reading residue informataion from AMBER parm7\n\n");

	fclose(parm);
}

void MD_INFORMATION::residue_information::Initial(CONTROLLER *controller, MD_INFORMATION *md_info)
{
	this->md_info = md_info;
	if (!(controller[0].Command_Exist("residue_in_file")))
	{
		if (controller[0].Command_Exist("amber_parm7"))
		{
			Read_AMBER_Parm7(controller[0].Command("amber_parm7"), controller[0]);
			is_initialized = 1;
		}
		//对于没有residue输入的模拟，默认每个粒子作为一个residue
		else
		{
			residue_numbers = md_info->atom_numbers;
			controller->printf("    Set default residue list:\n");
			controller->printf("        residue_numbers is %d\n", residue_numbers);
			Malloc_Safely((void**)&h_mass, sizeof(float)*this->residue_numbers);
			Malloc_Safely((void**)&h_mass_inverse, sizeof(float)*this->residue_numbers);
			Malloc_Safely((void**)&h_res_start, sizeof(int)*this->residue_numbers);
			Malloc_Safely((void**)&h_res_end, sizeof(int)*this->residue_numbers);
			Malloc_Safely((void**)&h_momentum, sizeof(float)*this->residue_numbers);
			Malloc_Safely((void**)&h_center_of_mass, sizeof(VECTOR)*this->residue_numbers);
			Malloc_Safely((void**)&h_sigma_of_res_ek, sizeof(float));

			Cuda_Malloc_Safely((void**)&d_mass, sizeof(float)*this->residue_numbers);
			Cuda_Malloc_Safely((void**)&d_mass_inverse, sizeof(float)*this->residue_numbers);
			Cuda_Malloc_Safely((void**)&d_res_start, sizeof(int)*this->residue_numbers);
			Cuda_Malloc_Safely((void**)&d_res_end, sizeof(int)*this->residue_numbers);
			Cuda_Malloc_Safely((void**)&d_momentum, sizeof(float)*this->residue_numbers);
			Cuda_Malloc_Safely((void**)&d_center_of_mass, sizeof(VECTOR)*this->residue_numbers);
			Cuda_Malloc_Safely((void**)&res_ek_energy, sizeof(float)*this->residue_numbers);
			Cuda_Malloc_Safely((void**)&sigma_of_res_ek, sizeof(float));
			int count = 0;
			int temp=1;//每个粒子作为一个residue
			for (int i = 0; i < residue_numbers; i++)
			{
				h_res_start[i] = count;
				count += temp;
				h_res_end[i] = count;
			}
			hipMemcpy(d_res_start, h_res_start, sizeof(int)* residue_numbers, hipMemcpyHostToDevice);
			hipMemcpy(d_res_end, h_res_end, sizeof(int)* residue_numbers, hipMemcpyHostToDevice);
			controller->printf("    End reading residue list\n\n");
			is_initialized = 1;
		}
	}
	else
	{
		FILE *fp = NULL;
		controller->printf("    Start reading residue list:\n");
		Open_File_Safely(&fp, controller[0].Command("residue_in_file"), "r");
		int atom_numbers = 0;
		int scanf_ret = fscanf(fp, "%d %d", &atom_numbers, &residue_numbers);
		if (md_info->atom_numbers > 0 && md_info->atom_numbers != atom_numbers)
		{
			controller->printf("        Error: atom_numbers is not equal: %d %d\n", md_info->atom_numbers, atom_numbers);
			getchar();
			exit(1);
		}
		else if (md_info->atom_numbers == 0)
		{
			md_info->atom_numbers = atom_numbers;
		}
		controller->printf("        residue_numbers is %d\n", residue_numbers);
		Malloc_Safely((void**)&h_mass, sizeof(float)*this->residue_numbers);
		Malloc_Safely((void**)&h_mass_inverse, sizeof(float)*this->residue_numbers);
		Malloc_Safely((void**)&h_res_start, sizeof(int)*this->residue_numbers);
		Malloc_Safely((void**)&h_res_end, sizeof(int)*this->residue_numbers);
		Malloc_Safely((void**)&h_momentum, sizeof(float)*this->residue_numbers);
		Malloc_Safely((void**)&h_center_of_mass, sizeof(VECTOR)*this->residue_numbers);
		Malloc_Safely((void**)&h_sigma_of_res_ek, sizeof(float));

		Cuda_Malloc_Safely((void**)&d_mass, sizeof(float)*this->residue_numbers);
		Cuda_Malloc_Safely((void**)&d_mass_inverse, sizeof(float)*this->residue_numbers);
		Cuda_Malloc_Safely((void**)&d_res_start, sizeof(int)*this->residue_numbers);
		Cuda_Malloc_Safely((void**)&d_res_end, sizeof(int)*this->residue_numbers);
		Cuda_Malloc_Safely((void**)&d_momentum, sizeof(float)*this->residue_numbers);
		Cuda_Malloc_Safely((void**)&d_center_of_mass, sizeof(VECTOR)*this->residue_numbers);
		Cuda_Malloc_Safely((void**)&res_ek_energy, sizeof(float)*this->residue_numbers);
		Cuda_Malloc_Safely((void**)&sigma_of_res_ek, sizeof(float));

		int count = 0;
		int temp;
		for (int i = 0; i < residue_numbers; i++)
		{
			h_res_start[i] = count;
			scanf_ret = fscanf(fp, "%d", &temp);
			count += temp;
			h_res_end[i] = count;
		}
		hipMemcpy(d_res_start, h_res_start, sizeof(int)* residue_numbers, hipMemcpyHostToDevice);
		hipMemcpy(d_res_end, h_res_end, sizeof(int)* residue_numbers, hipMemcpyHostToDevice);
		controller->printf("    End reading residue list\n\n");
		fclose(fp);
		is_initialized = 1;
	}
	if (is_initialized)
	{
		if (md_info->h_mass != NULL)
		{
			for (int i = 0; i < residue_numbers; i++)
			{
				float temp_mass = 0;
				for (int j = h_res_start[i]; j < h_res_end[i]; j++)
				{
					temp_mass += md_info->h_mass[j];
				}
				this->h_mass[i] = temp_mass;
				if (temp_mass == 0)
					this->h_mass_inverse[i] = 0;
				else
					this->h_mass_inverse[i] = 1.0 / temp_mass;
			}
			hipMemcpy(d_mass_inverse, h_mass_inverse, sizeof(float)* residue_numbers, hipMemcpyHostToDevice);
			hipMemcpy(d_mass, h_mass, sizeof(float)* residue_numbers, hipMemcpyHostToDevice);
		}
		else
		{
			controller->printf("    Error: atom mass should be initialized before residue mass\n");
			getchar();
			exit(1);
		}
	}
}

void MD_INFORMATION::Read_Coordinate_And_Velocity(CONTROLLER *controller)
{
	sys.start_time = 0.0;
	if (controller[0].Command_Exist("coordinate_in_file"))
	{
		Read_Coordinate_In_File(controller[0].Command("coordinate_in_file"), controller[0]);
		if (controller[0].Command_Exist("velocity_in_file"))
		{
			FILE *fp = NULL;
			controller->printf("    Start reading velocity_in_file:\n");
			Open_File_Safely(&fp, controller[0].Command("velocity_in_file"), "r");
			
			int atom_numbers = 0;
			char lin[CHAR_LENGTH_MAX];
			char *get_ret = fgets(lin, CHAR_LENGTH_MAX, fp);
			int scanf_ret = sscanf(lin, "%d", &atom_numbers);
			if (this->atom_numbers > 0 && this->atom_numbers != atom_numbers)
			{
				controller->printf("        Error: atom_numbers is not equal: %d %d\n", this->atom_numbers, atom_numbers);
				getchar();
				exit(1);
			}
			Malloc_Safely((void**)&velocity, sizeof(VECTOR)*this->atom_numbers);
			Cuda_Malloc_Safely((void**)&vel, sizeof(VECTOR)*this->atom_numbers);
			for (int i = 0; i < atom_numbers; i++)
			{
				scanf_ret = fscanf(fp, "%f %f %f", &velocity[i].x, &velocity[i].y, &velocity[i].z);
			}
			hipMemcpy(vel, velocity, sizeof(VECTOR)* atom_numbers, hipMemcpyHostToDevice);
			controller->printf("    End reading velocity_in_file\n\n");
			fclose(fp);
		}
		else
		{
			controller->printf("    Velocity is set to zero as default\n");
			Malloc_Safely((void**)&velocity, sizeof(VECTOR)*this->atom_numbers);
			Cuda_Malloc_Safely((void**)&vel, sizeof(VECTOR)*this->atom_numbers);
			for (int i = 0; i < atom_numbers; i++)
			{
				velocity[i].x = 0;
				velocity[i].y = 0;
				velocity[i].z = 0;
			}
			hipMemcpy(vel, velocity, sizeof(VECTOR)* atom_numbers, hipMemcpyHostToDevice);
		}
	}
	else if (controller[0].Command_Exist("amber_rst7"))
	{
		output.amber_irest = 1;
		if (controller[0].Command_Exist("amber_irest"))
			output.amber_irest = atoi(controller[0].Command("amber_irest"));
		Read_Rst7(controller[0].Command("amber_rst7"), output.amber_irest, controller[0]);
	}
	else
	{
		printf("MD basic information needed. Specify the coordinate in file.\n");
		getchar();
		exit(1);
	}
}

void MD_INFORMATION::Read_Mass(CONTROLLER *controller)
{
	if (controller[0].Command_Exist("mass_in_file"))
	{
		FILE *fp = NULL;
		controller->printf("    Start reading mass:\n");
		Open_File_Safely(&fp, controller[0].Command("mass_in_file"), "r");
		int atom_numbers = 0;
		char lin[CHAR_LENGTH_MAX];
		char *get_ret = fgets(lin, CHAR_LENGTH_MAX, fp);
		int scanf_ret = sscanf(lin, "%d", &atom_numbers);
		if (this->atom_numbers > 0 && this->atom_numbers != atom_numbers)
		{
			controller->printf("        Error: atom_numbers is not equal: %d %d\n", this->atom_numbers, atom_numbers);
			getchar();
			exit(1);
		}
		else if (this->atom_numbers == 0)
		{
			this->atom_numbers = atom_numbers;
		}
		Malloc_Safely((void**)&h_mass, sizeof(float)* atom_numbers);
		Malloc_Safely((void**)&h_mass_inverse, sizeof(float)* atom_numbers);
		Cuda_Malloc_Safely((void**)&d_mass, sizeof(float)* atom_numbers);
		Cuda_Malloc_Safely((void**)&d_mass_inverse, sizeof(float)* atom_numbers);
		sys.total_mass = 0;
		for (int i = 0; i < atom_numbers; i++)
		{
			scanf_ret = fscanf(fp, "%f", &h_mass[i]);
			sys.total_mass += h_mass[i];
			if (h_mass[i] == 0)
				h_mass_inverse[i] = 0;
			else
				h_mass_inverse[i] = 1.0 / h_mass[i];

		}
		controller->printf("    End reading mass\n\n");
		fclose(fp);
	}
	else if (controller[0].Command_Exist("amber_parm7"))
	{
		FILE *parm = NULL;
		Open_File_Safely(&parm, controller[0].Command("amber_parm7"), "r");
		controller[0].printf("    Start reading mass from AMBER parm7:\n");
		while (true)
		{
			char temps[CHAR_LENGTH_MAX];
			char temp_first_str[CHAR_LENGTH_MAX];
			char temp_second_str[CHAR_LENGTH_MAX];
			if (fgets(temps, CHAR_LENGTH_MAX, parm) == NULL)
			{
				break;
			}
			if (sscanf(temps, "%s %s", temp_first_str, temp_second_str) != 2)
			{
				continue;
			}
			//read in atomnumber atomljtypenumber
			if (strcmp(temp_first_str, "%FLAG") == 0
				&& strcmp(temp_second_str, "POINTERS") == 0)
			{
				char *get_ret = fgets(temps, CHAR_LENGTH_MAX, parm);

				int atom_numbers = 0;
				int scanf_ret = fscanf(parm, "%d", &atom_numbers);
				if (this->atom_numbers > 0 && this->atom_numbers != atom_numbers)
				{
					controller->printf("        Error: atom_numbers is not equal: %d %d\n", this->atom_numbers, atom_numbers);
					getchar();
					exit(1);
				}
				else if (this->atom_numbers == 0)
				{
					this->atom_numbers = atom_numbers;
				}
				Malloc_Safely((void**)&h_mass, sizeof(float)* atom_numbers);
				Malloc_Safely((void**)&h_mass_inverse, sizeof(float)* atom_numbers);
				Cuda_Malloc_Safely((void**)&d_mass, sizeof(float)* atom_numbers);
				Cuda_Malloc_Safely((void**)&d_mass_inverse, sizeof(float)* atom_numbers);
			}
			if (strcmp(temp_first_str, "%FLAG") == 0
				&& strcmp(temp_second_str, "MASS") == 0)
			{
				char *get_ret = fgets(temps, CHAR_LENGTH_MAX, parm);
				double lin;
				sys.total_mass = 0;
				for (int i = 0; i < this->atom_numbers; i = i + 1)
				{
					int scanf_ret = fscanf(parm, "%lf\n", &lin);
					this->h_mass[i] = (float)lin;
					if (h_mass[i] == 0)
						h_mass_inverse[i] = 0;
					else
						h_mass_inverse[i] = 1.0f / h_mass[i];
					sys.total_mass += h_mass[i];
				}
			}
		}
		controller[0].printf("    End reading mass from AMBER parm7\n\n");
		fclose(parm);
	}
	else if (atom_numbers > 0)
	{
		controller[0].printf("    mass is set to 20 as default\n");
		sys.total_mass = 0;
		Malloc_Safely((void**)&h_mass, sizeof(float)* atom_numbers);
		Malloc_Safely((void**)&h_mass_inverse, sizeof(float)* atom_numbers);
		Cuda_Malloc_Safely((void**)&d_mass, sizeof(float)* atom_numbers);
		Cuda_Malloc_Safely((void**)&d_mass_inverse, sizeof(float)* atom_numbers);
		for (int i = 0; i < atom_numbers; i++)
		{
			h_mass[i] = 20;
			h_mass_inverse[i] = 1.0 / h_mass[i];
			sys.total_mass += h_mass[i];
		}
	}
	else
	{
		controller[0].printf("    Error: failed to initialze mass, because no atom_numbers found\n");
		getchar();
		exit(1);
	}
	if (atom_numbers > 0)
	{
		hipMemcpy(d_mass, h_mass, sizeof(float)* atom_numbers, hipMemcpyHostToDevice);
		hipMemcpy(d_mass_inverse, h_mass_inverse, sizeof(float)* atom_numbers, hipMemcpyHostToDevice);
	}
}

void MD_INFORMATION::Read_Charge(CONTROLLER *controller)
{
	if (controller[0].Command_Exist("charge_in_file"))
	{
		FILE *fp = NULL;
		controller->printf("    Start reading charge:\n");
		Open_File_Safely(&fp, controller[0].Command("charge_in_file"), "r");
		int atom_numbers = 0;
		char lin[CHAR_LENGTH_MAX];
		char *get_ret = fgets(lin, CHAR_LENGTH_MAX, fp);
		int scanf_ret = sscanf(lin, "%d", &atom_numbers);
		if (this->atom_numbers > 0 && this->atom_numbers != atom_numbers)
		{
			controller->printf("        Error: atom_numbers is not equal: %d %d\n", this->atom_numbers, atom_numbers);
			getchar();
			exit(1);
		}
		else if (this->atom_numbers == 0)
		{
			this->atom_numbers = atom_numbers;
		}
		Malloc_Safely((void**)&h_charge, sizeof(float)* atom_numbers);
		Cuda_Malloc_Safely((void**)&d_charge, sizeof(float)* atom_numbers);
		for (int i = 0; i < atom_numbers; i++)
		{
			scanf_ret = fscanf(fp, "%f", &h_charge[i]);
		}
		controller->printf("    End reading charge\n\n");
		fclose(fp);
	}
	else if (controller[0].Command_Exist("amber_parm7"))
	{
		FILE *parm = NULL;
		Open_File_Safely(&parm, controller[0].Command("amber_parm7"), "r");
		controller[0].printf("    Start reading charge from AMBER parm7:\n");
		while (true)
		{
			char temps[CHAR_LENGTH_MAX];
			char temp_first_str[CHAR_LENGTH_MAX];
			char temp_second_str[CHAR_LENGTH_MAX];
			if (fgets(temps, CHAR_LENGTH_MAX, parm) == NULL)
			{
				break;
			}
			if (sscanf(temps, "%s %s", temp_first_str, temp_second_str) != 2)
			{
				continue;
			}
			//read in atomnumber atomljtypenumber
			if (strcmp(temp_first_str, "%FLAG") == 0
				&& strcmp(temp_second_str, "POINTERS") == 0)
			{
				char *get_ret = fgets(temps, CHAR_LENGTH_MAX, parm);

				int atom_numbers = 0;
				int scanf_ret = fscanf(parm, "%d", &atom_numbers);
				if (this->atom_numbers > 0 && this->atom_numbers != atom_numbers)
				{
					controller->printf("        Error: atom_numbers is not equal: %d %d\n", this->atom_numbers, atom_numbers);
					getchar();
					exit(1);
				}
				else if (this->atom_numbers == 0)
				{
					this->atom_numbers = atom_numbers;
				}
				Malloc_Safely((void**)&h_charge, sizeof(float)* atom_numbers);
				Cuda_Malloc_Safely((void**)&d_charge, sizeof(float)* atom_numbers);
			}
			if (strcmp(temp_first_str, "%FLAG") == 0
				&& strcmp(temp_second_str, "CHARGE") == 0)
			{
				char *get_ret = fgets(temps, CHAR_LENGTH_MAX, parm);
				for (int i = 0; i < this->atom_numbers; i = i + 1)
				{
					int scanf_ret = fscanf(parm, "%f", &h_charge[i]);
				}
			}
		}
		controller[0].printf("    End reading charge from AMBER parm7\n\n");
		fclose(parm);
	}
	else if (atom_numbers > 0)
	{
		controller[0].printf("    charge is set to 0 as default\n");
		Malloc_Safely((void**)&h_charge, sizeof(float)* atom_numbers);
		Cuda_Malloc_Safely((void**)&d_charge, sizeof(float)* atom_numbers);
		for (int i = 0; i < atom_numbers; i++)
		{
			h_charge[i] = 0;
		}
	}
	else
	{
		controller[0].printf("    Error: failed to initialze charge, because no atom_numbers found\n");
		getchar();
		exit(1);
	}
	if (atom_numbers > 0)
	{
		hipMemcpy(d_charge, h_charge, sizeof(float)* atom_numbers, hipMemcpyHostToDevice);
	}
}

//MD_INFORMATION成员函数
void MD_INFORMATION::Initial(CONTROLLER *controller)
{
	controller->printf("START INITIALZING MD CORE:\n");
	atom_numbers = 0; //初始化，使得能够进行所有原子数目是否相等的判断
	
	strcpy(md_name, controller[0].Command("md_name"));
	Read_Mode(controller);
	Read_dt(controller);

	Read_Coordinate_And_Velocity(controller);
	
	
	Read_Mass(controller);
	Read_Charge(controller);
    
	sys.Initial(controller, this);  //!需要先初始化坐标和速度
	nb.Initial(controller, this);
	
	output.Initial(controller, this);

	nve.Initial(controller, this);
	
	min.Initial(controller, this);

	res.Initial(controller, this);

	mol.Initial(controller, this);

	pbc.Initial(controller, sys.box_length);
	
	Atom_Information_Initial();

	is_initialized = 1;
	controller->printf("    structure last modify date is %d\n", last_modify_date);
	controller->printf("END INITIALZING MD CORE\n\n");
}

void MD_INFORMATION::Atom_Information_Initial()
{
	Malloc_Safely((void**)&this->force, sizeof(VECTOR)*this->atom_numbers);
	Malloc_Safely((void**)&this->h_atom_energy, sizeof(float)* atom_numbers);
	Malloc_Safely((void**)&this->h_atom_virial, sizeof(double)* atom_numbers);
	Cuda_Malloc_Safely((void**)&this->acc, sizeof(VECTOR)*this->atom_numbers);
	Cuda_Malloc_Safely((void**)&this->frc, sizeof(VECTOR)*this->atom_numbers);
	Cuda_Malloc_Safely((void **)&this->uint_crd, sizeof(UNSIGNED_INT_VECTOR)*this->atom_numbers);
	Cuda_Malloc_Safely((void**)&this->d_atom_energy, sizeof(float)* atom_numbers);
	Cuda_Malloc_Safely((void**)&this->d_atom_virial, sizeof(float)* atom_numbers);
	Cuda_Malloc_Safely((void**)&this->d_atom_ek, sizeof(float)* atom_numbers);
	Reset_List << <ceilf((float)3.*this->atom_numbers / 32), 32 >> >
		(3 * this->atom_numbers, (float*)this->acc, 0.);
	Reset_List << <ceilf((float)3.*this->atom_numbers / 32), 32 >> >
		(3 * this->atom_numbers, (float*)this->frc, 0.);
	sys.freedom = 3 * atom_numbers; //最大自由度，后面减
}

void MD_INFORMATION::Read_Coordinate_In_File(const char* file_name, CONTROLLER controller)
{
	FILE *fp =NULL;
	controller.printf("    Start reading coordinate_in_file:\n");
	Open_File_Safely(&fp, file_name, "r");
	char lin[CHAR_LENGTH_MAX];
	char *get_ret = fgets(lin, CHAR_LENGTH_MAX, fp);
	int atom_numbers = 0;
	int scanf_ret = sscanf(lin, "%d %lf", &atom_numbers, &sys.start_time);
	if (this->atom_numbers > 0 && this->atom_numbers != atom_numbers)
	{
		controller.printf("        Error: atom_numbers is not equal: %d %d\n", this->atom_numbers, atom_numbers);
		getchar();
		exit(1);
	}
	else if (this->atom_numbers == 0)
	{
		this->atom_numbers = atom_numbers;
	}
	if (scanf_ret == 0)
	{
		controller.printf("        Error: Atom_numbers not found.\n");
		getchar();
		exit(1);
	}
	else  if (scanf_ret == 1)
	{
		sys.start_time = 0;
	}

	controller.printf("        atom_numbers is %d\n", this->atom_numbers);
	controller.printf("        system start_time is %lf\n", this->sys.start_time);
	Malloc_Safely((void**)&coordinate, sizeof(VECTOR)*this->atom_numbers);
	Cuda_Malloc_Safely((void**)&crd, sizeof(VECTOR)*this->atom_numbers);

	for (int i = 0; i < atom_numbers; i++)
	{
		scanf_ret = fscanf(fp, "%f %f %f", &coordinate[i].x, &coordinate[i].y, &coordinate[i].z);
	}
	scanf_ret = fscanf(fp, "%f %f %f", &sys.box_length.x, &sys.box_length.y, &sys.box_length.z);
	controller.printf("        box_length is\n            x: %f\n            y: %f\n            z: %f\n", sys.box_length.x, sys.box_length.y, sys.box_length.z);
	hipMemcpy(crd, coordinate, sizeof(VECTOR)* atom_numbers, hipMemcpyHostToDevice);
	controller.printf("    End reading coordinate_in_file\n\n");
	fclose(fp);
}
void MD_INFORMATION::Read_Rst7(const char* file_name, int irest, CONTROLLER controller)
{
	FILE *fin = NULL;
	Open_File_Safely(&fin, file_name, "r");
	controller.printf("    Start reading AMBER rst7:\n");
	char lin[CHAR_LENGTH_MAX];
	int atom_numbers = 0;
	char *get_ret = fgets(lin, CHAR_LENGTH_MAX, fin);
	get_ret = fgets(lin, CHAR_LENGTH_MAX, fin);
	int has_vel = 0;
	int scanf_ret = sscanf(lin, "%d %lf", &atom_numbers, &sys.start_time);
	if (this->atom_numbers > 0 && this->atom_numbers != atom_numbers)
	{
		controller.printf("        Error: atom_numbers is not equal: %d %d\n", this->atom_numbers, atom_numbers);
		getchar();
		exit(1);
	}
	else if (this->atom_numbers == 0)
	{
		this->atom_numbers = atom_numbers;
	}
	if (scanf_ret == 0)
	{
		controller.printf("        Error: Atom_numbers not found.\n");
		getchar();
		exit(1);
	}
	else  if (scanf_ret == 2)
	{
		has_vel = 1;
	}
	else
	{
		sys.start_time = 0;
	}

	Malloc_Safely((void**)&coordinate, sizeof(VECTOR)*this->atom_numbers);
	Cuda_Malloc_Safely((void**)&crd, sizeof(VECTOR)*this->atom_numbers);
	Malloc_Safely((void**)&velocity, sizeof(VECTOR)*this->atom_numbers);
	Cuda_Malloc_Safely((void**)&vel, sizeof(VECTOR)*this->atom_numbers);

	controller.printf("        atom_numbers is %d\n", this->atom_numbers);
	controller.printf("        system start time is %lf\n", this->sys.start_time);

	if (has_vel == 0 || irest == 0)
	{
		controller.printf("        All velocity will be set to 0\n");
	}


	for (int i = 0; i < this->atom_numbers; i = i + 1)
	{
		scanf_ret = fscanf(fin, "%f %f %f",
			&this->coordinate[i].x,
			&this->coordinate[i].y,
			&this->coordinate[i].z);
	}
	if (has_vel)
	{
		for (int i = 0; i < this->atom_numbers; i = i + 1)
		{
			scanf_ret = fscanf(fin, "%f %f %f",
				&this->velocity[i].x,
				&this->velocity[i].y,
				&this->velocity[i].z);
		}
	}
	if (irest == 0 || !has_vel)
	{
		for (int i = 0; i < this->atom_numbers; i = i + 1)
		{
			this->velocity[i].x = 0.0;
			this->velocity[i].y = 0.0;
			this->velocity[i].z = 0.0;
		}
	}
	scanf_ret = fscanf(fin, "%f %f %f", &this->sys.box_length.x, &this->sys.box_length.y, &this->sys.box_length.z);
	controller.printf("        system size is %f %f %f\n", this->sys.box_length.x, this->sys.box_length.y, this->sys.box_length.z);
	hipMemcpy(this->crd, this->coordinate, sizeof(VECTOR)*this->atom_numbers, hipMemcpyHostToDevice);
	hipMemcpy(this->vel, this->velocity, sizeof(VECTOR)*this->atom_numbers, hipMemcpyHostToDevice);
	//in some bad rst7, the coordinates will be extremly bad, so need a full box map
	for (int i = 0; i < 10; i = i + 1)
	{
		Crd_Periodic_Map << <ceilf((float)this->atom_numbers / 32), 32 >> >
			(this->atom_numbers, this->crd, this->sys.box_length);
	}
	fclose(fin);
	controller.printf("    End reading AMBER rst7\n\n");
	
}

void MD_INFORMATION::trajectory_output::Append_Crd_Traj_File(FILE *fp)
{
	if (md_info->is_initialized)
	{
		md_info->Crd_Vel_Device_To_Host();
		if (fp == NULL)
		{
			fp = crd_traj;
		}
		fwrite(&md_info->coordinate[0].x, sizeof(VECTOR), md_info->atom_numbers, fp);
	}
}

// 20210827用于输出速度和力
void MD_INFORMATION::trajectory_output::Append_Frc_Traj_File(FILE *fp)
{
	if (md_info->is_initialized)
	{
		hipMemcpy(md_info->force, md_info->frc, sizeof(VECTOR)*md_info->atom_numbers, hipMemcpyDeviceToHost);
		if (fp == NULL)//默认的frc输出位置
		{
			fp = frc_traj;
			if (fp != NULL)
			{
				fwrite(&md_info->force[0].x, sizeof(VECTOR), md_info->atom_numbers, fp);
			}
		}
		else
		{
			fwrite(&md_info->force[0].x, sizeof(VECTOR), md_info->atom_numbers, fp);
		}
	}
}
void MD_INFORMATION::trajectory_output::Append_Vel_Traj_File(FILE *fp)
{
	if (md_info->is_initialized)
	{
		hipMemcpy(md_info->velocity, md_info->vel, sizeof(VECTOR)*md_info->atom_numbers, hipMemcpyDeviceToHost);
		if (fp == NULL)//默认的vel输出位置
		{
			fp = vel_traj;
			if (fp != NULL)
			{
				fwrite(&md_info->velocity[0].x, sizeof(VECTOR), md_info->atom_numbers, fp);
			}
		}
		else
		{
			fwrite(&md_info->velocity[0].x, sizeof(VECTOR), md_info->atom_numbers, fp);
		}
	}
}

void MD_INFORMATION::trajectory_output::Append_Box_Traj_File(FILE *fp)
{
	if (md_info->is_initialized)
	{
		if (fp == NULL)
		{
			fp = box_traj;
		}
		fprintf(fp, "%f %f %f %.0f %.0f %.0f\n", md_info->sys.box_length.x, md_info->sys.box_length.y, md_info->sys.box_length.z, 90.0f, 90.0f, 90.0f);
	}
}

void MD_INFORMATION::trajectory_output::Export_Restart_File(const char *rst7_name)
{
	if (!md_info->is_initialized)
		return;

	char filename[CHAR_LENGTH_MAX];
	if (rst7_name == NULL)
		strcpy(filename, restart_name);
	else
		strcpy(filename, rst7_name);
	md_info->Crd_Vel_Device_To_Host();
	if (amber_irest >= 0)
	{
		const char *sys_name = md_info->md_name;
		FILE *lin = NULL;
		Open_File_Safely(&lin, filename, "w");
		fprintf(lin, "%s\n", sys_name);
		fprintf(lin, "%8d %.3lf\n", md_info->atom_numbers, md_info->sys.Get_Current_Time());
		int s = 0;
		for (int i = 0; i < md_info->atom_numbers; i = i + 1)
		{
			fprintf(lin, "%12.7f%12.7f%12.7f", md_info->coordinate[i].x, md_info->coordinate[i].y, md_info->coordinate[i].z);
			s = s + 1;
			if (s == 2)
			{
				s = 0;
				fprintf(lin, "\n");
			}
		}
		if (s == 1)
		{
			s = 0;
			fprintf(lin, "\n");
		}
		for (int i = 0; i < md_info->atom_numbers; i = i + 1)
		{
			fprintf(lin, "%12.7f%12.7f%12.7f", md_info->velocity[i].x, md_info->velocity[i].y, md_info->velocity[i].z);
			s = s + 1;
			if (s == 2)
			{
				s = 0;
				fprintf(lin, "\n");
			}
		}
		if (s == 1)
		{
			s = 0;
			fprintf(lin, "\n");
		}
		fprintf(lin, "%12.7f%12.7f%12.7f", (float)md_info->sys.box_length.x, (float)md_info->sys.box_length.y, (float)md_info->sys.box_length.z);
		fprintf(lin, "%12.7f%12.7f%12.7f", (float)90., (float)90., (float)90.);
		fclose(lin);
	}
	else
	{
		FILE *lin = NULL;
		FILE *lin2 = NULL;
		char buffer[CHAR_LENGTH_MAX];
		sprintf(buffer, "%s_%s.txt", filename, "coordinate");
		Open_File_Safely(&lin, buffer, "w");
		sprintf(buffer, "%s_%s.txt", filename, "velocity");
		Open_File_Safely(&lin2, buffer, "w");
		fprintf(lin, "%d %.3lf\n", md_info->atom_numbers, md_info->sys.Get_Current_Time());
		fprintf(lin2, "%d %.3lf\n", md_info->atom_numbers, md_info->sys.Get_Current_Time());
		for (int i = 0; i < md_info->atom_numbers; i++)
		{
			fprintf(lin, "%12.7f %12.7f %12.7f\n", md_info->coordinate[i].x, md_info->coordinate[i].y, md_info->coordinate[i].z);
			fprintf(lin2, "%12.7f %12.7f %12.7f\n", md_info->velocity[i].x, md_info->velocity[i].y, md_info->velocity[i].z);
		}
		fprintf(lin, "%12.7f %12.7f %12.7f %12.7f %12.7f %12.7f", md_info->sys.box_length.x, md_info->sys.box_length.y, md_info->sys.box_length.z, 90.0f, 90.0f, 90.0f);
		fclose(lin);
		fclose(lin2);
	}
}


void MD_INFORMATION::Update_Volume(double factor)
{
	sys.box_length = factor * sys.box_length;
	pbc.crd_to_uint_crd_cof = CONSTANT_UINT_MAX_FLOAT / sys.box_length;
	pbc.quarter_crd_to_uint_crd_cof = 0.25 * pbc.crd_to_uint_crd_cof;
	pbc.uint_dr_to_dr_cof = 1.0f / pbc.crd_to_uint_crd_cof;
	MD_Information_Crd_To_Uint_Crd();
}


float MD_INFORMATION::system_information::Get_Density()
{
	density = total_mass * 1e24f / 6.023e23f / Get_Volume();
	return density;
}

double MD_INFORMATION::system_information::Get_Current_Time()
{
	current_time = start_time + (double)dt_in_ps * steps;
	return current_time;
}

float MD_INFORMATION::system_information::Get_Volume()
{
	volume = box_length.x * box_length.y * box_length.z;
	return volume;
}

void MD_INFORMATION::MD_Information_Crd_To_Uint_Crd()
{
	Crd_To_Uint_Crd << <ceilf((float)this->atom_numbers / 128), 128 >> >
		(this->atom_numbers, pbc.quarter_crd_to_uint_crd_cof, crd, uint_crd);
}


void MD_INFORMATION::NVE_iteration::Leap_Frog()
{
	if (max_velocity <= 0)
	{
		MD_Iteration_Leap_Frog << <ceilf((float)md_info->atom_numbers / 128), 128 >> >
			(md_info->atom_numbers, md_info->vel, md_info->crd, md_info->frc, md_info->acc, md_info->d_mass_inverse, md_info->dt);
	}
	else
	{
		MD_Iteration_Leap_Frog_With_Max_Velocity << <ceilf((float)md_info->atom_numbers / 128), 128 >> >
			(md_info->atom_numbers, md_info->vel, md_info->crd, md_info->frc, md_info->acc, md_info->d_mass_inverse, md_info->dt, max_velocity);
	}
}

void MD_INFORMATION::MINIMIZATION_iteration::Initial(CONTROLLER *controller, MD_INFORMATION *md_info)
{
	this->md_info = md_info;
	if (md_info->mode == MINIMIZATION)
	{
		controller->printf("    Start initializing minimization:\n");
		max_move = 0.1;
		if (controller[0].Command_Exist("minimization_max_move"))
		{
			max_move = atof(controller[0].Command("minimization_max_move"));
		}
		controller->printf("        minimization max move is %f A\n", max_move);

		dynamic_dt = 0;
		if (controller[0].Command_Exist("minimization_dynamic_dt"))
		{
			dynamic_dt = atoi(controller[0].Command("minimization_dynamic_dt"));
		}
		controller->printf("        minimization dynamic dt is %d\n", dynamic_dt);

		dt_decreasing_rate = 0.01;
		if (controller[0].Command_Exist("minimization_dt_decreasing_rate"))
		{
			dt_decreasing_rate = atof(controller[0].Command("minimization_dt_decreasing_rate"));
		}
		controller->printf("        minimization dt decreasing rate is %f\n", dt_decreasing_rate);

		dt_increasing_rate = 1.01;
		if (controller[0].Command_Exist("minimization_dt_increasing_rate"))
		{
			dt_increasing_rate = atof(controller[0].Command("minimization_dt_increasing_rate"));
		}
		controller->printf("        minimization dt increasing rate is %f\n", dt_increasing_rate);

		momentum_keep = 0;
		if (controller[0].Command_Exist("minimization_momentum_keep"))
		{
			momentum_keep = atof(controller[0].Command("minimization_momentum_keep"));
		}
		controller->printf("        minimization momentum keep is %f\n", momentum_keep);

		controller->printf("    End initializing minimization\n\n");
	}
}

void MD_INFORMATION::MINIMIZATION_iteration::Gradient_Descent()
{
	if (dynamic_dt)
	{
		if (md_info->sys.steps != 1)
		{
			if (last_potential > md_info->sys.h_potential)
			{
				md_info->dt *= dt_increasing_rate;
			}
			else
			{
				if (md_info->dt > 1e-8)
				{
					md_info->dt *= dt_decreasing_rate;
				}
			}
		}
		last_potential = md_info->sys.h_potential;
	}
	if (max_move <= 0)
	{
		MD_Iteration_Gradient_Descent << <ceilf((float)md_info->atom_numbers / 128), 128 >> >
			(md_info->atom_numbers, md_info->crd, md_info->frc, md_info->d_mass_inverse, md_info->dt, md_info->vel, momentum_keep);
	}
	else
	{
		MD_Iteration_Gradient_Descent_With_Max_Move << <ceilf((float)md_info->atom_numbers / 128), 128 >> >
			(md_info->atom_numbers, md_info->crd, md_info->frc, md_info->d_mass_inverse, md_info->dt, md_info->vel, momentum_keep, max_move);
	}
}



void MD_INFORMATION::NVE_iteration::Velocity_Verlet_1()
{
	MD_Iteration_Speed_Verlet_1 << <ceilf((float)md_info->atom_numbers / 32), 32 >> >
		(md_info->atom_numbers, 0.5*md_info->dt, md_info->dt, md_info->acc, md_info->vel, md_info->crd, md_info->frc);
}

void MD_INFORMATION::NVE_iteration::Velocity_Verlet_2()
{
	if (max_velocity <= 0)
	{
		MD_Iteration_Speed_Verlet_2 << <ceilf((float)md_info->atom_numbers / 32), 32 >> >
			(md_info->atom_numbers, 0.5*md_info->dt, md_info->d_mass_inverse, md_info->frc, md_info->vel, md_info->acc);
	}
	else
	{
		MD_Iteration_Speed_Verlet_2_With_Max_Velocity << <ceilf((float)md_info->atom_numbers / 32), 32 >> >
			(md_info->atom_numbers, 0.5*md_info->dt, md_info->d_mass_inverse, md_info->frc, md_info->vel, md_info->acc, max_velocity);
	}
}


float MD_INFORMATION::system_information::Get_Total_Atom_Ek(int is_download)
{
	MD_Atom_Ek << <ceilf((float)md_info->atom_numbers / 32.), 32 >> >(md_info->atom_numbers, md_info->d_atom_ek, md_info->vel, md_info->d_mass);
	Sum_Of_List << <1, 1024 >> >(md_info->atom_numbers, md_info->d_atom_ek, d_sum_of_atom_ek);
	if (is_download)
	{
		hipMemcpy(&h_sum_of_atom_ek, d_sum_of_atom_ek, sizeof(float), hipMemcpyDeviceToHost);
		return h_sum_of_atom_ek;
	}
	else
	{
		return 0;
	}	
}

float MD_INFORMATION::system_information::Get_Atom_Temperature()
{
	h_temperature = Get_Total_Atom_Ek() * 2. / CONSTANT_kB / freedom;
	return h_temperature;
}

float MD_INFORMATION::residue_information::Get_Total_Residue_Ek(int is_download)
{
	MD_Residue_Ek << <ceilf((float)residue_numbers / 32.), 32 >> >
		(residue_numbers, d_res_start, d_res_end, res_ek_energy,
		md_info->vel, md_info->d_mass);
	Sum_Of_List << <1, 1024 >> >(residue_numbers, res_ek_energy, sigma_of_res_ek);
	if (is_download)
	{
		hipMemcpy(h_sigma_of_res_ek, sigma_of_res_ek, sizeof(float), hipMemcpyDeviceToHost);
		return h_sigma_of_res_ek[0];
	}
	else
	{
		return 0;
	}
}

float MD_INFORMATION::residue_information::Get_Residue_Temperature()
{
	h_temperature = Get_Total_Residue_Ek() * 2. / CONSTANT_kB / residue_numbers / 3;
	return h_temperature;
}

void MD_INFORMATION::residue_information::Residue_Crd_Map(VECTOR *no_wrap_crd, float scaler)
{
	Get_Center_Of_Mass << <20, 32 >> >(residue_numbers, d_res_start, d_res_end, no_wrap_crd, md_info->d_mass, d_mass_inverse, d_center_of_mass);
	Map_Center_Of_Mass << <20, { 32, 4 } >> >(residue_numbers, d_res_start, d_res_end, scaler, d_center_of_mass, md_info->sys.box_length, no_wrap_crd, md_info->crd);
}


void MD_INFORMATION::MD_Reset_Atom_Energy_And_Virial_And_Force()
{
	need_potential = 0;
	hipMemset(d_atom_energy, 0, sizeof(float)* atom_numbers);
	hipMemset(sys.d_potential, 0, sizeof(float));

	need_pressure = 0;
	hipMemset(d_atom_virial, 0, sizeof(float)* atom_numbers);
	hipMemset(sys.d_virial, 0, sizeof(float));

	hipMemset(frc, 0, sizeof(VECTOR)* atom_numbers);
}


void MD_INFORMATION::Calculate_Pressure_And_Potential_If_Needed(int is_download)
{
	if (need_pressure > 0)
	{
		sys.Get_Pressure(is_download);
	}
	if (need_potential > 0)
	{
		sys.Get_Potential(is_download);
	}
}

float MD_INFORMATION::system_information::Get_Pressure(int is_download)
{
	//计算动能
	MD_Atom_Ek << <ceilf((float)md_info->atom_numbers / 32.), 32 >> >(md_info->atom_numbers, md_info->d_atom_ek, md_info->vel, md_info->d_mass);
	Sum_Of_List << <1, 1024 >> >(md_info->atom_numbers, md_info->d_atom_ek, d_sum_of_atom_ek);

	//计算维里
	Add_Sum_List << <1, 1024 >> >(md_info->atom_numbers, md_info->d_atom_virial, d_virial);

	//合并起来
	Calculate_Pressure_Cuda << <1, 1 >> >(1.0 / Get_Volume(), d_sum_of_atom_ek, d_virial, d_pressure);

	if (is_download)
	{
		hipMemcpy(&h_pressure, d_pressure, sizeof(float), hipMemcpyDeviceToHost);
		return h_pressure;
	}
	else
	{
		return 0;
	}
}

float MD_INFORMATION::system_information::Get_Potential(int is_download)
{

	Add_Sum_List << <1, 1024 >> >(md_info->atom_numbers, md_info->d_atom_energy, d_potential);

	if (is_download)
	{
		hipMemcpy(&h_potential, d_potential, sizeof(float), hipMemcpyDeviceToHost);
		return h_potential;
	}
	else
	{
		return 0;
	}
}

void MD_INFORMATION::MD_Information_Frc_Device_To_Host()
{
	hipMemcpy(this->force, this->frc, sizeof(VECTOR)*this->atom_numbers, hipMemcpyDeviceToHost);
}

void MD_INFORMATION::MD_Information_Frc_Host_To_Device()
{
	hipMemcpy(this->frc, this->force, sizeof(VECTOR)*this->atom_numbers, hipMemcpyHostToDevice);
}

void MD_INFORMATION::Crd_Vel_Device_To_Host(int Do_Translation, int forced)
{
	if (output.current_crd_synchronized_step != sys.steps || forced)
	{	
		output.current_crd_synchronized_step = sys.steps;
		if (Do_Translation)
		{	
			hipMemcpy(coordinate, crd, sizeof(VECTOR) * atom_numbers, hipMemcpyDeviceToHost);
		}
		else
		{
			hipMemcpy(this->coordinate, this->crd, sizeof(VECTOR)*this->atom_numbers, hipMemcpyDeviceToHost);
		}
		hipMemcpy(this->velocity, this->vel, sizeof(VECTOR)*this->atom_numbers, hipMemcpyDeviceToHost);
	}
}

void MD_INFORMATION::Clear()
{

}


void MD_INFORMATION::molecule_information::Initial(CONTROLLER *controller, MD_INFORMATION *md_info)
{
	controller->printf("    Start initializing molecule list:\n");
	this->md_info = md_info;
	//分子拓扑是一个无向图，邻接表进行描述，通过排除表形成
	int edge_numbers = 2 * md_info->nb.excluded_atom_numbers;
	int *visited = NULL; //每个原子是否拜访过
	int *first_edge = NULL; //每个原子的第一个边（链表的头）
	int *edges = NULL;  //每个边的序号
	int *edge_next = NULL; //每个原子的边（链表结构）
	int *molecule_belongings = NULL; //每个原子属于的分子编号
	Malloc_Safely((void**)&visited, sizeof(int)*md_info->atom_numbers);
	Malloc_Safely((void**)&visited, sizeof(int)*md_info->atom_numbers);
	Malloc_Safely((void**)&first_edge, sizeof(int)*md_info->atom_numbers);
	Malloc_Safely((void**)&edges, sizeof(int)*edge_numbers);
	Malloc_Safely((void**)&edge_next, sizeof(int)*edge_numbers);
	Malloc_Safely((void**)&molecule_belongings, sizeof(int)*md_info->atom_numbers);
	//初始化链表
	for (int i = 0; i < md_info->atom_numbers; i++)
	{
		visited[i] = 0;
		first_edge[i] = -1;
	}
	int atom_i, atom_j, edge_count = 0;
	for (int i = 0; i < md_info->atom_numbers; i++)
	{
		atom_i = i;
		for (int j = md_info->nb.h_excluded_list_start[i] + md_info->nb.h_excluded_numbers[i] - 1; j >= md_info->nb.h_excluded_list_start[i]; j--) //这里使用倒序是因为链表构建是用的头插法
		{
			atom_j = md_info->nb.h_excluded_list[j];
			edge_next[edge_count] = first_edge[atom_i];
			first_edge[atom_i] = edge_count;
			edges[edge_count] = atom_j;
			edge_count++;
			edge_next[edge_count] = first_edge[atom_j];
			first_edge[atom_j] = edge_count;
			edges[edge_count] = atom_i;
			edge_count++;
		}
	}

	std::deque<int> queue;
	int atom;
	molecule_numbers = 0;
	for (int i = 0; i < md_info->atom_numbers; i++)
	{
		if (!visited[i])
		{
			visited[i] = 1;
			queue.push_back(i);
			while (!queue.empty())
			{
				atom = queue[0];
				queue.pop_front();
				molecule_belongings[atom] = molecule_numbers;
				edge_count = first_edge[atom];

				while (edge_count != -1)
				{
					atom = edges[edge_count];
					if (!visited[atom])
					{
						queue.push_back(atom);
						visited[atom] = 1;
					}
					edge_count = edge_next[edge_count];
				}
			}
			molecule_numbers += 1;
		}
	}
	printf("        molecule numbers is %d\n", molecule_numbers);
	Malloc_Safely((void**)&h_mass, sizeof(float)*molecule_numbers);
	Malloc_Safely((void**)&h_mass_inverse, sizeof(float)*molecule_numbers);
	Malloc_Safely((void**)&h_atom_start, sizeof(int)*molecule_numbers);
	Malloc_Safely((void**)&h_atom_end, sizeof(int)*molecule_numbers);
	Malloc_Safely((void**)&h_residue_start, sizeof(int)*molecule_numbers);
	Malloc_Safely((void**)&h_residue_end, sizeof(int)*molecule_numbers);
	Malloc_Safely((void**)&h_center_of_mass, sizeof(VECTOR)*molecule_numbers);

	Cuda_Malloc_Safely((void**)&d_mass, sizeof(float)*molecule_numbers);
	Cuda_Malloc_Safely((void**)&d_mass_inverse, sizeof(float)*molecule_numbers);
	Cuda_Malloc_Safely((void**)&d_atom_start, sizeof(int)*molecule_numbers);
	Cuda_Malloc_Safely((void**)&d_atom_end, sizeof(int)*molecule_numbers);
	Cuda_Malloc_Safely((void**)&d_residue_start, sizeof(int)*molecule_numbers);
	Cuda_Malloc_Safely((void**)&d_residue_end, sizeof(int)*molecule_numbers);
	Cuda_Malloc_Safely((void**)&d_center_of_mass, sizeof(VECTOR)*molecule_numbers);

	int molecule_j = 0;
	h_atom_start[0] = 0;
	//该判断基于一个分子的所有原子一定在列表里是连续的
	for (int i = 0; i < md_info->atom_numbers; i++)
	{
		if (molecule_belongings[i] != molecule_j)
		{
			h_atom_end[molecule_j] = i;
			molecule_j += 1;
			if (molecule_j < molecule_numbers)
				h_atom_start[molecule_j] = i;
		}
	}
	h_atom_end[molecule_numbers - 1] = md_info->atom_numbers;
	
	molecule_j = 0;
	h_residue_start[0] = 0;
	//该判断基于一个分子的所有残基一定在列表里是连续的，且原子在残基里也是连续的
	for (int i = 0; i < md_info->res.residue_numbers; i++)
	{
		if (md_info->res.h_res_start[i] == h_atom_end[molecule_j])
		{
			h_residue_end[molecule_j] = i;
			molecule_j += 1;
			if (molecule_j < molecule_numbers)
				h_residue_start[molecule_j] = i;
		}
	}
	h_residue_end[molecule_numbers - 1] = md_info->res.residue_numbers;

	for (int i = 0; i < molecule_numbers; i++)
	{
		h_mass[i] = 0;
		for (molecule_j = h_atom_start[i]; molecule_j < h_atom_end[i]; molecule_j++)
		{
			h_mass[i] += md_info->h_mass[molecule_j];
		}
		h_mass_inverse[i] = 1.0f / h_mass[i];
	}

	hipMemcpy(d_mass, h_mass, sizeof(float)*molecule_numbers, hipMemcpyHostToDevice);
	hipMemcpy(d_mass_inverse, h_mass_inverse, sizeof(float)*molecule_numbers, hipMemcpyHostToDevice);
	hipMemcpy(d_atom_start, h_atom_start, sizeof(int)*molecule_numbers, hipMemcpyHostToDevice);
	hipMemcpy(d_atom_end, h_atom_end, sizeof(int)*molecule_numbers, hipMemcpyHostToDevice);
	hipMemcpy(d_residue_start, h_residue_start, sizeof(int)*molecule_numbers, hipMemcpyHostToDevice);
	hipMemcpy(d_residue_end, h_residue_end, sizeof(int)*molecule_numbers, hipMemcpyHostToDevice);

	free(visited);
	free(first_edge);
	free(edges);
	free(edge_next);
	free(molecule_belongings);
	controller->printf("    End initializing molecule list\n\n");
}

void MD_INFORMATION::molecule_information::Molecule_Crd_Map(VECTOR *no_wrap_crd, float scaler)
{
	//为了有一个分子有很多残基，而其他分子都很小这种情况的并行，先求残基的质心
	Get_Center_Of_Mass << <64, 128 >> >(md_info->res.residue_numbers, md_info->res.d_res_start, md_info->res.d_res_end, no_wrap_crd, md_info->d_mass, md_info->res.d_mass_inverse, md_info->res.d_center_of_mass);
	//再用残基的质心求分子的质心
	Get_Center_Of_Mass << <32, 64 >> >(molecule_numbers, d_residue_start, d_residue_end, md_info->res.d_center_of_mass, md_info->res.d_mass, d_mass_inverse, d_center_of_mass);

	Map_Center_Of_Mass << <20, { 32, 4 } >> >(molecule_numbers, d_atom_start, d_atom_end, scaler, d_center_of_mass, md_info->sys.box_length, no_wrap_crd, md_info->crd);
}
