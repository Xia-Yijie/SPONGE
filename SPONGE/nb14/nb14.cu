#include "hip/hip_runtime.h"
﻿#include "nb14.cuh"

static __global__ void Dihedral_14_LJ_Energy(const int dihedral_14_numbers, const UNSIGNED_INT_VECTOR *uint_crd, const VECTOR boxlength,
	const int *a_14, const int *b_14, const float *lj_A, const float *lj_B, float *ene)
{
	int dihedral_14_i = blockDim.x*blockIdx.x + threadIdx.x;
	if (dihedral_14_i < dihedral_14_numbers)
	{
		int atom_i = a_14[dihedral_14_i];
		int atom_j = b_14[dihedral_14_i];

		UNSIGNED_INT_VECTOR r1 = uint_crd[atom_i];
		UNSIGNED_INT_VECTOR r2 = uint_crd[atom_j];

		int int_x;
		int int_y;
		int int_z;
		VECTOR dr;
		float dr2;
		float dr_2;
		float dr_4;
		float dr_6;
		float dr_12;
		float ene_lin = 0.;

		int_x = r2.uint_x - r1.uint_x;
		int_y = r2.uint_y - r1.uint_y;
		int_z = r2.uint_z - r1.uint_z;
		dr.x = boxlength.x*int_x;
		dr.y = boxlength.y*int_y;
		dr.z = boxlength.z*int_z;
		dr2 = dr.x*dr.x + dr.y*dr.y + dr.z*dr.z;

		dr_2 = 1. / dr2;
		dr_4 = dr_2*dr_2;
		dr_6 = dr_4*dr_2;
		dr_12 = dr_6*dr_6;


		ene_lin = 0.08333333*lj_A[dihedral_14_i] * dr_12
			- 0.1666666*lj_B[dihedral_14_i] * dr_6;//LJ的A,B系数已经乘以12和6因此要反乘

		ene[dihedral_14_i] = ene_lin;
	}
}


static __global__ void Dihedral_14_CF_Energy(const int dihedral_14_numbers, const UNSIGNED_INT_VECTOR *uint_crd, const float *charge, const VECTOR boxlength,
	const int *a_14, const int *b_14, const float *cf_scale_factor, float *ene)
{
	int dihedral_14_i = blockDim.x*blockIdx.x + threadIdx.x;
	if (dihedral_14_i < dihedral_14_numbers)
	{
		int atom_i = a_14[dihedral_14_i];
		int atom_j = b_14[dihedral_14_i];

		UNSIGNED_INT_VECTOR r1 = uint_crd[atom_i];
		UNSIGNED_INT_VECTOR r2 = uint_crd[atom_j];

		int int_x;
		int int_y;
		int int_z;
		VECTOR dr;
		float r_1;
		float ene_lin = 0.;

		int_x = r2.uint_x - r1.uint_x;
		int_y = r2.uint_y - r1.uint_y;
		int_z = r2.uint_z - r1.uint_z;
		dr.x = boxlength.x*int_x;
		dr.y = boxlength.y*int_y;
		dr.z = boxlength.z*int_z;
		r_1 = rnorm3df(dr.x, dr.y, dr.z);

		ene_lin = charge[atom_i] * charge[atom_j] * r_1;

		ene_lin *= cf_scale_factor[dihedral_14_i];

		ene[dihedral_14_i] = ene_lin;
	}
}


static __global__ void Dihedral_14_LJ_CF_Force_With_Atom_Energy_And_Virial_Cuda(const int dihedral_14_numbers, const UNSIGNED_INT_VECTOR *uint_crd, const VECTOR scaler,
	const int *a_14, const int *b_14, const float *cf_scale_factor, const float *charge, const float *lj_A, const float *lj_B, VECTOR *frc,	float *atom_energy, float *atom_virial)
{
	int dihedral_14_i = blockDim.x*blockIdx.x + threadIdx.x;
	if (dihedral_14_i < dihedral_14_numbers)
	{
		UNSIGNED_INT_VECTOR r1, r2;
		VECTOR dr;
		float dr_abs;
		float dr2;
		float dr_1;
		float dr_2;
		float dr_4;
		float dr_8;
		float dr_14;
		float frc_abs = 0.;
		VECTOR temp_frc;

		float ene_lin;
		float ene_lin2;

		int atom_i = a_14[dihedral_14_i];
		int atom_j = b_14[dihedral_14_i];

		r1 = uint_crd[atom_i];
		r2 = uint_crd[atom_j];

		dr = Get_Periodic_Displacement(r2, r1, scaler);

		dr2 = dr.x*dr.x + dr.y*dr.y + dr.z*dr.z;

		dr_2 = 1.0 / dr2;
		dr_4 = dr_2*dr_2;
		dr_8 = dr_4*dr_4;
		dr_14 = dr_8*dr_4*dr_2;
		dr_abs = norm3df(dr.x, dr.y, dr.z);
		dr_1 = 1. / dr_abs;

		//CF
		float charge_i = charge[atom_i];
		float charge_j = charge[atom_j];
		float frc_cf_abs;
		frc_cf_abs = cf_scale_factor[dihedral_14_i] * dr_2 *dr_1;
		frc_cf_abs = -charge_i * charge_j*frc_cf_abs;
		//LJ
		frc_abs = -lj_A[dihedral_14_i] * dr_14
			+ lj_B[dihedral_14_i] * dr_8;

		frc_abs += frc_cf_abs;
		temp_frc.x = frc_abs*dr.x;
		temp_frc.y = frc_abs*dr.y;
		temp_frc.z = frc_abs*dr.z;



		atomicAdd(&frc[atom_j].x, -temp_frc.x);
		atomicAdd(&frc[atom_j].y, -temp_frc.y);
		atomicAdd(&frc[atom_j].z, -temp_frc.z);
		atomicAdd(&frc[atom_i].x, temp_frc.x);
		atomicAdd(&frc[atom_i].y, temp_frc.y);
		atomicAdd(&frc[atom_i].z, temp_frc.z);

		//能量
		ene_lin = charge_i*charge_j*dr_1;
		ene_lin *= cf_scale_factor[dihedral_14_i];
		ene_lin2 = 0.08333333*lj_A[dihedral_14_i] * dr_4*dr_8
			- 0.1666666*lj_B[dihedral_14_i] * dr_4*dr_2;//LJ的A,B系数已经乘以12和6因此要反乘

		atomicAdd(&atom_energy[atom_i], ene_lin + ene_lin2);

		//维里
		atomicAdd(&atom_virial[atom_i], -temp_frc * dr);
	}
}

void NON_BOND_14::Initial(CONTROLLER *controller, const float *LJ_type_A, const float *LJ_type_B, const int *lj_atom_type, char *module_name)
{	
	if (module_name == NULL)
	{
		strcpy(this->module_name, "nb14");
	}
	else
	{
		strcpy(this->module_name, module_name);
	}

	char file_name_suffix[CHAR_LENGTH_MAX], file_name_suffix2[CHAR_LENGTH_MAX];
	FILE *fp = NULL, *fp2 = NULL;
	float h_lj_scale_factor = 0;

	sprintf(file_name_suffix, "in_file");
	if (controller[0].Command_Exist(this->module_name, file_name_suffix))
	{
		Open_File_Safely(&fp, controller[0].Command(this->module_name, file_name_suffix), "r");
		int scanf_ret = fscanf(fp, "%d", &nb14_numbers);
	}

	int extra_numbers = 0;
	sprintf(file_name_suffix2, "extra_in_file");
	if (controller[0].Command_Exist(this->module_name, file_name_suffix2))
	{
		Open_File_Safely(&fp2, controller[0].Command(this->module_name, file_name_suffix2), "r");
		int scanf_ret = fscanf(fp2, "%d", &extra_numbers);
	}
	nb14_numbers += extra_numbers;
	if (nb14_numbers > 0)
	{
		controller[0].printf("START INITIALIZING NB14 (%s_%s):\n", this->module_name, file_name_suffix);
		controller[0].printf("    non-bond 14 numbers is %d\n", nb14_numbers);
		Memory_Allocate();
		int smallertype, biggertype, temp;
		for (int i = extra_numbers; i < nb14_numbers; i++)
		{
			int scanf_ret = fscanf(fp, "%d %d %f %f", h_atom_a + i, h_atom_b + i, &h_lj_scale_factor, h_cf_scale_factor + i);
			smallertype = lj_atom_type[h_atom_a[i]];
			biggertype = lj_atom_type[h_atom_b[i]];
			if (smallertype > biggertype)
			{
				temp = smallertype;
				smallertype = biggertype;
				biggertype = temp;
			}
			temp = biggertype * (biggertype + 1) / 2 + smallertype;
			h_A[i] = h_lj_scale_factor * LJ_type_A[temp];
			h_B[i] = h_lj_scale_factor * LJ_type_B[temp];
		}
		for (int i = 0; i < extra_numbers; i++)
		{
			int scanf_ret = fscanf(fp, "%d %d %f %f %f", h_atom_a + i, h_atom_b + i, h_A + i, h_B + i, h_cf_scale_factor + i);
			h_A[i] *= 12;
			h_B[i] *= 6;
		}
		if (fp != NULL)  fclose(fp);
		if (fp2 != NULL) fclose(fp2);
		Parameter_Host_To_Device();
		is_initialized = 1;
	}
	else if (controller[0].Command_Exist("amber_parm7"))
	{
		controller[0].printf("START INITIALIZING NB14 (amber_parm7):\n");
		Read_Information_From_AMBERFILE(controller[0].Command("amber_parm7"), controller[0], LJ_type_A, LJ_type_B, lj_atom_type);
		if (nb14_numbers > 0)
			is_initialized = 1;
	}
	else
	{
		controller[0].printf("NB14 IS NOT INITIALIZED\n\n");
	}


	if (is_initialized && !is_controller_printf_initialized)
	{
		controller[0].Step_Print_Initial("nb14_LJ", "%.2f");
		controller[0].Step_Print_Initial("nb14_EE", "%.2f");
		is_controller_printf_initialized = 1;
		controller[0].printf("    structure last modify date is %d\n", last_modify_date);
	}
	if (is_initialized == 1)
	{
		controller[0].printf("END INITIALIZING NB14\n\n");
	}
}

void NON_BOND_14::Parameter_Host_To_Device()
{
	hipMemcpy(this->d_atom_a, this->h_atom_a, sizeof(int)*this->nb14_numbers, hipMemcpyHostToDevice);
	hipMemcpy(this->d_atom_b, this->h_atom_b, sizeof(int)*this->nb14_numbers, hipMemcpyHostToDevice);
	hipMemcpy(this->d_A, this->h_A, sizeof(int)*this->nb14_numbers, hipMemcpyHostToDevice);
	hipMemcpy(this->d_B, this->h_B, sizeof(int)*this->nb14_numbers, hipMemcpyHostToDevice);
	hipMemcpy(this->d_cf_scale_factor, this->h_cf_scale_factor, sizeof(int)*this->nb14_numbers, hipMemcpyHostToDevice);
}

void NON_BOND_14::Memory_Allocate()
{
	if (!Malloc_Safely((void**)&this->h_atom_a, sizeof(int)* this->nb14_numbers))
		printf("Error occurs when malloc NON_BOND_14::h_atom_a in NON_BOND_14::Nb14_Initial");
	if (!Malloc_Safely((void**)&this->h_atom_b, sizeof(int)* this->nb14_numbers))
		printf("Error occurs when malloc NON_BOND_14::h_atom_b in NON_BOND_14::Nb14_Initial");
	if (!Cuda_Malloc_Safely((void**)&this->d_atom_a, sizeof(int)* this->nb14_numbers))
		printf("Error occurs when malloc NON_BOND_14::d_atom_a in NON_BOND_14::Nb14_Initial");
	if (!Cuda_Malloc_Safely((void**)&this->d_atom_b, sizeof(int)* this->nb14_numbers))
		printf("Error occurs when malloc NON_BOND_14::d_atom_b in NON_BOND_14::Nb14_Initial");

	if (!Malloc_Safely((void**)&this->h_A, sizeof(float)* this->nb14_numbers))
		printf("Error occurs when malloc NON_BOND_14::h_A in NON_BOND_14::Nb14_Initial");
	if (!Malloc_Safely((void**)&this->h_B, sizeof(float)* this->nb14_numbers))
		printf("Error occurs when malloc NON_BOND_14::h_B in NON_BOND_14::Nb14_Initial");
	if (!Malloc_Safely((void**)&this->h_cf_scale_factor, sizeof(float)* this->nb14_numbers))
		printf("Error occurs when malloc NON_BOND_14::h_cf_scale_factor in NON_BOND_14::Nb14_Initial");
	if (!Cuda_Malloc_Safely((void**)&this->d_A, sizeof(float)* this->nb14_numbers))
		printf("Error occurs when malloc NON_BOND_14::d_A in NON_BOND_14::Nb14_Initial");
	if (!Cuda_Malloc_Safely((void**)&this->d_B, sizeof(float)* this->nb14_numbers))
		printf("Error occurs when malloc NON_BOND_14::d_B in NON_BOND_14::Nb14_Initial");
	if (!Cuda_Malloc_Safely((void**)&this->d_cf_scale_factor, sizeof(float)* this->nb14_numbers))
		printf("Error occurs when malloc NON_BOND_14::d_cf_scale_factor in NON_BOND_14::Nb14_Initial");
	if (!Cuda_Malloc_Safely((void**)&this->d_nb14_energy, sizeof(float)* this->nb14_numbers))
		printf("Error occurs when malloc NON_BOND_14::d_nb14_ene in NON_BOND_14::Nb14_Initial");
	if (!Cuda_Malloc_Safely((void**)&this->d_nb14_lj_energy_sum, sizeof(float)))
		printf("Error occurs when malloc NON_BOND_14::d_nb14_lj_ene_sum in NON_BOND_14::Nb14_Initial");
	if (!Cuda_Malloc_Safely((void**)&this->d_nb14_cf_energy_sum, sizeof(float)))
		printf("Error occurs when malloc NON_BOND_14::d_nb14_cf_ene_sum in NON_BOND_14::Nb14_Initial");
}

void NON_BOND_14::Clear()
{
	if (is_initialized)
	{
		is_initialized = 0;

		free(h_atom_a);
		free(h_atom_b);
		free(h_A);
		free(h_B);
		free(h_cf_scale_factor);

		hipFree(d_atom_a);
		hipFree(d_atom_b);
		free(d_A);
		free(d_B);
		hipFree(d_cf_scale_factor);
		hipFree(d_nb14_energy);
		hipFree(d_nb14_lj_energy_sum);
		hipFree(d_nb14_cf_energy_sum);


		h_atom_a = NULL;
		h_atom_b = NULL;
		h_A = NULL;
		h_B = NULL;
		h_cf_scale_factor = NULL;

		d_atom_a = NULL;
		d_atom_b = NULL;
		d_A = NULL;
		d_B = NULL;
		d_cf_scale_factor = NULL;
		d_nb14_energy = NULL;
		d_nb14_lj_energy_sum = NULL;
		d_nb14_cf_energy_sum = NULL;
	}
}

void NON_BOND_14::Read_Information_From_AMBERFILE(const char *file_name, CONTROLLER controller, const float *LJ_type_A, const float *LJ_type_B, const int *lj_atom_type)
{
	int dihedral_numbers, dihedral_type_numbers, dihedral_with_hydrogen;
	FILE *parm = NULL;
	Open_File_Safely(&parm, file_name, "r");
	char temps[CHAR_LENGTH_MAX];
	char temp_first_str[CHAR_LENGTH_MAX];
	char temp_second_str[CHAR_LENGTH_MAX];
	int i, tempi, tempi2, tempa, tempb;
	float *lj_scale_type_cpu = NULL, *cf_scale_type_cpu = NULL;
	controller.printf("    Reading non-bond 14 information from AMBER file:\n");
	while (true)
	{
		if (fgets(temps, CHAR_LENGTH_MAX, parm) == NULL)
		{
			break;
		}
		if (sscanf(temps, "%s %s", temp_first_str, temp_second_str) != 2)
		{
			continue;
		}
		if (strcmp(temp_first_str, "%FLAG") == 0
			&& strcmp(temp_second_str, "POINTERS") == 0)
		{
			char *get_ret = fgets(temps, CHAR_LENGTH_MAX, parm);

			for (i = 0; i < 6; i++)
				int scanf_ret = fscanf(parm, "%d", &tempi);

			int scanf_ret = fscanf(parm, "%d", &dihedral_with_hydrogen);
			scanf_ret = fscanf(parm, "%d", &dihedral_numbers);
			dihedral_numbers += dihedral_with_hydrogen;

			for (i = 0; i < 9; i++)
				scanf_ret = fscanf(parm, "%d", &tempi);

			scanf_ret = fscanf(parm, "%d", &dihedral_type_numbers);


			nb14_numbers = dihedral_numbers;
			Memory_Allocate();
			nb14_numbers = 0;
			Malloc_Safely((void**)&cf_scale_type_cpu, sizeof(float)* dihedral_type_numbers);
			Malloc_Safely((void**)&lj_scale_type_cpu, sizeof(float)* dihedral_type_numbers);
		}

		if (strcmp(temp_first_str, "%FLAG") == 0
			&& strcmp(temp_second_str, "SCEE_SCALE_FACTOR") == 0)
		{
			controller.printf("\tread dihedral 1-4 CF scale factor\n");
			char *get_ret = fgets(temps, CHAR_LENGTH_MAX, parm);
			for (i = 0; i < dihedral_type_numbers; i++)
			{
				int scanf_ret = fscanf(parm, "%f", &cf_scale_type_cpu[i]);
			}
				

		}
		if (strcmp(temp_first_str, "%FLAG") == 0
			&& strcmp(temp_second_str, "SCNB_SCALE_FACTOR") == 0)
		{
			controller.printf("\tread dihedral 1-4 LJ scale factor\n");
			char *get_ret = fgets(temps, CHAR_LENGTH_MAX, parm);
			for (i = 0; i < dihedral_type_numbers; i++)
				int scanf_ret = fscanf(parm, "%f", &lj_scale_type_cpu[i]);
		}
		if (strcmp(temp_first_str, "%FLAG") == 0
			&& strcmp(temp_second_str, "DIHEDRALS_INC_HYDROGEN") == 0)
		{
			float h_lj_scale_factor;
			int smallertype, biggertype, temptype;
			char *get_ret = fgets(temps, CHAR_LENGTH_MAX, parm);
			for (i = 0; i < dihedral_with_hydrogen; i++)
			{
				int scanf_ret = fscanf(parm, "%d\n", &tempa);
				scanf_ret = fscanf(parm, "%d\n", &tempi);
				scanf_ret = fscanf(parm, "%d\n", &tempi2);
				scanf_ret = fscanf(parm, "%d\n", &tempb);
				scanf_ret = fscanf(parm, "%d\n", &tempi);

				tempi -= 1;
				if (tempi2>0)
				{
					h_atom_a[nb14_numbers] = tempa / 3;
					h_atom_b[nb14_numbers] = abs(tempb / 3);
					h_lj_scale_factor = lj_scale_type_cpu[tempi];

					if (h_lj_scale_factor != 0)
					{
						h_lj_scale_factor = 1.0f / h_lj_scale_factor;
					}
					h_cf_scale_factor[nb14_numbers] = cf_scale_type_cpu[tempi];
					if (h_cf_scale_factor[nb14_numbers] != 0)
						h_cf_scale_factor[nb14_numbers] = 1.0f / h_cf_scale_factor[nb14_numbers];
					
					smallertype = lj_atom_type[h_atom_a[nb14_numbers]];
					biggertype = lj_atom_type[h_atom_b[nb14_numbers]];
					if (smallertype > biggertype)
					{
						temptype = smallertype;
						smallertype = biggertype;
						biggertype = temptype;
					}
					temptype = biggertype * (biggertype + 1) / 2 + smallertype;
					h_A[nb14_numbers] = h_lj_scale_factor * LJ_type_A[temptype];
					h_B[nb14_numbers] = h_lj_scale_factor * LJ_type_B[temptype];
					nb14_numbers += 1;
				}
			}
		}
		if (strcmp(temp_first_str, "%FLAG") == 0
			&& strcmp(temp_second_str, "DIHEDRALS_WITHOUT_HYDROGEN") == 0)
		{
			char *get_ret = fgets(temps, CHAR_LENGTH_MAX, parm);
			float h_lj_scale_factor;
			int smallertype, biggertype, temptype;
			for (i = dihedral_with_hydrogen; i < dihedral_numbers; i++)
			{
				int scanf_ret = fscanf(parm, "%d\n", &tempa);
				scanf_ret = fscanf(parm, "%d\n", &tempi);
				scanf_ret = fscanf(parm, "%d\n", &tempi2);
				scanf_ret = fscanf(parm, "%d\n", &tempb);
				scanf_ret = fscanf(parm, "%d\n", &tempi);

				tempi -= 1;
				if (tempi2>0)
				{
					h_atom_a[nb14_numbers] = tempa / 3;
					h_atom_b[nb14_numbers] = abs(tempb / 3);
					h_lj_scale_factor = lj_scale_type_cpu[tempi];

					if (h_lj_scale_factor != 0)
					{
						h_lj_scale_factor = 1.0f / h_lj_scale_factor;
					}
					
					smallertype = lj_atom_type[h_atom_a[nb14_numbers]];
					biggertype = lj_atom_type[h_atom_b[nb14_numbers]];
					if (smallertype > biggertype)
					{
						temptype = smallertype;
						smallertype = biggertype;
						biggertype = temptype;
					}
					temptype = biggertype * (biggertype + 1) / 2 + smallertype;
					h_A[nb14_numbers] = h_lj_scale_factor * LJ_type_A[temptype];
					h_B[nb14_numbers] = h_lj_scale_factor * LJ_type_B[temptype];


					h_cf_scale_factor[nb14_numbers] = cf_scale_type_cpu[tempi];
					if (h_cf_scale_factor[nb14_numbers] != 0)
						h_cf_scale_factor[nb14_numbers] = 1.0f / h_cf_scale_factor[nb14_numbers];
					nb14_numbers += 1;
				}
			}
		}
	}
	
	free(lj_scale_type_cpu);
	free(cf_scale_type_cpu);
	fclose(parm);
	controller.printf("        nb14_number is %d\n",nb14_numbers);
	controller.printf("    End reading nb14 information from AMBER file\n");
	Parameter_Host_To_Device();
}


float NON_BOND_14::Get_14_LJ_Energy(const UNSIGNED_INT_VECTOR *uint_crd, const VECTOR scaler, int is_download)
{
	if (is_initialized)
	{
		Dihedral_14_LJ_Energy << <(unsigned int)ceilf((float)nb14_numbers / threads_per_block), threads_per_block >> >
			(nb14_numbers, uint_crd, scaler,
			d_atom_a, d_atom_b, d_A, d_B,
			d_nb14_energy);
		Sum_Of_List << <1, 1024 >> >
			(nb14_numbers, d_nb14_energy, d_nb14_lj_energy_sum);
		if (is_download)
		{
			hipMemcpy(&h_nb14_lj_energy_sum, this->d_nb14_lj_energy_sum, sizeof(float), hipMemcpyDeviceToHost);
			return h_nb14_lj_energy_sum;
		}
		else
		{
			return 0;
		}
	}
	return NAN;
}


float NON_BOND_14::Get_14_CF_Energy(const UNSIGNED_INT_VECTOR *uint_crd, const float *charge, const VECTOR scaler, int is_download)
{
	if (is_initialized)
	{
		Dihedral_14_CF_Energy << <(unsigned int)ceilf((float)nb14_numbers / threads_per_block), threads_per_block >> >
			(nb14_numbers, uint_crd, charge, scaler,
			d_atom_a, d_atom_b, d_cf_scale_factor,
			d_nb14_energy);
		Sum_Of_List << <1, 1024 >> >
			(nb14_numbers, d_nb14_energy, d_nb14_cf_energy_sum);
		if (is_download)
		{
			hipMemcpy(&h_nb14_cf_energy_sum, this->d_nb14_cf_energy_sum, sizeof(float), hipMemcpyDeviceToHost);
			return h_nb14_cf_energy_sum;
		}
		else
		{
			return 0;
		}
	}
	return NAN;
}


void NON_BOND_14::Non_Bond_14_LJ_CF_Force_With_Atom_Energy_And_Virial(const UNSIGNED_INT_VECTOR *uint_crd, const float *charge, const VECTOR scaler, VECTOR *frc, float *atom_energy, float *atom_virial)
{
	if (is_initialized)
	{
		Dihedral_14_LJ_CF_Force_With_Atom_Energy_And_Virial_Cuda << <(unsigned int)ceilf((float)nb14_numbers / threads_per_block), threads_per_block >> >(nb14_numbers, uint_crd, scaler,
			d_atom_a, d_atom_b, d_cf_scale_factor, charge, d_A, d_B, frc, atom_energy, atom_virial);
	}
}

