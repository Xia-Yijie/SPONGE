#include "hip/hip_runtime.h"
#include "main.cuh"

CONTROLLER controller;
MD_INFORMATION md_info;
MIDDLE_Langevin_INFORMATION middle_langevin;
ANDERSEN_THERMOSTAT_INFORMATION ad_thermo;
BERENDSEN_THERMOSTAT_INFORMATION bd_thermo;
NOSE_HOOVER_CHAIN_INFORMATION nhc;
BOND bond;
ANGLE angle;
DIHEDRAL dihedral;
NON_BOND_14 nb14;
NEIGHBOR_LIST neighbor_list;
LENNARD_JONES_INFORMATION lj;
Particle_Mesh_Ewald pme;
RESTRAIN_INFORMATION restrain;
SIMPLE_CONSTARIN simple_constrain;
VIRTUAL_INFORMATION vatom;
CoordinateMolecularMap mol_map;
MC_BAROSTAT_INFORMATION mc_baro;
BERENDSEN_BAROSTAT_INFORMATION bd_baro;

int main(int argc, char *argv[])
{
	Main_Initial(argc, argv);

	for (md_info.sys.steps = 1; md_info.sys.steps <= md_info.sys.step_limit; md_info.sys.steps++)
	{
		Main_Calculate_Force();
		Main_Iteration();
		Main_Print();
	}

	Main_Clear();
	return 0;
}

void Main_Initial(int argc, char *argv[])
{
	controller.Initial(argc, argv);
	md_info.Initial(&controller);
	controller.Command_Exist("end_pause");

	if  (md_info.mode >= md_info.NVT && controller.Command_Choice("thermostat", "langevin"))
	{
		middle_langevin.Initial(&controller, md_info.atom_numbers, md_info.sys.target_temperature, md_info.h_mass);
	}
	if (md_info.mode >= md_info.NVT && controller.Command_Choice("thermostat", "berendsen_thermostat"))
	{
		bd_thermo.Initial(&controller, md_info.sys.target_temperature);
	}
	if (md_info.mode >= md_info.NVT && controller.Command_Choice("thermostat", "andersen_thermostat"))
	{
		ad_thermo.Initial(&controller, md_info.sys.target_temperature, md_info.atom_numbers, md_info.h_mass);
	}
	if (md_info.mode >= md_info.NVT && controller.Command_Choice("thermostat", "nose_hoover_chain"))
	{
		nhc.Initial(&controller, md_info.sys.target_temperature);
	}

	neighbor_list.Initial(&controller, md_info.atom_numbers, md_info.sys.box_length, md_info.nb.cutoff, md_info.nb.skin);
	neighbor_list.Neighbor_List_Update(md_info.crd, md_info.nb.d_excluded_list_start, md_info.nb.d_excluded_list, md_info.nb.d_excluded_numbers, neighbor_list.FORCED_UPDATE);
	lj.Initial(&controller, md_info.nb.cutoff, md_info.sys.box_length);
	pme.Initial(&controller, md_info.atom_numbers, md_info.sys.box_length, md_info.nb.cutoff);
	
	nb14.Initial(&controller);
	bond.Initial(&controller);
	angle.Initial(&controller);
	dihedral.Initial(&controller);
	
	restrain.Initial(&controller, md_info.atom_numbers, md_info.crd);

	if (controller.Command_Exist("constrain_mode") && controller.Command_Choice("constrain_mode", "simple_constrain"))
	{	
		simple_constrain.Add_HBond_To_Constrain_Pair(&controller, bond.bond_numbers, bond.h_atom_a, bond.h_atom_b, bond.h_r0, md_info.h_mass);
		simple_constrain.Add_HAngle_To_Constrain_Pair(&controller, angle.angle_numbers, angle.h_atom_a, angle.h_atom_b, angle.h_atom_c, angle.h_angle_theta0, md_info.h_mass);
		if (middle_langevin.is_initialized)
			simple_constrain.Initial_Simple_Constrain(&controller, md_info.atom_numbers, md_info.dt, md_info.sys.box_length, middle_langevin.exp_gamma, 0, md_info.h_mass, &md_info.sys.freedom);
		else
			simple_constrain.Initial_Simple_Constrain(&controller, md_info.atom_numbers, md_info.dt, md_info.sys.box_length, 1.0, md_info.mode == md_info.MINIMIZATION, md_info.h_mass, &md_info.sys.freedom);
	}

	if (md_info.mode == md_info.NPT && controller.Command_Choice("barostat", "monte_carlo_barostat"))
	{
		mc_baro.Initial(&controller, md_info.atom_numbers, md_info.sys.target_pressure, md_info.sys.box_length, md_info.res.is_initialized);
	}
	if (md_info.mode == md_info.NPT && controller.Command_Choice("barostat", "berendsen_barostat"))
	{
		bd_baro.Initial(&controller, md_info.sys.target_pressure, md_info.sys.box_length);
	}

	vatom.Initial(&controller, md_info.atom_numbers, &md_info.sys.freedom);
	mol_map.Initial(md_info.atom_numbers, md_info.sys.box_length, md_info.crd,
		md_info.nb.excluded_atom_numbers, md_info.nb.h_excluded_numbers, md_info.nb.h_excluded_list_start, md_info.nb.h_excluded_list);

	controller.Input_Check();
	controller.Print_First_Line_To_Mdout();
	controller.core_time.Start();
}

void Main_Clear()
{
	controller.core_time.Stop();
	controller.printf("Core Run Wall Time: %f second(s)\n", controller.core_time.time);
	if (md_info.mode != md_info.MINIMIZATION)
	{
		controller.simulation_speed = md_info.sys.steps * md_info.dt / CONSTANT_TIME_CONVERTION / controller.core_time.time * 86.4;
		controller.printf("Core Run Speed: %f ns/day\n", controller.simulation_speed);
	}
	else
	{
		controller.simulation_speed = md_info.sys.steps / controller.core_time.time * 3600;
		controller.printf("Core Run Speed: %f steps/hour\n", controller.simulation_speed);
	}
	fcloseall();

	if (controller.Command_Exist("end_pause"))
	{
		if (atoi(controller.Command("end_pause")) == 1)
		{
			printf("End Pause\n");
			getchar();
		}
	}
}

void Main_Calculate_Force()
{
	md_info.MD_Information_Crd_To_Uint_Crd();
	md_info.MD_Reset_Atom_Energy_And_Virial_And_Force();
	if (md_info.sys.steps % md_info.output.write_trajectory_interval == 0 || (md_info.mode == md_info.MINIMIZATION && md_info.min.dynamic_dt))
	{
		md_info.need_potential = 1;
	}
	mc_baro.Ask_For_Calculate_Potential(md_info.sys.steps, &md_info.need_potential);
	bd_baro.Ask_For_Calculate_Pressure(md_info.sys.steps, &md_info.need_pressure);

	lj.LJ_PME_Direct_Force_With_Atom_Energy_And_Virial(md_info.atom_numbers, md_info.uint_crd, md_info.d_charge, md_info.frc,
		neighbor_list.d_nl, pme.beta, md_info.need_potential, md_info.d_atom_energy, md_info.need_pressure, md_info.d_atom_virial, pme.d_direct_atom_energy);
	lj.Long_Range_Correction(md_info.need_pressure, md_info.sys.d_virial,
		md_info.need_potential, md_info.sys.d_potential);
	
	pme.PME_Excluded_Force_With_Atom_Energy(md_info.uint_crd, md_info.pbc.uint_dr_to_dr_cof, md_info.d_charge,
		md_info.nb.d_excluded_list_start, md_info.nb.d_excluded_list, md_info.nb.d_excluded_numbers, md_info.frc, pme.d_correction_atom_energy);

	pme.PME_Reciprocal_Force_With_Energy_And_Virial(md_info.uint_crd, md_info.d_charge, md_info.frc, md_info.need_pressure, md_info.need_potential, md_info.sys.d_virial, md_info.sys.d_potential);

	nb14.Non_Bond_14_LJ_CF_Force_With_Atom_Energy_And_Virial(lj.uint_crd_with_LJ, md_info.pbc.uint_dr_to_dr_cof, lj.d_LJ_A, lj.d_LJ_B, md_info.frc, md_info.d_atom_energy, md_info.d_atom_virial);

	bond.Bond_Force_With_Atom_Energy_And_Virial(md_info.uint_crd, md_info.pbc.uint_dr_to_dr_cof, md_info.frc, md_info.d_atom_energy, md_info.d_atom_virial);
	angle.Angle_Force_With_Atom_Energy(md_info.uint_crd, md_info.pbc.uint_dr_to_dr_cof, md_info.frc, md_info.d_atom_energy);
	dihedral.Dihedral_Force_With_Atom_Energy(md_info.uint_crd, md_info.pbc.uint_dr_to_dr_cof, md_info.frc, md_info.d_atom_energy);

	restrain.Restraint(md_info.crd, md_info.sys.box_length, md_info.d_atom_energy, md_info.d_atom_virial, md_info.frc);

	vatom.Force_Redistribute(md_info.uint_crd, md_info.pbc.uint_dr_to_dr_cof, md_info.frc);

	md_info.Calculate_Pressure_And_Potential_If_Needed();
}

void Main_Iteration()
{
	//����������mc��ѹ����
	if (mc_baro.is_initialized && md_info.sys.steps % mc_baro.update_interval == 0)
	{
		//������
		mc_baro.energy_old = md_info.sys.h_potential;
		hipMemcpy(mc_baro.frc_backup, md_info.frc, sizeof(VECTOR)*md_info.atom_numbers, hipMemcpyDeviceToDevice);
		hipMemcpy(mc_baro.crd_backup, md_info.crd, sizeof(VECTOR)*md_info.atom_numbers, hipMemcpyDeviceToDevice);

		mc_baro.Volume_Change_Attempt(md_info.sys.box_length);


		//�ı�����
		if (mc_baro.scale_coordinate_by_molecule)
		{
			mol_map.Calculate_No_Wrap_Crd(md_info.crd);
			md_info.mol.Molecule_Crd_Map(mol_map.nowrap_crd, mc_baro.crd_scale_factor);
			mol_map.Refresh_BoxMapTimes(md_info.crd);
		}
		else
		{
			Scale_List((float*)md_info.crd, mc_baro.crd_scale_factor, 3 * md_info.atom_numbers);
		}

		
		//�ı����
		Main_Volume_Change(mc_baro.crd_scale_factor);
		//������
		Main_Calculate_Force();
		mc_baro.energy_new = md_info.sys.h_potential;

		//������ܸ���
		if (mc_baro.scale_coordinate_by_molecule)
			mc_baro.extra_term = md_info.sys.target_pressure * mc_baro.DeltaV - (md_info.mol.molecule_numbers - 1) * CONSTANT_kB * md_info.sys.target_temperature * logf(mc_baro.VDevided);
		else
			mc_baro.extra_term = md_info.sys.target_pressure * mc_baro.DeltaV - (md_info.atom_numbers - 1) * CONSTANT_kB * md_info.sys.target_temperature * logf(mc_baro.VDevided);

		mc_baro.accept_possibility = mc_baro.energy_new - mc_baro.energy_old + mc_baro.extra_term;
		mc_baro.accept_possibility = expf(-mc_baro.accept_possibility / (CONSTANT_kB * md_info.sys.target_temperature));

		//�ж��Ƿ����
		if (mc_baro.Check_MC_Barostat_Accept())
		{
			//���ܾ��˾ͻ�ԭ
			mc_baro.crd_scale_factor = 1.0 / mc_baro.crd_scale_factor;
			hipMemcpy(md_info.crd, mc_baro.crd_backup, sizeof(VECTOR)*md_info.atom_numbers, hipMemcpyDeviceToDevice);
			Main_Volume_Change(mc_baro.crd_scale_factor);
			neighbor_list.Neighbor_List_Update(md_info.crd, md_info.nb.d_excluded_list_start, md_info.nb.d_excluded_list, md_info.nb.d_excluded_numbers, neighbor_list.CONDITIONAL_UPDATE, neighbor_list.FORCED_CHECK);
			hipMemcpy(md_info.frc, mc_baro.frc_backup, sizeof(VECTOR)*md_info.atom_numbers, hipMemcpyDeviceToDevice);
		}
		//���ܺ�����仯�����������������̫�ࣨ~1 ns���Ժ����¶Բ���ģ���ʼ��
		if ((!mc_baro.reject && (mc_baro.newV > 1.331 * mc_baro.V0 || mc_baro.newV < 0.729 * mc_baro.V0)))
		{
			Main_Volume_Change_Largely();
			mc_baro.V0 = mc_baro.newV;
		}

		//�����仯ֵ���е���
		mc_baro.Delta_V_Max_Update();
	}

	simple_constrain.Remember_Last_Coordinates(md_info.crd, md_info.uint_crd, md_info.pbc.uint_dr_to_dr_cof);

	if (md_info.mode == md_info.NVE)
	{
		md_info.nve.Leap_Frog();
	}
	else if (md_info.mode == md_info.MINIMIZATION)
	{
		md_info.min.Gradient_Descent();
	}
	else if (middle_langevin.is_initialized)
	{
		middle_langevin.MD_Iteration_Leap_Frog(md_info.frc, md_info.vel, md_info.acc, md_info.crd);
	}
	else if (bd_thermo.is_initialized)
	{
		bd_thermo.Record_Temperature(md_info.sys.Get_Atom_Temperature(), md_info.sys.freedom);
		md_info.nve.Leap_Frog();
		bd_thermo.Scale_Velocity(md_info.atom_numbers, md_info.vel);
	}
	else if (ad_thermo.is_initialized)
	{
		if ((md_info.sys.steps - 1) % ad_thermo.update_interval == 0)
		{
			ad_thermo.MD_Iteration_Leap_Frog(md_info.atom_numbers, md_info.vel, md_info.crd, md_info.frc, md_info.acc, md_info.d_mass_inverse, md_info.dt);
			simple_constrain.info.exp_gamma = 0;
			simple_constrain.half_exp_gamma_plus_half = 0.5;
			simple_constrain.settle.exp_gamma = 0;
			simple_constrain.settle.half_exp_gamma_plus_half = 0.5;
		}
		else
		{
			md_info.nve.Leap_Frog();
			simple_constrain.info.exp_gamma = 1;
			simple_constrain.half_exp_gamma_plus_half = 1;
			simple_constrain.settle.exp_gamma = 1;
			simple_constrain.settle.half_exp_gamma_plus_half = 1;
		}
	}
	else if (nhc.is_initialized)
	{
		nhc.MD_Iteration_Leap_Frog(md_info.atom_numbers, md_info.vel, md_info.crd, md_info.frc, md_info.acc, md_info.d_mass_inverse, md_info.dt, md_info.sys.Get_Total_Atom_Ek(), md_info.sys.freedom);
	}


	simple_constrain.Constrain(md_info.crd, md_info.vel, md_info.d_mass_inverse, md_info.d_mass, md_info.sys.box_length, md_info.need_pressure, md_info.sys.d_pressure);
	
	if (bd_baro.is_initialized && md_info.sys.steps % bd_baro.update_interval == 0)
	{
		hipMemcpy(&md_info.sys.h_pressure, md_info.sys.d_pressure, sizeof(float), hipMemcpyDeviceToHost);
		float p_now = md_info.sys.h_pressure;
		bd_baro.crd_scale_factor = 1 - bd_baro.update_interval * bd_baro.compressibility * bd_baro.dt / bd_baro.taup / 3 * (md_info.sys.target_pressure - p_now);

		Main_Volume_Change(bd_baro.crd_scale_factor);
		bd_baro.newV = md_info.sys.Get_Volume();
		if (bd_baro.newV > 1.331 * bd_baro.V0 || bd_baro.newV < 0.729 * bd_baro.V0)
		{
			Main_Volume_Change_Largely();
			bd_baro.V0 = bd_baro.newV;
		}
	}

	md_info.MD_Information_Crd_To_Uint_Crd();
	vatom.Coordinate_Refresh(md_info.uint_crd, md_info.pbc.uint_dr_to_dr_cof, md_info.crd);//ע�����uint crd
	neighbor_list.Neighbor_List_Update(md_info.crd, md_info.nb.d_excluded_list_start, md_info.nb.d_excluded_list, md_info.nb.d_excluded_numbers);
	mol_map.Refresh_BoxMapTimes(md_info.crd);
}

void Main_Print()
{
	if (md_info.sys.steps % md_info.output.write_trajectory_interval == 0)
	{
		controller.Step_Print("step", md_info.sys.steps);
		controller.Step_Print("time", md_info.sys.Get_Current_Time());
		controller.Step_Print("temperature", md_info.sys.Get_Atom_Temperature());
		controller.Step_Print("potential", md_info.sys.h_potential);
		controller.Step_Print("PME", pme.Get_Energy(md_info.uint_crd,md_info.d_charge, neighbor_list.d_nl, md_info.pbc.uint_dr_to_dr_cof, 
			md_info.nb.d_excluded_list_start, md_info.nb.d_excluded_list, md_info.nb.d_excluded_numbers));
		controller.Step_Print("LJ", lj.Get_Energy(md_info.uint_crd, neighbor_list.d_nl));
		controller.Step_Print("nb14_LJ", nb14.Get_14_LJ_Energy(lj.uint_crd_with_LJ, md_info.pbc.uint_dr_to_dr_cof, lj.d_LJ_A, lj.d_LJ_B));
		controller.Step_Print("nb14_EE", nb14.Get_14_CF_Energy(lj.uint_crd_with_LJ, md_info.pbc.uint_dr_to_dr_cof));
		controller.Step_Print("bond", bond.Get_Energy(md_info.uint_crd, md_info.pbc.uint_dr_to_dr_cof));
		controller.Step_Print("angle", angle.Get_Energy(md_info.uint_crd, md_info.pbc.uint_dr_to_dr_cof));
		controller.Step_Print("restrain", restrain.Get_Energy(md_info.crd, md_info.sys.box_length));
		controller.Step_Print("dihedral", dihedral.Get_Energy(md_info.uint_crd, md_info.pbc.uint_dr_to_dr_cof));
		controller.Step_Print("density", md_info.sys.Get_Density());
		controller.Step_Print("pressure", md_info.sys.h_pressure * CONSTANT_PRES_CONVERTION);

		controller.Print_To_Screen_And_Mdout();

		if (md_info.output.is_molecule_map_output)
		{
			mol_map.Calculate_No_Wrap_Crd(md_info.crd);
			md_info.mol.Molecule_Crd_Map(mol_map.nowrap_crd);
			mol_map.Refresh_BoxMapTimes(md_info.crd);
		}
		md_info.output.Append_Crd_Traj_File();
		md_info.output.Append_Box_Traj_File();
		// 20210827��������ٶȺ���
		if (md_info.output.is_vel_traj)
		{
			md_info.output.Append_Vel_Traj_File();
		}
		if (md_info.output.is_frc_traj)
		{
			md_info.output.Append_Frc_Traj_File();
		}
		nhc.Save_Trajectory_File();
	}
	if (md_info.sys.steps % md_info.output.write_restart_file_interval == 0)
	{
		md_info.output.Export_Restart_File();
		nhc.Save_Restart_File();
	}
}

void Main_Volume_Change(double factor)
{
	md_info.Update_Volume(factor);
	neighbor_list.Update_Volume(md_info.sys.box_length);
	neighbor_list.Neighbor_List_Update(md_info.crd, md_info.nb.d_excluded_list_start, md_info.nb.d_excluded_list, md_info.nb.d_excluded_numbers, neighbor_list.CONDITIONAL_UPDATE, neighbor_list.FORCED_CHECK);
	lj.Update_Volume(md_info.sys.box_length);
	pme.Update_Volume(md_info.sys.box_length);
	simple_constrain.Update_Volume(md_info.sys.box_length);
	mol_map.Update_Volume(md_info.sys.box_length);
}

void Main_Volume_Change_Largely()
{
	controller.printf("Some modules are based on the meshing methods, and it is more precise to re-initialize these modules now for a long time or a large volume change.\n");
	neighbor_list.Clear();
	pme.Clear();
	neighbor_list.Initial(&controller, md_info.atom_numbers, md_info.sys.box_length, md_info.nb.cutoff, md_info.nb.skin);
	neighbor_list.Neighbor_List_Update(md_info.crd, md_info.nb.d_excluded_list_start, md_info.nb.d_excluded_list, md_info.nb.d_excluded_numbers, 1);
	pme.Initial(&controller, md_info.atom_numbers, md_info.sys.box_length ,md_info.nb.cutoff );
	controller.printf("---------------------------------------------------------------------------------------\n"); 
}

