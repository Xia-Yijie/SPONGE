#include "hip/hip_runtime.h"
﻿#include "TI_core.cuh"

#define PI 3.1415926

#define TRAJ_COMMAND "crd"
#define TRAJ_DEFAULT_FILENAME "mdcrd.dat"
#define BOX_COMMAND "box"
#define BOX_DEFAULT_FILENAME "box.txt"
#define TI_RESULT_COMMAND "TI"
#define TI_RESULT_DEFUALT_FILENAME "TI.txt"


static __global__ void device_add(float * ene, float factor, float * charge_sum1, float * charge_sum2)
{
	ene[0] += factor * charge_sum1[0] * charge_sum2[0];
}

static __global__ void PME_Cross_Direct_Energy(
	const int atom_numbers, const ATOM_GROUP *nl,
	const UNSIGNED_INT_VECTOR *uint_crd, const VECTOR boxlength, const float *charge, const float *charge2,
	const float beta, const float cutoff_square, float *direct_ene)
{
	int atom_i = blockDim.x*blockIdx.x + threadIdx.x;
	if (atom_i < atom_numbers)
	{
		ATOM_GROUP nl_i = nl[atom_i];
		int N = nl_i.atom_numbers;
		int atom_j;
		int int_x;
		int int_y;
		int int_z;
		UNSIGNED_INT_VECTOR r1 = uint_crd[atom_i], r2;
		VECTOR dr;
		float dr2;
		float dr_abs;
		float ene_temp;
		float charge_i = charge[atom_i], charge_i2 = charge2[atom_i];
		float ene_lin = 0.;

		for (int j = threadIdx.y; j < N; j = j + blockDim.y)
		{

			atom_j = nl_i.atom_serial[j];
			r2 = uint_crd[atom_j];

			int_x = r2.uint_x - r1.uint_x;
			int_y = r2.uint_y - r1.uint_y;
			int_z = r2.uint_z - r1.uint_z;
			dr.x = boxlength.x*int_x;
			dr.y = boxlength.y*int_y;
			dr.z = boxlength.z*int_z;

			dr2 = dr.x*dr.x + dr.y*dr.y + dr.z*dr.z;
			if (dr2 < cutoff_square)
			{

				dr_abs = norm3df(dr.x, dr.y, dr.z);
				ene_temp = (charge_i * charge2[atom_j] + charge_i2 * charge[atom_j]) * erfcf(beta * dr_abs) / dr_abs;
				ene_lin = ene_lin + ene_temp;
				//printf("ene_temp: %f, dr_abs = %f, r1.uint_x, uy, yz: %u %u %u\n", ene_temp, dr_abs, r1.uint_x, r1.uint_y, r1.uint_z);
			}

		}//atom_j cycle
		atomicAdd(direct_ene, ene_lin);
	}
}


static __global__ void PME_Cross_Excluded_Energy_Correction
(const int atom_numbers, const UNSIGNED_INT_VECTOR *uint_crd, const VECTOR sacler,
const float *charge, const float * charge2, const float pme_beta, const float sqrt_pi,
const int *excluded_list_start, const int *excluded_list, const int *excluded_atom_numbers,
float *ene)
{
	int atom_i = blockDim.x*blockIdx.x + threadIdx.x;
	if (atom_i < atom_numbers)
	{
		int excluded_number = excluded_atom_numbers[atom_i];
		if (excluded_number > 0)
		{
			int list_start = excluded_list_start[atom_i];
			int list_end = list_start + excluded_number;
			int atom_j;
			int int_x;
			int int_y;
			int int_z;

			float charge_i = charge[atom_i];
			float charge_i2 = charge2[atom_i];
			float charge_j, charge_j2;
			float dr_abs;
			float beta_dr;

			UNSIGNED_INT_VECTOR r1 = uint_crd[atom_i], r2;
			VECTOR dr;
			float dr2;

			float ene_lin = 0.;

			for (int i = list_start; i < list_end; i = i + 1)
			{
				atom_j = excluded_list[i];
				r2 = uint_crd[atom_j];
				charge_j = charge2[atom_j];
				charge_j2 = charge[atom_j];

				int_x = r2.uint_x - r1.uint_x;
				int_y = r2.uint_y - r1.uint_y;
				int_z = r2.uint_z - r1.uint_z;
				dr.x = sacler.x*int_x;
				dr.y = sacler.y*int_y;
				dr.z = sacler.z*int_z;
				dr2 = dr.x*dr.x + dr.y*dr.y + dr.z*dr.z;
				//假设剔除表中的原子对距离总是小于cutoff的，正常体系
				dr_abs = sqrtf(dr2);
				beta_dr = pme_beta*dr_abs;

				ene_lin -= (charge_i * charge_j + charge_i2 * charge_j2)* erff(beta_dr) / dr_abs;

			}//atom_j cycle
			atomicAdd(ene + atom_i, ene_lin);
		}//if need excluded
	}
}


void TI_CORE::non_bond_information::Initial(CONTROLLER *controller, TI_CORE *TI_core )
{
	if (controller[0].Command_Exist("skin"))
	{
		skin = atof(controller[0].Command("skin"));
	}
	else
	{
		skin = 2.0;
	}
	controller->printf("    skin set to %.2f Angstram\n", skin);

	if (controller[0].Command_Exist("cutoff"))
	{
		cutoff = atof(controller[0].Command("cutoff"));
	}
	else
	{
		cutoff = 10.0;
	}
	controller->printf("    cutoff set to %.2f Angstram\n", cutoff);
	/*===========================
	读取排除表相关信息
	============================*/
	if (controller[0].Command_Exist("exclude_in_file"))
	{
		FILE *fp = NULL;
		controller->printf("    Start reading excluded list:\n");
		Open_File_Safely(&fp, controller[0].Command("exclude_in_file"), "r");
		
		int atom_numbers = 0;
		int toscan = fscanf(fp, "%d %d", &atom_numbers, &excluded_atom_numbers);
		if (TI_core->atom_numbers > 0 && TI_core->atom_numbers != atom_numbers)
		{
			controller->printf("        Error: atom_numbers is not equal: %d %d\n", TI_core->atom_numbers, atom_numbers);
			getchar();
			exit(1);
		}
		else if (TI_core->atom_numbers == 0)
		{
			TI_core->atom_numbers = atom_numbers;
		}
		controller->printf("        excluded list total length is %d\n", excluded_atom_numbers);

		Cuda_Malloc_Safely((void**)&d_excluded_list_start, sizeof(int)*atom_numbers);
		Cuda_Malloc_Safely((void**)&d_excluded_numbers, sizeof(int)*atom_numbers);
		Cuda_Malloc_Safely((void**)&d_excluded_list, sizeof(int)*excluded_atom_numbers);

		Malloc_Safely((void**)&h_excluded_list_start, sizeof(int)*atom_numbers);
		Malloc_Safely((void**)&h_excluded_numbers, sizeof(int)*atom_numbers);
		Malloc_Safely((void**)&h_excluded_list, sizeof(int)*excluded_atom_numbers);
		int count = 0;
		for (int i = 0; i < atom_numbers; i++)
		{
			toscan = fscanf(fp, "%d", &h_excluded_numbers[i]);
			h_excluded_list_start[i] = count;
			for (int j = 0; j < h_excluded_numbers[i]; j++)
			{
				toscan = fscanf(fp, "%d", &h_excluded_list[count]);
				count++;
			}
		}
		hipMemcpy(d_excluded_list_start, h_excluded_list_start, sizeof(int)*atom_numbers, hipMemcpyHostToDevice);
		hipMemcpy(d_excluded_numbers, h_excluded_numbers, sizeof(int)*atom_numbers, hipMemcpyHostToDevice);
		hipMemcpy(d_excluded_list, h_excluded_list, sizeof(int)*excluded_atom_numbers, hipMemcpyHostToDevice);
		controller->printf("    End reading excluded list\n\n");
		fclose(fp);
	}
	else if (controller[0].Command_Exist("amber_parm7"))
	{
		/*===========================
		从parm中读取排除表相关信息
		============================*/
		FILE *parm = NULL;
		Open_File_Safely(&parm, controller[0].Command("amber_parm7"), "r");
		controller->printf("    Start reading excluded list from AMBER file:\n");
		while (true)
		{
			char temps[CHAR_LENGTH_MAX];
			char temp_first_str[CHAR_LENGTH_MAX];
			char temp_second_str[CHAR_LENGTH_MAX];
			if (!fgets(temps, CHAR_LENGTH_MAX, parm))
			{
				break;
			}
			if (sscanf(temps, "%s %s", temp_first_str, temp_second_str) != 2)
			{
				continue;
			}
			if (strcmp(temp_first_str, "%FLAG") == 0
				&& strcmp(temp_second_str, "POINTERS") == 0)
			{
				char *toget = fgets(temps, CHAR_LENGTH_MAX, parm);

				int atom_numbers = 0;
				int toscan = fscanf(parm, "%d\n", &atom_numbers);
				if (TI_core->atom_numbers > 0 && TI_core->atom_numbers != atom_numbers)
				{
					controller->printf("        Error: atom_numbers is not equal: %d %d\n", TI_core->atom_numbers, atom_numbers);
					getchar();
					exit(1);
				}
				else if (TI_core->atom_numbers == 0)
				{
					TI_core->atom_numbers = atom_numbers;
				}
				Cuda_Malloc_Safely((void**)&d_excluded_list_start, sizeof(int)*atom_numbers);
				Cuda_Malloc_Safely((void**)&d_excluded_numbers, sizeof(int)*atom_numbers);

				Malloc_Safely((void**)&h_excluded_list_start, sizeof(int)*atom_numbers);
				Malloc_Safely((void**)&h_excluded_numbers, sizeof(int)*atom_numbers);
				for (int i = 0; i < 9; i = i + 1)
				{
					toscan = fscanf(parm, "%d\n", &excluded_atom_numbers);
				}
				toscan = fscanf(parm, "%d\n", &excluded_atom_numbers);
				controller->printf("        excluded list total length is %d\n", excluded_atom_numbers);

				Cuda_Malloc_Safely((void**)&d_excluded_list, sizeof(int)*excluded_atom_numbers);
				Malloc_Safely((void**)&h_excluded_list, sizeof(int)*excluded_atom_numbers);
			}

			//read atom_excluded_number for every atom
			if (strcmp(temp_first_str, "%FLAG") == 0
				&& strcmp(temp_second_str, "NUMBER_EXCLUDED_ATOMS") == 0)
			{
				char *toget = fgets(temps, CHAR_LENGTH_MAX, parm);
				for (int i = 0; i<TI_core->atom_numbers; i = i + 1)
				{
					int toscan = fscanf(parm, "%d\n", &h_excluded_numbers[i]);
				}
			}
			//read every atom's excluded atom list
			if (strcmp(temp_first_str, "%FLAG") == 0
				&& strcmp(temp_second_str, "EXCLUDED_ATOMS_LIST") == 0)
			{
				int count = 0;
				//int none_count = 0;
				int lin = 0;
				char *toget = fgets(temps, CHAR_LENGTH_MAX, parm);
				for (int i = 0; i<TI_core->atom_numbers; i = i + 1)
				{
					h_excluded_list_start[i] = count;
					for (int j = 0; j<h_excluded_numbers[i]; j = j + 1)
					{
						int toscan = fscanf(parm, "%d\n", &lin);
						if (lin == 0)
						{
							h_excluded_numbers[i] = 0;
							break;
						}
						else
						{
							h_excluded_list[count] = lin - 1;
							count = count + 1;
						}
					}
					if (h_excluded_numbers[i] > 0)
						thrust::sort(&h_excluded_list[h_excluded_list_start[i]], &h_excluded_list[h_excluded_list_start[i]] + h_excluded_numbers[i]);
				}
			}
		}

		hipMemcpy(d_excluded_list_start, h_excluded_list_start, sizeof(int)*TI_core->atom_numbers, hipMemcpyHostToDevice);
		hipMemcpy(d_excluded_numbers, h_excluded_numbers, sizeof(int)*TI_core->atom_numbers, hipMemcpyHostToDevice);
		hipMemcpy(d_excluded_list, h_excluded_list, sizeof(int)*excluded_atom_numbers, hipMemcpyHostToDevice);
		controller->printf("    End reading excluded list from AMBER file\n\n");
		fclose(parm);
	}
	else
	{
		int atom_numbers = TI_core->atom_numbers;
		excluded_atom_numbers = 0;
		controller->printf("    Set all atom exclude no atoms as default\n"); 

		Cuda_Malloc_Safely((void**)&d_excluded_list_start, sizeof(int)*atom_numbers);
		Cuda_Malloc_Safely((void**)&d_excluded_numbers, sizeof(int)*atom_numbers);
		Cuda_Malloc_Safely((void**)&d_excluded_list, sizeof(int)*excluded_atom_numbers);

		Malloc_Safely((void**)&h_excluded_list_start, sizeof(int)*atom_numbers);
		Malloc_Safely((void**)&h_excluded_numbers, sizeof(int)*atom_numbers);
		Malloc_Safely((void**)&h_excluded_list, sizeof(int)*excluded_atom_numbers);


		int count = 0;
		for (int i = 0; i < atom_numbers; i++)
		{
			h_excluded_numbers[i] = 0;
			h_excluded_list_start[i] = count;
			for (int j = 0; j < h_excluded_numbers[i]; j++)
			{
				h_excluded_list[count] = 0;
				count++;
			}
		}
		hipMemcpy(d_excluded_list_start, h_excluded_list_start, sizeof(int)*atom_numbers, hipMemcpyHostToDevice);
		hipMemcpy(d_excluded_numbers, h_excluded_numbers, sizeof(int)*atom_numbers, hipMemcpyHostToDevice);
		hipMemcpy(d_excluded_list, h_excluded_list, sizeof(int)*excluded_atom_numbers, hipMemcpyHostToDevice);
	}
}

void TI_CORE::periodic_box_condition_information::Update_Volume(VECTOR box_length)
{
	crd_to_uint_crd_cof = CONSTANT_UINT_MAX_FLOAT / box_length;
	quarter_crd_to_uint_crd_cof = 0.25 * crd_to_uint_crd_cof;
	uint_dr_to_dr_cof = 1.0f / crd_to_uint_crd_cof;
}

void TI_CORE::trajectory_input::Initial(CONTROLLER *controller, TI_CORE * TI_core)
{
	this->TI_core = TI_core;
	if (controller[0].Command_Exist("frame_numbers"))
	{
		frame_numbers = atoi(controller[0].Command("frame_numbers"));
	}
	else
	{
		printf("	warning: missing value of frame numbers, set to default 1000.\n");
		frame_numbers = 1000;
	}
	current_frame = 0;
	bytes_per_frame = TI_core->atom_numbers * 3 * sizeof(float);
	if (controller[0].Command_Exist(TRAJ_COMMAND))
	{
		Open_File_Safely(&crd_traj, controller[0].Command(TRAJ_COMMAND), "rb");
	}
	else
	{
		printf("	Error: missing trajectory file.\n");
		getchar();
		exit(1);
	}
	if (controller[0].Command_Exist(BOX_COMMAND))
	{
		Open_File_Safely(&box_traj, controller[0].Command(BOX_COMMAND), "r");
	}
	else
	{
		printf("	Error: missing box trajectory file.\n");
		getchar();
		exit(1);
	}
}

void TI_CORE::Initial(CONTROLLER *controller)
{
	controller[0].printf("START INITIALZING TI CORE:\n");
	
	if (controller[0].Command_Exist("atom_numbers"))
	{
		atom_numbers = atoi(controller[0].Command("atom_numbers"));
	}
	else
	{
		printf("	Error: missing value of atom numbers.\n");
		getchar();
		exit(1);
	}

	box_length.x = box_length.y = box_length.z = 1.0;
	last_box_length.x = last_box_length.y = last_box_length.z = 1.0;
	volume_change_factor = 0.0;
	box_angle.x = box_angle.y = box_angle.z = 0.0;
	if (controller[0].Command_Exist("charge_pertubated"))
	{
		charge_pertubated = atoi(controller[0].Command("charge_pertubated"));
	}
	else
	{
		printf("	Warning: missing value of charge pertubated, set to default 0.\n");
		charge_pertubated = 0;
	}

	Malloc_Safely((void**)&h_charge, sizeof(float) * atom_numbers);
	Malloc_Safely((void**)&h_charge_A, sizeof(float) * atom_numbers);
	Malloc_Safely((void**)&h_charge_B, sizeof(float) * atom_numbers);
	Malloc_Safely((void**)&h_charge_B_A, sizeof(float) * atom_numbers);
	Malloc_Safely((void**)&h_subsys_division, sizeof(int) * atom_numbers);
	Cuda_Malloc_Safely((void**)&d_charge, sizeof(float) * atom_numbers);
	Cuda_Malloc_Safely((void**)&d_charge_B_A, sizeof(float) * atom_numbers);
	Cuda_Malloc_Safely((void**)&d_subsys_division, sizeof(int) * atom_numbers);

	if (controller->Command_Exist(TI_RESULT_COMMAND))
	{
		Open_File_Safely(&ti_result, controller->Command(TI_RESULT_COMMAND), "wb");
	}
	else
	{
		Open_File_Safely(&ti_result, TI_RESULT_DEFUALT_FILENAME, "wb");
	}
	
	
	if (charge_pertubated > 0)
	{
		if (controller[0].Command_Exist("chargeA_in_file") && controller[0].Command_Exist("chargeB_in_file"))
		{
			controller[0].printf("    Start reading chargeA:\n");
			int atom_numbers_in_file = 0;
			FILE * fp = NULL;
			Open_File_Safely(&fp, controller[0].Command("chargeA_in_file"), "r");
			char lin[CHAR_LENGTH_MAX];
			char *toget = fgets(lin, CHAR_LENGTH_MAX, fp);
			int scanf_ret = sscanf(lin, "%d", &atom_numbers_in_file);
			if (atom_numbers != atom_numbers_in_file)
			{
				controller->printf("        Error: atom_numbers is not equal: %d %d\n", atom_numbers, atom_numbers_in_file);
				getchar();
				exit(1);
			}
			for (int i = 0; i < atom_numbers; ++i)
			{
				scanf_ret = fscanf(fp, "%f", &h_charge_A[i]);
			}
			fclose(fp);
			controller[0].printf("    End reading chargeA\n\n");

			controller[0].printf("    Start reading chargeB:\n");
			Open_File_Safely(&fp, controller[0].Command("chargeB_in_file"), "r");
			toget = fgets(lin, CHAR_LENGTH_MAX, fp);
			scanf_ret = sscanf(lin, "%d", &atom_numbers_in_file);
			if (atom_numbers != atom_numbers_in_file)
			{
				controller->printf("        Error: atom_numbers is not equal: %d %d\n", atom_numbers, atom_numbers_in_file);
				getchar();
				exit(1);
			}
			for (int i = 0; i < atom_numbers; ++i)
			{
				scanf_ret = fscanf(fp, "%f", &h_charge_B[i]);
			}
			fclose(fp);
			controller[0].printf("    End reading chargeB\n\n");
			for (int i = 0; i < atom_numbers; ++i)
			{
				h_charge_B_A[i] = h_charge_B[i] - h_charge_A[i];
			}
			hipMemcpy(d_charge_B_A, h_charge_B_A, sizeof(float)*atom_numbers, hipMemcpyHostToDevice);
		}
		else
		{
			printf("	Error: missing value of charge A and charge B, These value must be given in TI mode if charge is pertubated.\n");
			getchar();
			exit(1);
		}
	}

	if (controller[0].Command_Exist("charge_in_file"))
	{
		FILE *fp = NULL;
		controller->printf("    Start reading charge:\n");
		Open_File_Safely(&fp, controller[0].Command("charge_in_file"), "r");
		int atom_numbers = 0;
		char lin[CHAR_LENGTH_MAX];
		char *toget = fgets(lin, CHAR_LENGTH_MAX, fp);
		int scanf_ret = sscanf(lin, "%d", &atom_numbers);
		if (this->atom_numbers > 0 && this->atom_numbers != atom_numbers)
		{
			controller->printf("        Error: atom_numbers is not equal: %d %d\n", this->atom_numbers, atom_numbers);
			getchar();
			exit(1);
		}
		else if (this->atom_numbers == 0)
		{
			this->atom_numbers = atom_numbers;
		}
		for (int i = 0; i < atom_numbers; i++)
		{
			scanf_ret = fscanf(fp, "%f", &h_charge[i]);
		}
		controller->printf("    End reading charge\n\n");
		fclose(fp);
	}
	else if (atom_numbers > 0)
	{
		controller[0].printf("    charge is set to 0 as default\n");
		for (int i = 0; i < atom_numbers; i++)
		{
			h_charge[i] = 0;
		}
	}
	hipMemcpy(d_charge, h_charge, sizeof(float)*atom_numbers, hipMemcpyHostToDevice);


	if (controller[0].Command_Exist("subsys_division_in_file"))
	{
		FILE *fp = NULL;
		controller->printf("    Start reading subsystem division information:\n");
		Open_File_Safely(&fp, controller[0].Command("subsys_division_in_file"), "r");
		int atom_numbers = 0;
		char lin[CHAR_LENGTH_MAX];
		char *toget = fgets(lin, CHAR_LENGTH_MAX, fp);
		int scanf_ret = sscanf(lin, "%d", &atom_numbers);
		if (this->atom_numbers > 0 && this->atom_numbers != atom_numbers)
		{
			controller->printf("        Error: atom_numbers is not equal: %d %d\n", this->atom_numbers, atom_numbers);
			getchar();
			exit(1);
		}
		else if (this->atom_numbers == 0)
		{
			this->atom_numbers = atom_numbers;
		}
		for (int i = 0; i < atom_numbers; i++)
		{
			scanf_ret = fscanf(fp, "%d", &h_subsys_division[i]);
		}
		controller->printf("    End reading subsystem information\n\n");
		fclose(fp);
	}
	else if (atom_numbers > 0)
	{
		controller[0].printf("    subsystem mask is set to 0 as default\n");
		for (int i = 0; i < atom_numbers; i++)
		{
			h_subsys_division[i] = 0;
		}
	}
	hipMemcpy(d_subsys_division, h_subsys_division, sizeof(float)*atom_numbers, hipMemcpyHostToDevice);

	//Malloc_Safely((void**)&velocity, sizeof(VECTOR) * atom_numbers);
	Malloc_Safely((void**)&coordinate, sizeof(VECTOR) * atom_numbers);
	//Cuda_Malloc_Safely((void**)&vel, sizeof(VECTOR) * atom_numbers);
	Cuda_Malloc_Safely((void**)&crd, sizeof(VECTOR) * atom_numbers);
	Cuda_Malloc_Safely((void**)&uint_crd, sizeof(UNSIGNED_INT_VECTOR)*atom_numbers);

	nb.Initial(controller, this);
	input.Initial(controller, this);

	controller[0].Step_Print_Initial("frame", "%d");
	controller[0].Step_Print_Initial("dH_dlambda", "%.2f");
	Read_Next_Frame();

	printf("END INITIALZING TI CORE\n\n");
}

void TI_CORE::TI_Core_Crd_To_Uint_Crd()
{
	Crd_To_Uint_Crd << <ceilf((float)this->atom_numbers / 128), 128 >> >
		(this->atom_numbers, pbc.quarter_crd_to_uint_crd_cof, crd, uint_crd);
}

void TI_CORE::Read_Next_Frame()
{
	size_t toread = fread(coordinate, sizeof(VECTOR), atom_numbers, input.crd_traj);
	hipMemcpy(crd, coordinate, sizeof(VECTOR) *atom_numbers, hipMemcpyHostToDevice);
	last_box_length.x = box_length.x;
	last_box_length.y = box_length.y;
	last_box_length.z = box_length.z;
	int toscan = fscanf(input.box_traj, "%f %f %f %f %f %f", &box_length.x, &box_length.y, &box_length.z, &box_angle.x, &box_angle.y, &box_angle.z);
	volume_change_factor = box_length.x / last_box_length.x;
	/*if (mass_pertubated != 0)
	{
		fread(velocity, sizeof(VECTOR), atom_numbers, input.vel_traj);
		hipMemcpy(vel, velocity, sizeof(VECTOR) * atom_numbers, hipMemcpyHostToDevice);
	}*/
	pbc.Update_Volume(box_length);
	TI_Core_Crd_To_Uint_Crd();
}


void TI_CORE::Clear()
{
	free(coordinate);
	hipFree(crd);
	hipFree(uint_crd);

	free(h_charge_A);
	free(h_charge_B);
	free(h_charge);
	free(h_charge_B_A);
	hipFree(d_charge);
	hipFree(d_charge_B_A);
	free(h_subsys_division);
	hipFree(d_subsys_division);
	

	coordinate = NULL;
	crd = NULL;
	uint_crd = NULL;
	h_charge_A = NULL;
	h_charge_B = NULL;
	h_charge = NULL;
	h_charge_B_A = NULL;
	d_charge = NULL;
	d_charge_B_A = NULL;
	h_subsys_division = NULL;
	d_subsys_division = NULL;

	free(nb.h_excluded_list);
	free(nb.h_excluded_numbers);
	free(nb.h_excluded_list_start);
	hipFree(nb.d_excluded_list);
	hipFree(nb.d_excluded_numbers);
	hipFree(nb.d_excluded_list_start);
	nb.h_excluded_list = NULL;
	nb.h_excluded_numbers = NULL;
	nb.h_excluded_list_start = NULL;
	nb.d_excluded_list_start = NULL;
	nb.d_excluded_numbers = NULL;
	nb.d_excluded_list = NULL;

	fclose(input.crd_traj);

	fclose(input.box_traj);
}

void TI_CORE::dH_dlambda_data::Sum_One_Frame()
{
	dH_dlambda_current_frame = bond_soft_dH_dlambda + lj_soft_dH_dlambda + coul_direct_dH_dlambda +lj_soft_long_range_correction + pme_dH_dlambda /*+ kinetic_dH_dlambda*/ + (bondB_ene - bondA_ene) + (angleB_ene - angleA_ene) + (dihedralB_ene - dihedralA_ene) + (nb14B_EE_ene - nb14A_EE_ene) + (nb14B_LJ_ene - nb14A_LJ_ene);
	
	total_dH_dlambda += dH_dlambda_current_frame;
}

void TI_CORE::TI_Core_Crd_Device_To_Host()
{
	hipMemcpy(coordinate, crd, sizeof(VECTOR) * atom_numbers, hipMemcpyDeviceToHost);
}

void TI_CORE::Print_dH_dlambda_Average_To_Screen_And_Result_File()
{
	data.average_dH_dlambda = data.total_dH_dlambda/input.frame_numbers;
	fprintf(stdout, "Ensemble Average <dH/dlambda>: %.6f\n", data.average_dH_dlambda);
	fprintf(ti_result, "%.6f\n", data.average_dH_dlambda);
}

void TI_CORE::cross_pme::Initial(const int atom_numbers, const int PME_Nall)
{
	Cuda_Malloc_Safely((void**)&PME_Q_B_A, sizeof(float) * PME_Nall);
	Cuda_Malloc_Safely((void**)&d_cross_reciprocal_ene, sizeof(float));
	Cuda_Malloc_Safely((void**)&d_cross_self_ene, sizeof(float));
	Cuda_Malloc_Safely((void**)&charge_sum_B_A, sizeof(float));
	Cuda_Malloc_Safely((void**)&d_cross_correction_atom_energy, sizeof(float) * atom_numbers);
	Cuda_Malloc_Safely((void**)&d_cross_correction_ene, sizeof(float));
	Cuda_Malloc_Safely((void**)&d_cross_direct_ene, sizeof(float));
}

float TI_CORE::Get_Cross_PME_Partial_H_Partial_Lambda(Particle_Mesh_Ewald * pme, const ATOM_GROUP
	* nl, int lj_pertubated, int is_download)
{
   if (charge_pertubated)
   {
	   PME_Atom_Near << <atom_numbers / 32 + 1, 32 >> >
	   (uint_crd, pme->PME_atom_near, pme->PME_Nin,
	   CONSTANT_UINT_MAX_INVERSED * pme->fftx, CONSTANT_UINT_MAX_INVERSED * pme->ffty, CONSTANT_UINT_MAX_INVERSED * pme->fftz,
	   atom_numbers, pme->fftx, pme->ffty, pme->fftz,
	   pme->PME_kxyz, pme->PME_uxyz, pme->PME_frxyz);

	   Reset_List << < pme->PME_Nall / 1024 + 1, 1024 >> >(pme->PME_Nall, pme->PME_Q, 0);
	   Reset_List << < pme->PME_Nall / 1024 + 1, 1024 >> >(pme->PME_Nall, cross_pme.PME_Q_B_A, 0);

	   PME_Q_Spread << < atom_numbers / pme->thread_PME.x + 1, pme->thread_PME >> >
			   (pme->PME_atom_near, d_charge, pme->PME_frxyz,
				pme->PME_Q, pme->PME_kxyz, atom_numbers);

	   PME_Q_Spread << < atom_numbers / pme->thread_PME.x + 1, pme->thread_PME >> >
			   (pme->PME_atom_near, d_charge_B_A, pme->PME_frxyz, cross_pme.PME_Q_B_A, pme->PME_kxyz, atom_numbers);
	   
	   hipfftExecR2C(pme->PME_plan_r2c, (float*)pme->PME_Q, (hipfftComplex*)pme->PME_FQ);

	   PME_BCFQ << < pme->PME_Nfft / 1024 + 1, 1024 >> > (pme->PME_FQ, pme->PME_BC, pme->PME_Nfft);

	   hipfftExecC2R(pme->PME_plan_c2r, (hipfftComplex*)pme->PME_FQ, (float*)pme->PME_FBCFQ);

	   PME_Energy_Product << < 1, 1024 >> >(pme->PME_Nall, cross_pme.PME_Q_B_A, pme->PME_FBCFQ, cross_pme.d_cross_reciprocal_ene);

	   PME_Energy_Product << < 1, 1024 >> >(atom_numbers, d_charge, d_charge_B_A, cross_pme.d_cross_self_ene);

	   Scale_List << <1, 1 >> >(1, cross_pme.d_cross_self_ene, -2 * pme->beta / sqrtf(PI));

	   Sum_Of_List << <1, 1024 >> >(atom_numbers, d_charge, pme->charge_sum);
	   device_add << <1, 1 >> >(cross_pme.d_cross_self_ene, pme->neutralizing_factor, pme->charge_sum, cross_pme.charge_sum_B_A);
	   
	   Reset_List << <ceilf((float)atom_numbers / 1024.0f), 1024 >> >(atom_numbers, cross_pme.d_cross_correction_atom_energy, 0.0f);
	   PME_Cross_Excluded_Energy_Correction << < atom_numbers / 32 + 1, 32 >> >
		   (atom_numbers, uint_crd, pbc.uint_dr_to_dr_cof,
		   d_charge, d_charge_B_A, pme->beta, sqrtf(PI), nb.d_excluded_list_start, nb.d_excluded_list, nb.d_excluded_numbers, cross_pme.d_cross_correction_atom_energy);
	   Sum_Of_List << <1, 1024 >> >(atom_numbers, cross_pme.d_cross_correction_atom_energy, cross_pme.d_cross_correction_ene);

	   hipMemset(cross_pme.d_cross_direct_ene, 0, sizeof(float));
	   if (!lj_pertubated)
	   {
		   PME_Cross_Direct_Energy << < atom_numbers / pme->thread_PME.x + 1,  pme->thread_PME >> >
		   (atom_numbers, nl,
		   uint_crd, pbc.uint_dr_to_dr_cof, d_charge, d_charge_B_A, 
		   pme->beta, nb.cutoff*nb.cutoff, cross_pme.d_cross_direct_ene);
	   }

	   if (is_download)
	   {
		   hipMemcpy(&cross_pme.cross_reciprocal_ene, cross_pme.d_cross_reciprocal_ene, sizeof(float), hipMemcpyDeviceToHost);
		   hipMemcpy(&cross_pme.cross_self_ene, cross_pme.d_cross_self_ene, sizeof(float), hipMemcpyDeviceToHost);
		   hipMemcpy(&cross_pme.cross_correction_ene, cross_pme.d_cross_correction_ene, sizeof(float), hipMemcpyDeviceToHost);
		   hipMemcpy(&cross_pme.cross_direct_ene, cross_pme.d_cross_direct_ene, sizeof(float), hipMemcpyDeviceToHost);
		   cross_pme.dH_dlambda = cross_pme.cross_reciprocal_ene + cross_pme.cross_self_ene + cross_pme.cross_correction_ene;
		   return cross_pme.dH_dlambda;
	   }
	   else
	   {
		   return 0.0;
	   }
   }
   else
   {
	   return NAN;
   }
}


