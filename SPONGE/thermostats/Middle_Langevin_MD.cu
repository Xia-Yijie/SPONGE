#include "hip/hip_runtime.h"
﻿#include "Middle_Langevin_MD.cuh"

static __global__ void MD_Iteration_Leap_Frog_With_LiuJian
(const int atom_numbers, const float half_dt, const float dt,const float exp_gamma, 
const float *inverse_mass, const float *sqrt_mass_inverse, 
VECTOR *vel, VECTOR *crd, VECTOR *frc, VECTOR *acc, VECTOR *random_frc)
{
	int i = blockDim.x*blockIdx.x + threadIdx.x;
	if (i < atom_numbers)
	{
		acc[i].x = inverse_mass[i] * frc[i].x;
		acc[i].y = inverse_mass[i] * frc[i].y;
		acc[i].z = inverse_mass[i] * frc[i].z;

		vel[i].x = vel[i].x + dt*acc[i].x;
		vel[i].y = vel[i].y + dt*acc[i].y;
		vel[i].z = vel[i].z + dt*acc[i].z;

		
		crd[i].x = crd[i].x + half_dt*vel[i].x;
		crd[i].y = crd[i].y + half_dt*vel[i].y;
		crd[i].z = crd[i].z + half_dt*vel[i].z;


		vel[i].x = exp_gamma*vel[i].x + sqrt_mass_inverse[i] * random_frc[i].x;
		vel[i].y = exp_gamma*vel[i].y + sqrt_mass_inverse[i] * random_frc[i].y;
		vel[i].z = exp_gamma*vel[i].z + sqrt_mass_inverse[i] * random_frc[i].z;

		crd[i].x = crd[i].x + half_dt*vel[i].x;
		crd[i].y = crd[i].y + half_dt*vel[i].y;
		crd[i].z = crd[i].z + half_dt*vel[i].z;


	}
}
static __global__ void MD_Iteration_Leap_Frog_With_LiuJian_With_Max_Velocity
(const int atom_numbers, const float half_dt, const float dt, const float exp_gamma,
const float *inverse_mass, const float *sqrt_mass_inverse,
VECTOR *vel, VECTOR *crd, VECTOR *frc, VECTOR *acc, VECTOR *random_frc,const float max_vel)
{
	int i = blockDim.x*blockIdx.x + threadIdx.x;
	float abs_vel;
	if (i < atom_numbers)
	{
		acc[i].x = inverse_mass[i] * frc[i].x;
		acc[i].y = inverse_mass[i] * frc[i].y;
		acc[i].z = inverse_mass[i] * frc[i].z;

		vel[i].x = vel[i].x + dt*acc[i].x;
		vel[i].y = vel[i].y + dt*acc[i].y;
		vel[i].z = vel[i].z + dt*acc[i].z;


		abs_vel = fminf(1.0, max_vel * rnorm3df(vel[i].x, vel[i].y, vel[i].z));
		vel[i].x = abs_vel* vel[i].x;
		vel[i].y = abs_vel* vel[i].y;
		vel[i].z = abs_vel* vel[i].z;
	

		crd[i].x = crd[i].x + half_dt*vel[i].x;
		crd[i].y = crd[i].y + half_dt*vel[i].y;
		crd[i].z = crd[i].z + half_dt*vel[i].z;


		vel[i].x = exp_gamma*vel[i].x + sqrt_mass_inverse[i] * random_frc[i].x;
		vel[i].y = exp_gamma*vel[i].y + sqrt_mass_inverse[i] * random_frc[i].y;
		vel[i].z = exp_gamma*vel[i].z + sqrt_mass_inverse[i] * random_frc[i].z;

		crd[i].x = crd[i].x + half_dt*vel[i].x;
		crd[i].y = crd[i].y + half_dt*vel[i].y;
		crd[i].z = crd[i].z + half_dt*vel[i].z;


	}
}


void MIDDLE_Langevin_INFORMATION::Initial(CONTROLLER *controller, const int atom_numbers, const float target_temperature, const float *h_mass, const char *module_name)
{
	controller[0].printf("START INITIALIZING MIDDLE LANGEVIN DYNAMICS:\n");
	if (module_name == NULL)
	{
		strcpy(this->module_name, "middle_langevin");
	}
	else
	{
		strcpy(this->module_name, module_name);
	}

	float *h_mass_temp=NULL;
	this->atom_numbers = atom_numbers;
	this->target_temperature = target_temperature;
	controller[0].printf("    atom_numbers is %d\n", atom_numbers);
	Malloc_Safely((void**)&h_mass_temp, sizeof(float)*atom_numbers);
	hipMemcpy(h_mass_temp, h_mass, sizeof(float)*atom_numbers, hipMemcpyHostToHost);

	gamma_ln = 1.0f;
	if (controller[0].Command_Exist(this->module_name, "gamma"))
	{
		gamma_ln = atof(controller[0].Command(this->module_name, "gamma"));
	}

	int random_seed = rand();
	if (controller[0].Command_Exist(this->module_name, "seed"))
	{
		random_seed = atoi(controller[0].Command(this->module_name, "seed"));
	}

	controller[0].printf("    target temperature is %.2f K\n", target_temperature);
	controller[0].printf("    friction coefficient is %.2f ps^-1\n", gamma_ln);
	controller[0].printf("    random seed is %d\n", random_seed);

	dt = 0.001;
	if (controller[0].Command_Exist("dt"))
		dt = atof(controller[0].Command("dt"));
	dt *= CONSTANT_TIME_CONVERTION;
	half_dt = 0.5 * dt;

	float4_numbers = ceil((double)3.* atom_numbers / 4.);
	Cuda_Malloc_Safely((void**)&random_force, sizeof(float4)* float4_numbers);
	Cuda_Malloc_Safely((void**)&rand_state, sizeof(hiprandStatePhilox4_32_10_t)* float4_numbers);

	Setup_Rand_Normal_Kernel << <(unsigned int)ceilf((float)float4_numbers / threads_per_block), threads_per_block >> >
		(float4_numbers, rand_state, random_seed);

	gamma_ln = gamma_ln / CONSTANT_TIME_CONVERTION;//单位换算

	exp_gamma = expf(-gamma_ln * dt);

	float sart_gamma = sqrtf((1. - exp_gamma * exp_gamma) * target_temperature * CONSTANT_kB);
	Cuda_Malloc_Safely((void**)&d_sqrt_mass, sizeof(float)* atom_numbers);
	Cuda_Malloc_Safely((void**)&d_mass_inverse, sizeof(float)* atom_numbers);
	Malloc_Safely((void**)&h_sqrt_mass, sizeof(float)* atom_numbers);
	for (int i = 0; i < atom_numbers; i = i + 1)
	{
		if (h_mass_temp[i] == 0)
			h_sqrt_mass[i] = 0;
		else
			h_sqrt_mass[i] = sart_gamma * sqrtf(1. / h_mass_temp[i]);
	}
	hipMemcpy(d_sqrt_mass, h_sqrt_mass, sizeof(float)* atom_numbers, hipMemcpyHostToDevice);

	//确定是否加上速度上限
	max_velocity = 0;
	if (controller[0].Command_Exist(this->module_name, "velocity_max"))
	{
		sscanf(controller[0].Command(this->module_name, "velocity_max"), "%f", &max_velocity);
		controller[0].printf("    max velocity is %.2f\n", max_velocity);
	}
	//记录质量的倒数
	for (int i = 0; i < atom_numbers; i = i + 1)
	{
		if (h_mass_temp[i] == 0)
			h_mass_temp[i] = 0;
		else
			h_mass_temp[i] = 1.0f / h_mass_temp[i];
	}
	hipMemcpy(d_mass_inverse, h_mass_temp, sizeof(float)*atom_numbers, hipMemcpyHostToDevice);
	free(h_mass_temp);
	is_initialized = 1;
	if (is_initialized && !is_controller_printf_initialized)
	{
		is_controller_printf_initialized = 1;
		controller[0].printf("    structure last modify date is %d\n", last_modify_date);
	}
	controller[0].printf("END INITIALIZING MIDDLE LANGEVIN DYNAMICS\n\n");
}

void MIDDLE_Langevin_INFORMATION::MD_Iteration_Leap_Frog(VECTOR *frc, VECTOR *vel, VECTOR *acc, VECTOR *crd)
{
	if (is_initialized)
	{
		Rand_Normal << <ceilf((float)float4_numbers / 32.), 32 >> >
			(float4_numbers, rand_state, (float4 *)random_force);

		if (max_velocity <= 0)
		{
			MD_Iteration_Leap_Frog_With_LiuJian
				<< <ceilf((float)atom_numbers / 32), 32 >> >
				(atom_numbers, half_dt, dt, exp_gamma, d_mass_inverse,
				d_sqrt_mass, vel, crd, frc, acc, random_force);
		}
		else
		{
			MD_Iteration_Leap_Frog_With_LiuJian_With_Max_Velocity
				<< <ceilf((float)atom_numbers / 32), 32 >> >
				(atom_numbers, half_dt, dt, exp_gamma, d_mass_inverse,
				d_sqrt_mass, vel, crd, frc, acc, random_force,max_velocity);
			//hipDeviceSynchronize();
		}
	}
}

void MIDDLE_Langevin_INFORMATION::Clear()
{
	if (is_initialized)
	{
		is_initialized = 0;
		hipFree(rand_state);
		hipFree(random_force);
		free(h_sqrt_mass);
		hipFree(d_sqrt_mass);
		hipFree(d_mass_inverse);

		rand_state = NULL;
		random_force = NULL;
		h_sqrt_mass = NULL;
		d_sqrt_mass = NULL;
		d_mass_inverse = NULL;
	}
}