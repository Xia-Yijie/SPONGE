#include "hip/hip_runtime.h"
#include "LJ_soft_core.cuh"

#define TWO_DIVIDED_BY_SQRT_PI 1.1283791670218446
#define ONE_DIVIDED_BY_3_SQRT_PI 0.18806319451591877

float Get_Beta_LJ(float cutoff, float tolerance)
{
    float beta, low, high, tempf;
    int ilow, ihigh;
    
    high = 1.0;
    ihigh = 1;
    
    while (1)
    {
        tempf = erfc(high * cutoff) / cutoff;
        if (tempf <= tolerance)
            break;
        high *= 2;
        ihigh++;
    }
    
    ihigh += 50;
    low = 0.0;
    for (ilow =1; ilow < ihigh; ilow++)
    {
        beta = (low + high) / 2;
        tempf = erfc(beta * cutoff) / cutoff;
        if (tempf >= tolerance)
            low = beta;
        else
            high = beta;
    }
    return beta;
}

__device__ __host__ VECTOR Get_Periodic_Displacement(const UINT_VECTOR_LJ_FEP_TYPE uvec_a, const UINT_VECTOR_LJ_FEP_TYPE uvec_b, const VECTOR scaler)
{
	VECTOR dr;
	dr.x = ((int)(uvec_a.uint_x - uvec_b.uint_x)) * scaler.x;
	dr.y = ((int)(uvec_a.uint_y - uvec_b.uint_y)) * scaler.y;
	dr.z = ((int)(uvec_a.uint_z - uvec_b.uint_z)) * scaler.z;
	return dr;
}

__global__ void Copy_LJ_Type_And_Mask_To_New_Crd(const int atom_numbers, UINT_VECTOR_LJ_FEP_TYPE *new_crd, const int *LJ_type_A, const int * LJ_type_B, const int * mask)
{
	int atom_i = blockDim.x*blockIdx.x + threadIdx.x;
	if (atom_i < atom_numbers)
	{
		new_crd[atom_i].LJ_type_A = LJ_type_A[atom_i];
		new_crd[atom_i].LJ_type_B = LJ_type_B[atom_i];
		new_crd[atom_i].mask = mask[atom_i];
	}
}

static __global__ void device_add(float *variable, const float adder)
{
	variable[0] += adder;
}

__global__ void Copy_Crd_And_Charge_To_New_Crd(const int atom_numbers, const UNSIGNED_INT_VECTOR *crd, UINT_VECTOR_LJ_FEP_TYPE *new_crd, const float *charge)
{
	int atom_i = blockDim.x*blockIdx.x + threadIdx.x;
	if (atom_i < atom_numbers)
	{
		new_crd[atom_i].uint_x = crd[atom_i].uint_x;
		new_crd[atom_i].uint_y = crd[atom_i].uint_y;
		new_crd[atom_i].uint_z = crd[atom_i].uint_z;
		new_crd[atom_i].charge = charge[atom_i];
	}
}

__global__ void Copy_Crd_To_New_Crd(const int atom_numbers, const UNSIGNED_INT_VECTOR *crd, UINT_VECTOR_LJ_FEP_TYPE *new_crd)
{
	int atom_i = blockDim.x*blockIdx.x + threadIdx.x;
	if (atom_i < atom_numbers)
	{
		new_crd[atom_i].uint_x = crd[atom_i].uint_x;
		new_crd[atom_i].uint_y = crd[atom_i].uint_y;
		new_crd[atom_i].uint_z = crd[atom_i].uint_z;
	}
}

static __global__ void Total_C6_Get(int atom_numbers, int * atom_lj_type_A, int * atom_lj_type_B, float * d_lj_Ab, float * d_lj_Bb,float * d_factor, const float lambda)
{
	int i, j;
	float temp_sum = 0.0;
	int xA, yA, xB, yB;
	int itype_A, jtype_A, itype_B, jtype_B, atom_pair_LJ_type_A, atom_pair_LJ_type_B;
	float lambda_ = 1.0 - lambda;
	for (i = blockIdx.x * blockDim.x + threadIdx.x; i < atom_numbers; i += gridDim.x * blockDim.x)
	{
		itype_A = atom_lj_type_A[i];
		itype_B = atom_lj_type_B[i];
		for (j = blockIdx.y * blockDim.y + threadIdx.y; j < atom_numbers; j += gridDim.y * blockDim.y)
		{
			jtype_A = atom_lj_type_A[j];
			jtype_B = atom_lj_type_B[j];
			yA = (jtype_A - itype_A);
			xA = yA >> 31;
			yA = (yA^xA) - xA;
			xA = jtype_A + itype_A;
			jtype_A = (xA + yA) >> 1;
			xA = (xA - yA) >> 1;
			atom_pair_LJ_type_A = (jtype_A*(jtype_A + 1) >> 1) + xA;

			yB = (jtype_B - itype_B);
			xB = yB >> 31;
			yB = (yB^xB) - xB;
			xB = jtype_B + itype_B;
			jtype_B = (xB + yB) >> 1;
			xB = (xB - yB) >> 1;
			atom_pair_LJ_type_B = (jtype_B*(jtype_B + 1) >> 1) + xB;
			
			temp_sum += lambda_ * d_lj_Ab[atom_pair_LJ_type_A];
			temp_sum += lambda * d_lj_Bb[atom_pair_LJ_type_B];
		}
	}
	atomicAdd(d_factor, temp_sum);
}

static __global__ void LJ_Soft_Core_Force_With_Direct_CF_CUDA(
	const int atom_numbers, const ATOM_GROUP *nl,
	const UINT_VECTOR_LJ_FEP_TYPE *uint_crd, const VECTOR boxlength,
	const float *LJ_type_AA, const float *LJ_type_AB, const float * LJ_type_BA, const float * LJ_type_BB,const float cutoff,
	VECTOR *frc,const float pme_beta,const float sqrt_pi, const float lambda, const float alpha_lambda_p, const float alpha_lambda_p_, const float input_sigma_6, const float input_sigma_6_min)
{
	int atom_i = blockDim.x*blockIdx.x + threadIdx.x;
	float lambda_ = 1.0 - lambda;
	if (atom_i < atom_numbers)
	{
		ATOM_GROUP nl_i = nl[atom_i];
		int N = nl_i.atom_numbers;
		int atom_j;
		int int_x;
		int int_y;
		int int_z;
		UINT_VECTOR_LJ_FEP_TYPE r1 = uint_crd[atom_i], r2;
		VECTOR dr;
		float dr2, dr4, dr6;
		float dr_sc_A6, dr_sc_B6;
		float dr_sc_A, dr_sc_B;
		float dr_sc_A12, dr_sc_B12;
		float dr_2;
		float dr_4;
		float dr_8;
		float dr_6;
		float frc_abs = 0.;
		float AAij, ABij, BAij, BBij;
		float sigma_Aij, sigma_Bij;
		VECTOR frc_lin;
		VECTOR frc_record = { 0., 0., 0. };

		//CF
		float charge_i = r1.charge; //r1.charge;
		float charge_j;
		float dr_abs;
		float dr_1;
		float beta_dr;
		float beta_dr_sc_A, beta_dr_sc_B;
		float frc_cf_abs;
		//

		int xA, yA, xB, yB;
		int atom_pair_LJ_type_A, atom_pair_LJ_type_B;

		int mask_i = r1.mask, mask_j;
		bool soft_core;
		for (int j = threadIdx.y; j < N; j = j + blockDim.y)
		{
			atom_j = nl_i.atom_serial[j];
			r2 = uint_crd[atom_j];
			//CF
			charge_j = r2.charge;
			mask_j = r2.mask;

			int_x = r2.uint_x - r1.uint_x;
			int_y = r2.uint_y - r1.uint_y;
			int_z = r2.uint_z - r1.uint_z;
			dr.x = boxlength.x*int_x;
			dr.y = boxlength.y*int_y;
			dr.z = boxlength.z*int_z;
			dr_abs = norm3df(dr.x, dr.y, dr.z);
			if (dr_abs < cutoff)
			{
				yA = (r2.LJ_type_A - r1.LJ_type_A);
				xA = yA >> 31;
				yA = (yA^xA) - xA;
				xA = r2.LJ_type_A + r1.LJ_type_A;
				r2.LJ_type_A = (xA + yA) >> 1;
				xA = (xA - yA) >> 1;
				atom_pair_LJ_type_A = (r2.LJ_type_A*(r2.LJ_type_A + 1) >> 1) + xA;
				AAij = LJ_type_AA[atom_pair_LJ_type_A];
				ABij = LJ_type_AB[atom_pair_LJ_type_A];

				yB = (r2.LJ_type_B - r1.LJ_type_B);
				xB = yB >> 31;
				yB = (yB^xB) - xB;
				xB = r2.LJ_type_B + r1.LJ_type_B;
				r2.LJ_type_B = (xB + yB) >> 1;
				xB = (xB - yB) >> 1;
				atom_pair_LJ_type_B = (r2.LJ_type_B*(r2.LJ_type_B + 1) >> 1) + xB;
				BAij = LJ_type_BA[atom_pair_LJ_type_B];
				BBij = LJ_type_BB[atom_pair_LJ_type_B];
				
				soft_core = (mask_i != mask_j) || (BAij > 1e-6 && AAij < 1e-6) || (BAij < 1e-6 && AAij > 1e-6);
				if (!soft_core)
				{
					dr_1 = 1. / dr_abs;
					dr_2 = dr_1*dr_1;
					dr_4 = dr_2*dr_2;
					dr_8 = dr_4*dr_4;
					dr_6 = dr_4 * dr_2;
					frc_abs = (-(lambda_ * AAij + lambda * BAij) * dr_6
						+ (lambda_ * ABij + lambda * BBij)) * dr_8;
					
					beta_dr = pme_beta*dr_abs;
					frc_cf_abs = beta_dr *sqrt_pi * expf(-beta_dr*beta_dr) + erfcf(beta_dr);
					frc_cf_abs = frc_cf_abs * dr_2 *dr_1;
					frc_cf_abs = charge_i * charge_j*frc_cf_abs;
	
					frc_abs = frc_abs - frc_cf_abs;
				}
				else
				{
					dr2 = dr_abs * dr_abs;
					dr4 = dr2 * dr2;
					dr6 = dr4 * dr2;
					if (AAij < 1e-6 || ABij < 1e-6)
						sigma_Aij = input_sigma_6;
					else
						sigma_Aij = max(0.5 * AAij / ABij, input_sigma_6_min);
					if (BAij < 1e-6 || BBij < 1e-6)
						sigma_Bij = input_sigma_6;
					else
						sigma_Bij = max(0.5 * BAij / BBij, input_sigma_6_min);

					dr_sc_A6 = 1.0 / (dr6 + alpha_lambda_p * sigma_Aij);
					dr_sc_B6 = 1.0 / (dr6 + alpha_lambda_p_ * sigma_Bij);
					dr_sc_A12 = dr_sc_A6 * dr_sc_A6;
					dr_sc_B12 = dr_sc_B6 * dr_sc_B6;

					frc_abs = dr4 * (
						lambda_ * ( - AAij * dr_sc_A6 + ABij) * dr_sc_A12 
						+lambda * ( - BAij * dr_sc_B6 + BBij) * dr_sc_B12
					);

					dr_sc_A = pow(dr_sc_A6, 1.0/6.0);
					dr_sc_B = pow(dr_sc_B6, 1.0/6.0);
					beta_dr_sc_A = pme_beta / dr_sc_A;
					beta_dr_sc_B = pme_beta / dr_sc_B;

					frc_cf_abs = dr4 * (
						lambda_ * (expf(-beta_dr_sc_A * beta_dr_sc_A) * sqrt_pi * pme_beta + erfcf(beta_dr_sc_A) * dr_sc_A) * dr_sc_A6
						+ lambda * (expf(-beta_dr_sc_B * beta_dr_sc_B) * sqrt_pi * pme_beta + erfcf(beta_dr_sc_B) * dr_sc_B) * dr_sc_B6
					);
					frc_cf_abs = frc_cf_abs * charge_i * charge_j;

					frc_abs = frc_abs - frc_cf_abs;
				}


				frc_lin.x = frc_abs*dr.x;
				frc_lin.y = frc_abs*dr.y;
				frc_lin.z = frc_abs*dr.z;

				frc_record.x = frc_record.x + frc_lin.x;
				frc_record.y = frc_record.y + frc_lin.y;
				frc_record.z = frc_record.z + frc_lin.z;

				atomicAdd(&frc[atom_j].x, -frc_lin.x);
				atomicAdd(&frc[atom_j].y, -frc_lin.y);
				atomicAdd(&frc[atom_j].z, -frc_lin.z);
			}
		}//atom_j cycle
		atomicAdd(&frc[atom_i].x, frc_record.x);
		atomicAdd(&frc[atom_i].y, frc_record.y);
		atomicAdd(&frc[atom_i].z, frc_record.z);
	}
}


static __global__ void LJ_Soft_Core_Direct_CF_Force_With_Atom_Energy_CUDA(
	const int atom_numbers, const ATOM_GROUP *nl,
	const UINT_VECTOR_LJ_FEP_TYPE *uint_crd, const VECTOR boxlength,
	const float *LJ_type_AA, const float * LJ_type_AB, const float *LJ_type_BA, const float * LJ_type_BB,const float cutoff,
	VECTOR *frc, const float pme_beta, const float sqrt_pi,float *atom_energy, const float lambda, const float alpha_lambda_p, const float alpha_lambda_p_, const float input_sigma_6, const float input_sigma_6_min)
{
	int atom_i = blockDim.x*blockIdx.x + threadIdx.x;
	float lambda_ = 1.0 - lambda;
	if (atom_i < atom_numbers)
	{
		ATOM_GROUP nl_i = nl[atom_i];
		int N = nl_i.atom_numbers;
		int atom_j;
		int int_x;
		int int_y;
		int int_z;
		UINT_VECTOR_LJ_FEP_TYPE r1 = uint_crd[atom_i], r2;
		VECTOR dr;
		float dr2, dr4, dr6;
		float dr_sc_A6, dr_sc_B6;
		float dr_sc_A, dr_sc_B;
		float dr_sc_A12, dr_sc_B12;
		float dr_2;
		float dr_4;
		float dr_8;
		float dr_6;
		float frc_abs = 0.;
		float AAij, ABij, BAij, BBij;
		float sigma_Aij, sigma_Bij;
		VECTOR frc_lin;
		VECTOR frc_record = { 0., 0., 0. };

		float charge_i = r1.charge; //r1.charge;
		float charge_j;
		float dr_abs;
		float dr_1;
		float beta_dr;
		float beta_dr_sc_A, beta_dr_sc_B;
		float frc_cf_abs;

		float ene_lin = 0.0;
		float ene_lin2 = 0.0;

		int xA, yA, xB, yB;
		int atom_pair_LJ_type_A, atom_pair_LJ_type_B;

		int mask_i = r1.mask, mask_j;
		bool soft_core;
		for (int j = threadIdx.y; j < N; j = j + blockDim.y)
		{
			atom_j = nl_i.atom_serial[j];
			r2 = uint_crd[atom_j];
			charge_j = r2.charge;
			mask_j = r2.mask;

			int_x = r2.uint_x - r1.uint_x;
			int_y = r2.uint_y - r1.uint_y;
			int_z = r2.uint_z - r1.uint_z;
			dr.x = boxlength.x*int_x;
			dr.y = boxlength.y*int_y;
			dr.z = boxlength.z*int_z;
			dr_abs = norm3df(dr.x, dr.y, dr.z);
			if (dr_abs < cutoff)
			{
				yA = (r2.LJ_type_A - r1.LJ_type_A);
				xA = yA >> 31;
				yA = (yA^xA) - xA;
				xA = r2.LJ_type_A + r1.LJ_type_A;
				r2.LJ_type_A = (xA + yA) >> 1;
				xA = (xA - yA) >> 1;
				atom_pair_LJ_type_A = (r2.LJ_type_A*(r2.LJ_type_A + 1) >> 1) + xA;
				AAij = LJ_type_AA[atom_pair_LJ_type_A];
				ABij = LJ_type_AB[atom_pair_LJ_type_A];

				yB = (r2.LJ_type_B - r1.LJ_type_B);
				xB = yB >> 31;
				yB = (yB^xB) - xB;
				xB = r2.LJ_type_B + r1.LJ_type_B;
				r2.LJ_type_B = (xB + yB) >> 1;
				xB = (xB - yB) >> 1;
				atom_pair_LJ_type_B = (r2.LJ_type_B*(r2.LJ_type_B + 1) >> 1) + xB;
				BAij = LJ_type_BA[atom_pair_LJ_type_B];
				BBij = LJ_type_BB[atom_pair_LJ_type_B];

				soft_core = (mask_i != mask_j) || (BAij > 1e-6 && AAij < 1e-6) || (BAij < 1e-6 && AAij > 1e-6);
				if (!soft_core)
				{
					dr_1 = 1. / dr_abs;
					dr_2 = dr_1*dr_1;
					dr_4 = dr_2*dr_2;
					dr_8 = dr_4*dr_4;
					dr_6 = dr_4 * dr_2;
					frc_abs = (-(lambda_ * AAij + lambda * BAij) * dr_6
						+ (lambda_ * ABij + lambda * BBij)) * dr_8;
					
					beta_dr = pme_beta*dr_abs;
					frc_cf_abs = beta_dr *sqrt_pi * expf(-beta_dr*beta_dr) + erfcf(beta_dr);
					frc_cf_abs = frc_cf_abs * dr_2 *dr_1;
					frc_cf_abs = charge_i * charge_j*frc_cf_abs;
	
					frc_abs = frc_abs - frc_cf_abs;

					ene_lin2 = ene_lin2 + charge_i * charge_j * erfcf(beta_dr) * dr_1;
					ene_lin = ene_lin + (0.083333333* (lambda_ * AAij + lambda * BAij) * dr_6
						- 0.166666666*(lambda_ * ABij + lambda * BBij)) * dr_6;
				}
				else
				{
					dr2 = dr_abs * dr_abs;
					dr4 = dr2 * dr2;
					dr6 = dr4 * dr2;
					if (AAij < 1e-6 || ABij < 1e-6)
						sigma_Aij = input_sigma_6;
					else
						sigma_Aij = max(0.5 * AAij / ABij, input_sigma_6_min);
					if (BAij < 1e-6 || BBij < 1e-6)
						sigma_Bij = input_sigma_6;
					else
						sigma_Bij = max(0.5 * BAij / BBij, input_sigma_6_min);

					dr_sc_A6 = 1.0 / (dr6 + alpha_lambda_p * sigma_Aij);
					dr_sc_B6 = 1.0 / (dr6 + alpha_lambda_p_ * sigma_Bij);
					dr_sc_A12 = dr_sc_A6 * dr_sc_A6;
					dr_sc_B12 = dr_sc_B6 * dr_sc_B6;

					frc_abs = dr4 * (
						lambda_ * ( - AAij * dr_sc_A6 + ABij) * dr_sc_A12 
						+lambda * ( - BAij * dr_sc_B6 + BBij) * dr_sc_B12
					);
					
					dr_sc_A = pow(dr_sc_A6, 1.0/6.0);
					dr_sc_B = pow(dr_sc_B6, 1.0/6.0);
					beta_dr_sc_A = pme_beta / dr_sc_A;
					beta_dr_sc_B = pme_beta / dr_sc_B;

					frc_cf_abs = dr4 * (
						lambda_ * (expf(-beta_dr_sc_A * beta_dr_sc_A) * sqrt_pi * pme_beta + erfcf(beta_dr_sc_A) * dr_sc_A) * dr_sc_A6
						+ lambda * (expf(-beta_dr_sc_B * beta_dr_sc_B) * sqrt_pi * pme_beta + erfcf(beta_dr_sc_B) * dr_sc_B) * dr_sc_B6
					);
					frc_cf_abs = frc_cf_abs * charge_i * charge_j;

					frc_abs = frc_abs - frc_cf_abs;

					ene_lin2 = ene_lin2 + charge_i * charge_j * (lambda_ * erfcf(beta_dr_sc_A) * dr_sc_A + lambda * erfcf(beta_dr_sc_B) * dr_sc_B);

					ene_lin = ene_lin + 
					lambda_ * ( 0.083333333 * AAij * dr_sc_A6 - 0.166666666 * ABij) * dr_sc_A6 + lambda * ( 0.083333333 * BAij * dr_sc_B6 - 0.166666666 * BBij) * dr_sc_B6;
				}


				frc_lin.x = frc_abs*dr.x;
				frc_lin.y = frc_abs*dr.y;
				frc_lin.z = frc_abs*dr.z;

				frc_record.x = frc_record.x + frc_lin.x;
				frc_record.y = frc_record.y + frc_lin.y;
				frc_record.z = frc_record.z + frc_lin.z;

				atomicAdd(&frc[atom_j].x, -frc_lin.x);
				atomicAdd(&frc[atom_j].y, -frc_lin.y);
				atomicAdd(&frc[atom_j].z, -frc_lin.z);
			}
		}//atom_j cycle
		atomicAdd(&frc[atom_i].x, frc_record.x);
		atomicAdd(&frc[atom_i].y, frc_record.y);
		atomicAdd(&frc[atom_i].z, frc_record.z);

		atomicAdd(&atom_energy[atom_i], ene_lin + ene_lin2);
	}
}

void __global__ LJ_Soft_Core_Direct_CF_Force_With_LJ_Virial_Direct_CF_Energy_CUDA(
	const int atom_numbers, const ATOM_GROUP *nl,
	const UINT_VECTOR_LJ_FEP_TYPE *uint_crd, const VECTOR boxlength,
	const float *LJ_type_AA, const float *LJ_type_AB, const float * LJ_type_BA, const float * LJ_type_BB,const float cutoff,
	VECTOR *frc,const float pme_beta,const float sqrt_pi, float *atom_lj_virial, float *atom_direct_cf_energy, const float lambda, const float alpha_lambda_p, const float alpha_lambda_p_, const float input_sigma_6, const float input_sigma_6_min)
{
	int atom_i = blockDim.x*blockIdx.x + threadIdx.x;
	float lambda_ = 1.0 - lambda;
	if (atom_i < atom_numbers)
	{
		ATOM_GROUP nl_i = nl[atom_i];
		int N = nl_i.atom_numbers;
		int atom_j;
		int int_x;
		int int_y;
		int int_z;
		UINT_VECTOR_LJ_FEP_TYPE r1 = uint_crd[atom_i], r2;
		VECTOR dr;
		float dr2, dr4, dr6;
		float dr_sc_A6, dr_sc_B6;
		float dr_sc_A, dr_sc_B;
		float dr_sc_A12, dr_sc_B12;
		float dr_2;
		float dr_4;
		float dr_8;
		float dr_6;
		float frc_abs = 0.;
		float AAij, ABij, BAij, BBij;
		float sigma_Aij, sigma_Bij;
		VECTOR frc_lin;
		VECTOR frc_record = { 0., 0., 0. };

		//CF
		float charge_i = r1.charge; //r1.charge;
		float charge_j;
		float dr_abs;
		float dr_1;
		float beta_dr;
		float beta_dr_sc_A, beta_dr_sc_B;
		float frc_cf_abs;

		float virial_lin = 0.0;
		float energy_lin = 0.0;

		int xA, yA, xB, yB;
		int atom_pair_LJ_type_A, atom_pair_LJ_type_B;
		bool soft_core;

		int mask_i = r1.mask, mask_j;
		for (int j = threadIdx.y; j < N; j = j + blockDim.y)
		{
			atom_j = nl_i.atom_serial[j];
			r2 = uint_crd[atom_j];
			//CF
			charge_j = r2.charge;
			mask_j = r2.mask;

			int_x = r2.uint_x - r1.uint_x;
			int_y = r2.uint_y - r1.uint_y;
			int_z = r2.uint_z - r1.uint_z;
			dr.x = boxlength.x*int_x;
			dr.y = boxlength.y*int_y;
			dr.z = boxlength.z*int_z;
			dr_abs = norm3df(dr.x, dr.y, dr.z);
			if (dr_abs < cutoff)
			{
				yA = (r2.LJ_type_A - r1.LJ_type_A);
				xA = yA >> 31;
				yA = (yA^xA) - xA;
				xA = r2.LJ_type_A + r1.LJ_type_A;
				r2.LJ_type_A = (xA + yA) >> 1;
				xA = (xA - yA) >> 1;
				atom_pair_LJ_type_A = (r2.LJ_type_A*(r2.LJ_type_A + 1) >> 1) + xA;
				AAij = LJ_type_AA[atom_pair_LJ_type_A];
				ABij = LJ_type_AB[atom_pair_LJ_type_A];

				yB = (r2.LJ_type_B - r1.LJ_type_B);
				xB = yB >> 31;
				yB = (yB^xB) - xB;
				xB = r2.LJ_type_B + r1.LJ_type_B;
				r2.LJ_type_B = (xB + yB) >> 1;
				xB = (xB - yB) >> 1;
				atom_pair_LJ_type_B = (r2.LJ_type_B*(r2.LJ_type_B + 1) >> 1) + xB;
				BAij = LJ_type_BA[atom_pair_LJ_type_B];
				BBij = LJ_type_BB[atom_pair_LJ_type_B];
				
				soft_core = (mask_i != mask_j) || (BAij > 1e-6 && AAij < 1e-6) || (BAij < 1e-6 && AAij > 1e-6);
				if (!soft_core)
				{
					dr_1 = 1. / dr_abs;
					dr_2 = dr_1*dr_1;
					dr_4 = dr_2*dr_2;
					dr_8 = dr_4*dr_4;
					dr_6 = dr_4 * dr_2;
					frc_abs = (-(lambda_ * AAij + lambda * BAij) * dr_6
						+ (lambda_ * ABij + lambda * BBij)) * dr_8;
					
					beta_dr = pme_beta*dr_abs;
					frc_cf_abs = beta_dr *sqrt_pi * expf(-beta_dr*beta_dr) + erfcf(beta_dr);
					frc_cf_abs = frc_cf_abs * dr_2 *dr_1;
					frc_cf_abs = charge_i * charge_j*frc_cf_abs;

					virial_lin = virial_lin - frc_abs * dr_abs * dr_abs;
	
					frc_abs = frc_abs - frc_cf_abs;

					energy_lin = energy_lin + charge_i * charge_j * erfcf(beta_dr) * dr_1;
				}
				else
				{
					dr2 = dr_abs * dr_abs;
					dr4 = dr2 * dr2;
					dr6 = dr4 * dr2;
					if (AAij < 1e-6 || ABij < 1e-6)
						sigma_Aij = input_sigma_6;
					else
						sigma_Aij = max(0.5 * AAij / ABij, input_sigma_6_min);
					if (BAij < 1e-6 || BBij < 1e-6)
						sigma_Bij = input_sigma_6;
					else
						sigma_Bij = max(0.5 * BAij / BBij, input_sigma_6_min);

					dr_sc_A6 = 1.0 / (dr6 + alpha_lambda_p * sigma_Aij);
					dr_sc_B6 = 1.0 / (dr6 + alpha_lambda_p_ * sigma_Bij);
					dr_sc_A12 = dr_sc_A6 * dr_sc_A6;
					dr_sc_B12 = dr_sc_B6 * dr_sc_B6;

					frc_abs = dr4 * (
						lambda_ * ( - AAij * dr_sc_A6 + ABij) * dr_sc_A12 
						+lambda * ( - BAij * dr_sc_B6 + BBij) * dr_sc_B12
					);
					
					dr_sc_A = pow(dr_sc_A6, 1.0/6.0);
					dr_sc_B = pow(dr_sc_B6, 1.0/6.0);
					beta_dr_sc_A = pme_beta / dr_sc_A;
					beta_dr_sc_B = pme_beta / dr_sc_B;

					frc_cf_abs = dr4 * (
						lambda_ * (expf(-beta_dr_sc_A * beta_dr_sc_A) * sqrt_pi * pme_beta + erfcf(beta_dr_sc_A) * dr_sc_A) * dr_sc_A6
						+ lambda * (expf(-beta_dr_sc_B * beta_dr_sc_B) * sqrt_pi * pme_beta + erfcf(beta_dr_sc_B) * dr_sc_B) * dr_sc_B6
					);
					frc_cf_abs = frc_cf_abs * charge_i * charge_j;

					virial_lin = virial_lin - frc_abs * dr_abs * dr_abs;

					frc_abs = frc_abs - frc_cf_abs;

					energy_lin = energy_lin + charge_i * charge_j * (lambda_ * erfcf(beta_dr_sc_A) * dr_sc_A + lambda * erfcf(beta_dr_sc_B) * dr_sc_B);
				}


				frc_lin.x = frc_abs*dr.x;
				frc_lin.y = frc_abs*dr.y;
				frc_lin.z = frc_abs*dr.z;

				frc_record.x = frc_record.x + frc_lin.x;
				frc_record.y = frc_record.y + frc_lin.y;
				frc_record.z = frc_record.z + frc_lin.z;

				atomicAdd(&frc[atom_j].x, -frc_lin.x);
				atomicAdd(&frc[atom_j].y, -frc_lin.y);
				atomicAdd(&frc[atom_j].z, -frc_lin.z);
			}
		}//atom_j cycle
		atomicAdd(&frc[atom_i].x, frc_record.x);
		atomicAdd(&frc[atom_i].y, frc_record.y);
		atomicAdd(&frc[atom_i].z, frc_record.z);

		atomicAdd(&atom_direct_cf_energy[atom_i], energy_lin);
		atomicAdd(&atom_lj_virial[atom_i], virial_lin);
	}
}

static __global__ void LJ_Soft_Core_Direct_CF_Force_With_Atom_Energy_And_LJ_Virial_Direct_CF_Energy_CUDA(
	const int atom_numbers, const ATOM_GROUP *nl,
	const UINT_VECTOR_LJ_FEP_TYPE *uint_crd, const VECTOR boxlength,
	const float *LJ_type_AA, const float * LJ_type_AB, const float *LJ_type_BA, const float * LJ_type_BB,const float cutoff,
	VECTOR *frc, const float pme_beta, const float sqrt_pi,float *atom_energy, float * atom_lj_virial, float * atom_direct_cf_energy, const float lambda, const float alpha_lambda_p, const float alpha_lambda_p_, const float input_sigma_6, const float input_sigma_6_min)
{
	int atom_i = blockDim.x*blockIdx.x + threadIdx.x;
	float lambda_ = 1.0 - lambda;
	if (atom_i < atom_numbers)
	{
		ATOM_GROUP nl_i = nl[atom_i];
		int N = nl_i.atom_numbers;
		int atom_j;
		int int_x;
		int int_y;
		int int_z;
		UINT_VECTOR_LJ_FEP_TYPE r1 = uint_crd[atom_i], r2;
		VECTOR dr;
		float dr2, dr4, dr6;
		float dr_sc_A6, dr_sc_B6;
		float dr_sc_A, dr_sc_B;
		float dr_sc_A12, dr_sc_B12;
		float dr_2;
		float dr_4;
		float dr_8;
		float dr_6;
		float frc_abs = 0.;
		float AAij, ABij, BAij, BBij;
		float sigma_Aij, sigma_Bij;
		VECTOR frc_lin;
		VECTOR frc_record = { 0., 0., 0. };

		//CF
		float charge_i = r1.charge; //r1.charge;
		float charge_j;
		float dr_abs;
		float dr_1;
		float beta_dr;
		float beta_dr_sc_A, beta_dr_sc_B;
		float frc_cf_abs;
		//

		//能量
		float ene_lin = 0.0;
		float ene_lin2 = 0.0;
		float virial_lin = 0.0;

		int xA, yA, xB, yB;
		int atom_pair_LJ_type_A, atom_pair_LJ_type_B;
		int mask_i = r1.mask, mask_j;
		bool soft_core;
		for (int j = threadIdx.y; j < N; j = j + blockDim.y)
		{
			atom_j = nl_i.atom_serial[j];
			r2 = uint_crd[atom_j];
			charge_j = r2.charge;
			mask_j = r2.mask;

			int_x = r2.uint_x - r1.uint_x;
			int_y = r2.uint_y - r1.uint_y;
			int_z = r2.uint_z - r1.uint_z;
			dr.x = boxlength.x*int_x;
			dr.y = boxlength.y*int_y;
			dr.z = boxlength.z*int_z;
			dr_abs = norm3df(dr.x, dr.y, dr.z);
			if (dr_abs < cutoff)
			{
				yA = (r2.LJ_type_A - r1.LJ_type_A);
				xA = yA >> 31;
				yA = (yA^xA) - xA;
				xA = r2.LJ_type_A + r1.LJ_type_A;
				r2.LJ_type_A = (xA + yA) >> 1;
				xA = (xA - yA) >> 1;
				atom_pair_LJ_type_A = (r2.LJ_type_A*(r2.LJ_type_A + 1) >> 1) + xA;
				AAij = LJ_type_AA[atom_pair_LJ_type_A];
				ABij = LJ_type_AB[atom_pair_LJ_type_A];

				yB = (r2.LJ_type_B - r1.LJ_type_B);
				xB = yB >> 31;
				yB = (yB^xB) - xB;
				xB = r2.LJ_type_B + r1.LJ_type_B;
				r2.LJ_type_B = (xB + yB) >> 1;
				xB = (xB - yB) >> 1;
				atom_pair_LJ_type_B = (r2.LJ_type_B*(r2.LJ_type_B + 1) >> 1) + xB;
				BAij = LJ_type_BA[atom_pair_LJ_type_B];
				BBij = LJ_type_BB[atom_pair_LJ_type_B];

				soft_core = (mask_i != mask_j) || (BAij > 1e-6 && AAij < 1e-6) || (BAij < 1e-6 && AAij > 1e-6);
				if (!soft_core)
				{
					dr_1 = 1. / dr_abs;
					dr_2 = dr_1*dr_1;
					dr_4 = dr_2*dr_2;
					dr_8 = dr_4*dr_4;
					dr_6 = dr_4 * dr_2;
					frc_abs = (-(lambda_ * AAij + lambda * BAij) * dr_6
						+ (lambda_ * ABij + lambda * BBij)) * dr_8;
					
					beta_dr = pme_beta*dr_abs;
					frc_cf_abs = beta_dr *sqrt_pi * expf(-beta_dr*beta_dr) + erfcf(beta_dr);
					frc_cf_abs = frc_cf_abs * dr_2 *dr_1;
					frc_cf_abs = charge_i * charge_j*frc_cf_abs;
					
					virial_lin = virial_lin - frc_abs * dr_abs * dr_abs;
	
					frc_abs = frc_abs - frc_cf_abs;

					ene_lin2 = ene_lin2 + charge_i * charge_j * erfcf(beta_dr) * dr_1;
					ene_lin = ene_lin + (0.083333333* (lambda_ * AAij + lambda * BAij) * dr_6
						- 0.166666666*(lambda_ * ABij + lambda * BBij)) * dr_6;
				}
				else
				{
					dr2 = dr_abs * dr_abs;
					dr4 = dr2 * dr2;
					dr6 = dr4 * dr2;
					if (AAij < 1e-6 || ABij < 1e-6)
						sigma_Aij = input_sigma_6;
					else
						sigma_Aij = max(0.5 * AAij / ABij, input_sigma_6_min);
					if (BAij < 1e-6 || BBij < 1e-6)
						sigma_Bij = input_sigma_6;
					else
						sigma_Bij = max(0.5 * BAij / BBij, input_sigma_6_min);

					dr_sc_A6 = 1.0 / (dr6 + alpha_lambda_p * sigma_Aij);
					dr_sc_B6 = 1.0 / (dr6 + alpha_lambda_p_ * sigma_Bij);
					dr_sc_A12 = dr_sc_A6 * dr_sc_A6;
					dr_sc_B12 = dr_sc_B6 * dr_sc_B6;

					frc_abs = dr4 * (
						lambda_ * ( - AAij * dr_sc_A6 + ABij) * dr_sc_A12 
						+lambda * ( - BAij * dr_sc_B6 + BBij) * dr_sc_B12
					);
					
					dr_sc_A = pow(dr_sc_A6, 1.0/6.0);
					dr_sc_B = pow(dr_sc_B6, 1.0/6.0);
					beta_dr_sc_A = pme_beta / dr_sc_A;
					beta_dr_sc_B = pme_beta / dr_sc_B;

					frc_cf_abs = dr4 * (
						lambda_ * (expf(-beta_dr_sc_A * beta_dr_sc_A) * sqrt_pi * pme_beta + erfcf(beta_dr_sc_A) * dr_sc_A) * dr_sc_A6
						+ lambda * (expf(-beta_dr_sc_B * beta_dr_sc_B) * sqrt_pi * pme_beta + erfcf(beta_dr_sc_B) * dr_sc_B) * dr_sc_B6
					);
					frc_cf_abs = frc_cf_abs * charge_i * charge_j;

					virial_lin = virial_lin - frc_abs * dr_abs * dr_abs;

					frc_abs = frc_abs - frc_cf_abs;

					ene_lin2 = ene_lin2 + charge_i * charge_j * (lambda_ * erfcf(beta_dr_sc_A) * dr_sc_A + lambda * erfcf(beta_dr_sc_B) * dr_sc_B);

					ene_lin = ene_lin + 
					lambda_ * ( 0.083333333 * AAij * dr_sc_A6 - 0.166666666 * ABij) * dr_sc_A6 + lambda * ( 0.083333333 * BAij * dr_sc_B6 - 0.166666666 * BBij) * dr_sc_B6;
				}


				frc_lin.x = frc_abs*dr.x;
				frc_lin.y = frc_abs*dr.y;
				frc_lin.z = frc_abs*dr.z;

				frc_record.x = frc_record.x + frc_lin.x;
				frc_record.y = frc_record.y + frc_lin.y;
				frc_record.z = frc_record.z + frc_lin.z;

				atomicAdd(&frc[atom_j].x, -frc_lin.x);
				atomicAdd(&frc[atom_j].y, -frc_lin.y);
				atomicAdd(&frc[atom_j].z, -frc_lin.z);
			}
		}//atom_j cycle
		atomicAdd(&frc[atom_i].x, frc_record.x);
		atomicAdd(&frc[atom_i].y, frc_record.y);
		atomicAdd(&frc[atom_i].z, frc_record.z);

		atomicAdd(&atom_energy[atom_i], ene_lin + ene_lin2);
		atomicAdd(&atom_direct_cf_energy[atom_i], ene_lin2);
		atomicAdd(&atom_lj_virial[atom_i], virial_lin);
	}
}

static __global__ void LJ_Soft_Core_Energy_CUDA(
	const int atom_numbers, const ATOM_GROUP *nl,
	const UINT_VECTOR_LJ_FEP_TYPE *uint_crd, const VECTOR boxlength,
	const float *LJ_type_AA, const float *LJ_type_AB, const float * LJ_type_BA, const float * LJ_type_BB,const float cutoff, const float pme_beta,
	float * lj_ene, float * direct_ene, const float lambda, const float alpha_lambda_p, const float alpha_lambda_p_, const float input_sigma_6, const float input_sigma_6_min)
{
	int atom_i = blockDim.x*blockIdx.x + threadIdx.x;
	float lambda_ = 1.0 - lambda;
	if (atom_i < atom_numbers)
	{
		ATOM_GROUP nl_i = nl[atom_i];
		int N = nl_i.atom_numbers;
		int atom_j;
		int int_x;
		int int_y;
		int int_z;
		UINT_VECTOR_LJ_FEP_TYPE r1 = uint_crd[atom_i], r2;
		VECTOR dr;
		float dr2, dr4, dr6;
		float dr_sc_A6, dr_sc_B6;
		float dr_sc_A, dr_sc_B;
		float dr_2;
		float dr_4;
		float dr_6;
		float AAij, ABij, BAij, BBij;
		float sigma_Aij, sigma_Bij;

		float charge_i = r1.charge; //r1.charge;
		float charge_j;
		float dr_abs;
		float dr_1;
		float beta_dr;
		float beta_dr_sc_A, beta_dr_sc_B;

		float ene_lin = 0.0;
		float ene_lin2 = 0.0;

		int xA, yA, xB, yB;
		int atom_pair_LJ_type_A, atom_pair_LJ_type_B;
		int mask_i = r1.mask, mask_j;
		bool soft_core;
		for (int j = threadIdx.y; j < N; j = j + blockDim.y)
		{
			atom_j = nl_i.atom_serial[j];
			r2 = uint_crd[atom_j];
			charge_j = r2.charge;
			mask_j = r2.mask;

			int_x = r2.uint_x - r1.uint_x;
			int_y = r2.uint_y - r1.uint_y;
			int_z = r2.uint_z - r1.uint_z;
			dr.x = boxlength.x*int_x;
			dr.y = boxlength.y*int_y;
			dr.z = boxlength.z*int_z;
			dr_abs = norm3df(dr.x, dr.y, dr.z);
			if (dr_abs < cutoff)
			{
				yA = (r2.LJ_type_A - r1.LJ_type_A);
				xA = yA >> 31;
				yA = (yA^xA) - xA;
				xA = r2.LJ_type_A + r1.LJ_type_A;
				r2.LJ_type_A = (xA + yA) >> 1;
				xA = (xA - yA) >> 1;
				atom_pair_LJ_type_A = (r2.LJ_type_A*(r2.LJ_type_A + 1) >> 1) + xA;
				AAij = LJ_type_AA[atom_pair_LJ_type_A];
				ABij = LJ_type_AB[atom_pair_LJ_type_A];

				yB = (r2.LJ_type_B - r1.LJ_type_B);
				xB = yB >> 31;
				yB = (yB^xB) - xB;
				xB = r2.LJ_type_B + r1.LJ_type_B;
				r2.LJ_type_B = (xB + yB) >> 1;
				xB = (xB - yB) >> 1;
				atom_pair_LJ_type_B = (r2.LJ_type_B*(r2.LJ_type_B + 1) >> 1) + xB;
				BAij = LJ_type_BA[atom_pair_LJ_type_B];
				BBij = LJ_type_BB[atom_pair_LJ_type_B];

				soft_core = (mask_i != mask_j) || (BAij > 1e-6 && AAij < 1e-6) || (BAij < 1e-6 && AAij > 1e-6);
				if (!soft_core)
				{
					dr_1 = 1. / dr_abs;
					dr_2 = dr_1*dr_1;
					dr_4 = dr_2*dr_2;
					dr_6 = dr_4 * dr_2;
					
					beta_dr = pme_beta*dr_abs;

					ene_lin2 = ene_lin2 + charge_i * charge_j * erfcf(beta_dr) * dr_1;
					ene_lin = ene_lin + (0.083333333* (lambda_ * AAij + lambda * BAij) * dr_6
						- 0.166666666*(lambda_ * ABij + lambda * BBij)) * dr_6;
				}
				else
				{
					dr2 = dr_abs * dr_abs;
					dr4 = dr2 * dr2;
					dr6 = dr4 * dr2;
					if (AAij < 1e-6 || ABij < 1e-6)
						sigma_Aij = input_sigma_6;
					else
						sigma_Aij = max(0.5 * AAij / ABij, input_sigma_6_min);
					if (BAij < 1e-6 || BBij < 1e-6)
						sigma_Bij = input_sigma_6;
					else
						sigma_Bij = max(0.5 * BAij / BBij, input_sigma_6_min);

					dr_sc_A6 = 1.0 / (dr6 + alpha_lambda_p * sigma_Aij);
					dr_sc_B6 = 1.0 / (dr6 + alpha_lambda_p_ * sigma_Bij);
					
					dr_sc_A = pow(dr_sc_A6, 1.0/6.0);
					dr_sc_B = pow(dr_sc_B6, 1.0/6.0);
					beta_dr_sc_A = pme_beta / dr_sc_A;
					beta_dr_sc_B = pme_beta / dr_sc_B;

					ene_lin2 = ene_lin2 + charge_i * charge_j * (lambda_ * erfcf(beta_dr_sc_A) * dr_sc_A + lambda * erfcf(beta_dr_sc_B) * dr_sc_B);

					ene_lin = ene_lin + 
					lambda_ * ( 0.083333333 * AAij * dr_sc_A6 - 0.166666666 * ABij) * dr_sc_A6 + lambda * ( 0.083333333 * BAij * dr_sc_B6 - 0.166666666 * BBij) * dr_sc_B6;
				}
			}
		}//atom_j cycle

		atomicAdd(&lj_ene[atom_i], ene_lin);
		atomicAdd(direct_ene, ene_lin2);
	}
}

static __global__ void LJ_Soft_Core_Energy_With_Coulumb_Direct_CUDA(
	const int atom_numbers, const ATOM_GROUP *nl,
	const UINT_VECTOR_LJ_FEP_TYPE *uint_crd, const VECTOR boxlength,
	const float *LJ_type_AA, const float *LJ_type_AB, const float * LJ_type_BA, const float * LJ_type_BB,const float cutoff, const float pme_beta,
	float * lj_ene_intersys, float * lj_ene_intrasys, float * direct_ene_intersys, float * direct_ene_intrasys, const float lambda, const float alpha_lambda_p, const float alpha_lambda_p_, const float input_sigma_6, const float input_sigma_6_min)
{
	int atom_i = blockDim.x*blockIdx.x + threadIdx.x;
	float lambda_ = 1.0 - lambda;
	if (atom_i < atom_numbers)
	{
		ATOM_GROUP nl_i = nl[atom_i];
		int N = nl_i.atom_numbers;
		int atom_j;
		UINT_VECTOR_LJ_FEP_TYPE r1 = uint_crd[atom_i], r2;
		VECTOR dr;
		float dr2, dr4, dr6;
		float dr_scA6, dr_scB6;
		float dr_sc_A6, dr_sc_B6;
		float dr_scA, dr_scB;
		float dr_sc_A, dr_sc_B;
		float vdw_A, vdw_B, ee_A, ee_B;
		float dr_2;
		float dr_4;
		float dr_6;
		float AAij, ABij, BAij, BBij;
		float sigma_Aij, sigma_Bij;

		float charge_i = r1.charge; //r1.charge;
		float charge_j;
		float dr_abs;
		float dr_1;
		float beta_dr;
		float beta_dr_sc_A, beta_dr_sc_B;

		float ene_lin_intersys = 0.0;
		float ene_lin2_intersys = 0.0;
		float ene_lin_intrasys = 0.0;
		float ene_lin2_intrasys = 0.0;

		float ene_tmp = 0.0;
		float ene_tmp_2 = 0.0;

		int xA, yA, xB, yB;
		int atom_pair_LJ_type_A, atom_pair_LJ_type_B;
		int mask_i = r1.mask, mask_j;
		bool soft_core;
		for (int j = threadIdx.y; j < N; j = j + blockDim.y)
		{
			atom_j = nl_i.atom_serial[j];
			r2 = uint_crd[atom_j];
			charge_j = r2.charge;
			mask_j = r2.mask;

			dr = Get_Periodic_Displacement(r1, r2, boxlength);

			dr_abs = norm3df(dr.x, dr.y, dr.z);
			//dr_abs = sqrt(dr.x * dr.x + dr.y * dr.y + dr.z * dr.z);
			if (dr_abs < cutoff)
			{
				yA = (r2.LJ_type_A - r1.LJ_type_A);
				xA = yA >> 31;
				yA = (yA^xA) - xA;
				xA = r2.LJ_type_A + r1.LJ_type_A;
				r2.LJ_type_A = (xA + yA) >> 1;
				xA = (xA - yA) >> 1;
				atom_pair_LJ_type_A = (r2.LJ_type_A*(r2.LJ_type_A + 1) >> 1) + xA;
				AAij = LJ_type_AA[atom_pair_LJ_type_A];
				ABij = LJ_type_AB[atom_pair_LJ_type_A];

				yB = (r2.LJ_type_B - r1.LJ_type_B);
				xB = yB >> 31;
				yB = (yB^xB) - xB;
				xB = r2.LJ_type_B + r1.LJ_type_B;
				r2.LJ_type_B = (xB + yB) >> 1;
				xB = (xB - yB) >> 1;
				atom_pair_LJ_type_B = (r2.LJ_type_B*(r2.LJ_type_B + 1) >> 1) + xB;
				BAij = LJ_type_BA[atom_pair_LJ_type_B];
				BBij = LJ_type_BB[atom_pair_LJ_type_B];

				soft_core = (mask_i != mask_j) || (BAij > 1e-6 && AAij < 1e-6) || (BAij < 1e-6 && AAij > 1e-6);

				if (!soft_core)
				{
					dr_1 = 1. / dr_abs;
					dr_2 = dr_1*dr_1;
					dr_4 = dr_2*dr_2;
					dr_6 = dr_4 * dr_2;
					
					beta_dr = pme_beta*dr_abs;

					ene_tmp_2 =  charge_i * charge_j * erfcf(beta_dr) * dr_1;
					ene_tmp =  (0.083333333* (lambda_ * AAij + lambda * BAij) * dr_6
						- 0.166666666*(lambda_ * ABij + lambda * BBij)) * dr_6;
				}
				else
				{
					dr2 = dr_abs * dr_abs;
					dr4 = dr2 * dr2;
					dr6 = dr4 * dr2;
					if (AAij < 1e-6 || ABij < 1e-6)
						sigma_Aij = input_sigma_6;
					else
						sigma_Aij = max(0.5 * AAij / ABij, input_sigma_6_min);
					if (BAij < 1e-6 || BBij < 1e-6)
						sigma_Bij = input_sigma_6;
					else
						sigma_Bij = max(0.5 * BAij / BBij, input_sigma_6_min);

					//printf("%f %f\n", sigma_Aij, sigma_Bij);
					//printf("%f %f %f %f\n", AAij / 12.0, ABij / 6.0, BAij / 12.0f, BBij / 6.0f);

					dr_scA6 = (dr6 + alpha_lambda_p * sigma_Aij);
					dr_scB6 = (dr6 + alpha_lambda_p_ * sigma_Bij);

					dr_scA = pow(dr_scA6, 1.0/6.0);
					dr_scB = pow(dr_scB6, 1.0/6.0);

					if (dr_scA < cutoff)
					{
						dr_sc_A6 = 1.0 / dr_scA6;
						vdw_A = ( 0.083333333 * AAij * dr_sc_A6 - 0.166666666 * ABij) * dr_sc_A6;
						dr_sc_A = 1.0 / dr_scA;
						beta_dr_sc_A = pme_beta * dr_scA;
						ee_A = erfcf(beta_dr_sc_A) * dr_sc_A;
					}
					else
					{
						vdw_A = ee_A = 0.0f;
					}

					if (dr_scB < cutoff)
					{
						dr_sc_B6 = 1.0 / dr_scB6;
						vdw_B = ( 0.083333333 * BAij * dr_sc_B6 - 0.166666666 * BBij) * dr_sc_B6;
						dr_sc_B = 1.0 / dr_scB;
						beta_dr_sc_B = pme_beta * dr_scB;
						ee_B = erfc(beta_dr_sc_B) * dr_sc_B;
					}
					else
					{
						vdw_B = ee_B = 0.0f;
					}

					ene_tmp = lambda_ * vdw_A + lambda * vdw_B;
					ene_tmp_2 = charge_i * charge_j * (lambda_ * ee_A + lambda * ee_B);
				}

				if (mask_i == mask_j)
				{
					ene_lin_intrasys += ene_tmp;
					ene_lin2_intrasys += ene_tmp_2;
				}
				else
				{
					ene_lin_intersys += ene_tmp;
					ene_lin2_intersys += ene_tmp_2;
				}
			}
		}//atom_j cycle
		atomicAdd(&lj_ene_intersys[atom_i], ene_lin_intersys);
		atomicAdd(&lj_ene_intrasys[atom_i], ene_lin_intrasys);
		atomicAdd(direct_ene_intersys, ene_lin2_intersys);
		atomicAdd(direct_ene_intrasys, ene_lin2_intrasys);
	}
}


static __global__ void LJ_Soft_Core_With_Drect_Columb_dH_dlambda_CUDA(
	const int atom_numbers, const ATOM_GROUP *nl,
	const UINT_VECTOR_LJ_FEP_TYPE *uint_crd, const VECTOR boxlength,
	const float *LJ_type_AA, const float *LJ_type_AB, const float * LJ_type_BA, const float * LJ_type_BB, const float *charge_B_A,const float cutoff, 
	float * sigma_dH_dlambda_lj, float * sigma_dH_dlambda_direct, const float pme_beta, const float sqrt_pi, const float lambda, const float alpha_lambda_p, const float alpha_lambda_p_, const float alpha_lambda_p_1, const float alpha_lambda_p_1_, const float input_sigma_6, const float input_sigma_6_min, const float p)
{
	int atom_i = blockDim.x*blockIdx.x + threadIdx.x;
	float lambda_ = 1.0 - lambda;
	if (atom_i < atom_numbers)
	{
		ATOM_GROUP nl_i = nl[atom_i];
		int N = nl_i.atom_numbers;
		int atom_j;
		int int_x;
		int int_y;
		int int_z;
		UINT_VECTOR_LJ_FEP_TYPE r1 = uint_crd[atom_i], r2;
		VECTOR dr;
		float dr2, dr4, dr6;
		float dr_sc_A6, dr_sc_B6;
		float dr_sc_A, dr_sc_B;
		float dr_sc_A12, dr_sc_B12;
		float dr_2;
		float dr_4;
		float dr_6;
		float AAij, ABij, BAij, BBij;
		float sigma_Aij, sigma_Bij;
		int mask_i, mask_j;

		float charge_i = r1.charge; //r1.charge;
		float charge_j;
		float dr_abs;
		float beta_dr_sc_A, beta_dr_sc_B;

		float charge_B_A_i = charge_B_A[atom_i], charge_B_A_j;

		float dH_dlambda_lj_sum = 0.0;
		float dH_dlambda_cf_sum = 0.0;
		float dH_dlambda_lj = 0.0;
		float dH_dlambda_cf = 0.0;

		int xA, yA, xB, yB;
		int atom_pair_LJ_type_A, atom_pair_LJ_type_B;
		mask_i = r1.mask;
		bool soft_core;

		for (int j = threadIdx.y; j < N; j = j + blockDim.y)
		{
			atom_j = nl_i.atom_serial[j];
				r2 = uint_crd[atom_j];
			mask_j = r2.mask;
			charge_j = r2.charge;
			charge_B_A_j = charge_B_A[atom_j];

			int_x = r2.uint_x - r1.uint_x;
			int_y = r2.uint_y - r1.uint_y;
			int_z = r2.uint_z - r1.uint_z;
			dr.x = boxlength.x*int_x;
			dr.y = boxlength.y*int_y;
			dr.z = boxlength.z*int_z;
			dr_abs = norm3df(dr.x, dr.y, dr.z);
			if (dr_abs < cutoff)
			{
				yA = (r2.LJ_type_A - r1.LJ_type_A);
				xA = yA >> 31;
				yA = (yA^xA) - xA;
				xA = r2.LJ_type_A + r1.LJ_type_A;
				r2.LJ_type_A = (xA + yA) >> 1;
				xA = (xA - yA) >> 1;
				atom_pair_LJ_type_A = (r2.LJ_type_A*(r2.LJ_type_A + 1) >> 1) + xA;
				AAij = LJ_type_AA[atom_pair_LJ_type_A];
				ABij = LJ_type_AB[atom_pair_LJ_type_A];

				yB = (r2.LJ_type_B - r1.LJ_type_B);
				xB = yB >> 31;
				yB = (yB^xB) - xB;
				xB = r2.LJ_type_B + r1.LJ_type_B;
				r2.LJ_type_B = (xB + yB) >> 1;
				xB = (xB - yB) >> 1;
				atom_pair_LJ_type_B = (r2.LJ_type_B*(r2.LJ_type_B + 1) >> 1) + xB;
				BAij = LJ_type_BA[atom_pair_LJ_type_B];
				BBij = LJ_type_BB[atom_pair_LJ_type_B];
				dr2 = dr_abs * dr_abs;
				
				soft_core = (mask_i != mask_j) || (BAij > 1e-6 && AAij < 1e-6) || (BAij < 1e-6 && AAij > 1e-6);
				if (soft_core)
				{
					dr4 = dr2 * dr2;
					dr6 = dr4 * dr2;
					if (AAij < 1e-6 || ABij < 1e-6)
						sigma_Aij = input_sigma_6;
					else
						sigma_Aij = max(0.5 * AAij / ABij, input_sigma_6_min);
					if (BAij < 1e-6 || BBij < 1e-6)
						sigma_Bij = input_sigma_6;
					else
						sigma_Bij = max(0.5 * BAij / BBij, input_sigma_6_min);
					dr_sc_A6 = 1.0 / (dr6 + alpha_lambda_p * sigma_Aij);
					dr_sc_B6 = 1.0 / (dr6 + alpha_lambda_p_ * sigma_Bij);
					dr_sc_A12 = dr_sc_A6 * dr_sc_A6;
					dr_sc_B12 = dr_sc_B6 * dr_sc_B6;

					dH_dlambda_lj =  dr_sc_B12 * 0.083333333 * BAij - 0.166666666 * BBij * dr_sc_B6 - 0.083333333 * AAij * dr_sc_A12 + 0.166666666 * ABij * dr_sc_A6;
					dH_dlambda_lj = dH_dlambda_lj + lambda * dr_sc_B12 * p * sigma_Bij * alpha_lambda_p_1_ * (0.166666666 * BAij * dr_sc_B6 - 0.166666666 * BBij) + lambda_ * dr_sc_A12 * p * sigma_Aij * alpha_lambda_p_1  * (-0.166666666 * AAij * dr_sc_A6 + 0.166666666 * ABij);

					dH_dlambda_lj_sum += dH_dlambda_lj;

					dr_sc_A = pow(dr_sc_A6, 1.0/6.0);
					dr_sc_B = pow(dr_sc_B6, 1.0/6.0);
					beta_dr_sc_A = pme_beta / dr_sc_A;
					beta_dr_sc_B = pme_beta / dr_sc_B;
					
					dr_sc_A = pow(dr_sc_A6, 1.0/6.0);
					dr_sc_B = pow(dr_sc_B6, 1.0/6.0);
					beta_dr_sc_A = pme_beta / dr_sc_A;
					beta_dr_sc_B = pme_beta / dr_sc_B;
					
					dH_dlambda_cf = lambda * p * sigma_Bij * alpha_lambda_p_1_ * dr_sc_B6 * (pme_beta * sqrt_pi * expf(- beta_dr_sc_B * beta_dr_sc_B) + erfcf(beta_dr_sc_B) * 0.166666666 * dr_sc_B);

					dH_dlambda_cf = dH_dlambda_cf - lambda_ * p * sigma_Aij * alpha_lambda_p_1 * dr_sc_A6 * (pme_beta * sqrt_pi * expf(-beta_dr_sc_A * beta_dr_sc_A) + erfcf(beta_dr_sc_A) * 0.166666666 * dr_sc_A);

					dH_dlambda_cf = dH_dlambda_cf - erfcf(beta_dr_sc_A)  * dr_sc_A + erfcf(beta_dr_sc_B) * dr_sc_B;
					
					dH_dlambda_cf = dH_dlambda_cf * charge_i * charge_j;

					dH_dlambda_cf = dH_dlambda_cf + (charge_B_A_j * charge_i + charge_j * charge_B_A_i) * (lambda_ * erfcf(beta_dr_sc_A) * dr_sc_A + lambda * erfcf(beta_dr_sc_B) * dr_sc_B);
					//printf("%f %f\n", record, dH_dlambda_cf - record);

					dH_dlambda_cf_sum += dH_dlambda_cf;
				}
				else
				{
					dr_2 = 1.0 / dr2;
					dr_4 = dr_2 * dr_2;
					dr_6 = dr_4 * dr_2;
					dH_dlambda_lj = (0.083333333 * (BAij - AAij) * dr_6 - 0.166666666 * (BBij - ABij)) * dr_6;
					dH_dlambda_cf = erfcf(pme_beta * dr_abs) * (charge_B_A_j * charge_i + charge_j * charge_B_A_i);

					dH_dlambda_lj_sum += dH_dlambda_lj;
					dH_dlambda_cf_sum += dH_dlambda_cf;
				}
			}
		}//atom_j cycle
		atomicAdd(sigma_dH_dlambda_lj, dH_dlambda_lj_sum);
		atomicAdd(sigma_dH_dlambda_direct, dH_dlambda_cf_sum);
	}
}


static __global__ void LJ_Soft_Core_With_Drect_Columb_dH_dlambda_Charge_Unpertubated_CUDA(
	const int atom_numbers, const ATOM_GROUP *nl,
	const UINT_VECTOR_LJ_FEP_TYPE *uint_crd, const VECTOR boxlength,
	const float *LJ_type_AA, const float *LJ_type_AB, const float * LJ_type_BA, const float * LJ_type_BB, const float cutoff, 
	float * sigma_dH_dlambda_lj, float * sigma_dH_dlambda_direct, const float pme_beta, const float sqrt_pi, const float lambda, const float alpha_lambda_p, const float alpha_lambda_p_, const float alpha_lambda_p_1, const float alpha_lambda_p_1_, const float input_sigma_6, const float input_sigma_6_min,const float p)
{
	int atom_i = blockDim.x*blockIdx.x + threadIdx.x;
	float lambda_ = 1.0 - lambda;
	if (atom_i < atom_numbers)
	{
		ATOM_GROUP nl_i = nl[atom_i];
		int N = nl_i.atom_numbers;
		int atom_j;
		int int_x;
		int int_y;
		int int_z;
		UINT_VECTOR_LJ_FEP_TYPE r1 = uint_crd[atom_i], r2;
		VECTOR dr;
		float dr2, dr4, dr6;
		float dr_sc_A6, dr_sc_B6;
		float dr_sc_A, dr_sc_B;
		float dr_sc_A12, dr_sc_B12;
		float dr_2;
		float dr_4;
		float dr_6;
		float AAij, ABij, BAij, BBij;
		float sigma_Aij, sigma_Bij;
		int mask_i = r1.mask, mask_j;

		float charge_i = r1.charge; //r1.charge;
		float charge_j;
		float dr_abs;
		float beta_dr_sc_A, beta_dr_sc_B;

		float dH_dlambda_lj_sum = 0.0;
		float dH_dlambda_cf_sum = 0.0;
		float dH_dlambda_lj = 0.0;
		float dH_dlambda_cf = 0.0;

		int xA, yA, xB, yB;
		int atom_pair_LJ_type_A, atom_pair_LJ_type_B;
		bool soft_core;

		for (int j = threadIdx.y; j < N; j = j + blockDim.y)
		{
			atom_j = nl_i.atom_serial[j];
				r2 = uint_crd[atom_j];
			mask_j = r2.mask;
			charge_j = r2.charge;

			int_x = r2.uint_x - r1.uint_x;
			int_y = r2.uint_y - r1.uint_y;
			int_z = r2.uint_z - r1.uint_z;
			dr.x = boxlength.x*int_x;
			dr.y = boxlength.y*int_y;
			dr.z = boxlength.z*int_z;
			dr_abs = norm3df(dr.x, dr.y, dr.z);
			if (dr_abs < cutoff)
			{
				yA = (r2.LJ_type_A - r1.LJ_type_A);
				xA = yA >> 31;
				yA = (yA^xA) - xA;
				xA = r2.LJ_type_A + r1.LJ_type_A;
				r2.LJ_type_A = (xA + yA) >> 1;
				xA = (xA - yA) >> 1;
				atom_pair_LJ_type_A = (r2.LJ_type_A*(r2.LJ_type_A + 1) >> 1) + xA;
				AAij = LJ_type_AA[atom_pair_LJ_type_A];
				ABij = LJ_type_AB[atom_pair_LJ_type_A];

				yB = (r2.LJ_type_B - r1.LJ_type_B);
				xB = yB >> 31;
				yB = (yB^xB) - xB;
				xB = r2.LJ_type_B + r1.LJ_type_B;
				r2.LJ_type_B = (xB + yB) >> 1;
				xB = (xB - yB) >> 1;
				atom_pair_LJ_type_B = (r2.LJ_type_B*(r2.LJ_type_B + 1) >> 1) + xB;
				BAij = LJ_type_BA[atom_pair_LJ_type_B];
				BBij = LJ_type_BB[atom_pair_LJ_type_B];
				dr2 = dr_abs * dr_abs;

				soft_core = (mask_i != mask_j) || (BAij > 1e-6 && AAij < 1e-6) || (BAij < 1e-6 && AAij > 1e-6);
				if (soft_core)
				{
					dr4 = dr2 * dr2;
					dr6 = dr4 * dr2;
					if (AAij < 1e-6 || ABij < 1e-6)
						sigma_Aij = input_sigma_6;
					else
						sigma_Aij = max(0.5 * AAij / ABij, input_sigma_6_min);
					if (BAij < 1e-6 || BBij < 1e-6)
						sigma_Bij = input_sigma_6;
					else
						sigma_Bij = max(0.5 * BAij / BBij, input_sigma_6_min);
					dr_sc_A6 = 1.0 / (dr6 + alpha_lambda_p * sigma_Aij);
					dr_sc_B6 = 1.0 / (dr6 + alpha_lambda_p_ * sigma_Bij);
					dr_sc_A12 = dr_sc_A6 * dr_sc_A6;
					dr_sc_B12 = dr_sc_B6 * dr_sc_B6;

					dH_dlambda_lj =  dr_sc_B12 * 0.083333333 * BAij - 0.166666666 * BBij * dr_sc_B6 - 0.083333333 * AAij * dr_sc_A12 + 0.166666666 * ABij * dr_sc_A6;
					dH_dlambda_lj = dH_dlambda_lj + lambda * dr_sc_B12 * p * sigma_Bij * alpha_lambda_p_1_ * (0.166666666 * BAij * dr_sc_B6 - 0.166666666 * BBij) + lambda_ * dr_sc_A12 * p * sigma_Aij * alpha_lambda_p_1  * (-0.166666666 * AAij * dr_sc_A6 + 0.166666666 * ABij);

					dH_dlambda_lj_sum += dH_dlambda_lj;

					dr_sc_A = pow(dr_sc_A6, 1.0/6.0);
					dr_sc_B = pow(dr_sc_B6, 1.0/6.0);
					beta_dr_sc_A = pme_beta / dr_sc_A;
					beta_dr_sc_B = pme_beta / dr_sc_B;
					
					dH_dlambda_cf = lambda * p * sigma_Bij * alpha_lambda_p_1_ * dr_sc_B6 * (pme_beta * sqrt_pi * expf(- beta_dr_sc_B * beta_dr_sc_B) + erfcf(beta_dr_sc_B) * 0.166666666 * dr_sc_B);

					dH_dlambda_cf = dH_dlambda_cf - lambda_ * p * sigma_Aij * alpha_lambda_p_1 * dr_sc_A6 * (pme_beta * sqrt_pi * expf(-beta_dr_sc_A * beta_dr_sc_A) + erfcf(beta_dr_sc_A) * 0.166666666 * dr_sc_A);

					dH_dlambda_cf = dH_dlambda_cf - erfcf(beta_dr_sc_A)  * dr_sc_A + erfcf(beta_dr_sc_B) * dr_sc_B;


					dH_dlambda_cf_sum = dH_dlambda_cf_sum + dH_dlambda_cf * charge_i * charge_j;
				}
				else
				{
					dr_2 = 1.0 / dr2;
					dr_4 = dr_2 * dr_2;
					dr_6 = dr_4 * dr_2;
					dH_dlambda_lj = (0.083333333 * (BAij - AAij) * dr_6 - 0.166666666 * (BBij - ABij)) * dr_6;

					dH_dlambda_lj_sum += dH_dlambda_lj;
				}
			}
		}//atom_j cycle
		atomicAdd(sigma_dH_dlambda_lj, dH_dlambda_lj_sum);
		atomicAdd(sigma_dH_dlambda_direct, dH_dlambda_cf_sum);
	}
}


void LJ_SOFT_CORE::Initial(CONTROLLER *controller, float cutoff, VECTOR box_length, const char *module_name)
{
	if (module_name == NULL)
	{
		strcpy(this->module_name, "LJ_soft_core");
	}
	else
	{
		strcpy(this->module_name, module_name);
	}
		controller[0].printf("START INITIALIZING FEP SOFT CORE FOR LJ AND COULOMB:\n");
		if (controller[0].Command_Exist(this->module_name, "in_file"))
		{
			if (controller[0].Command_Exist("lambda_lj"))
			{
				this->lambda = atof(controller[0].Command("lambda_lj"));
			}
			else
			{
				printf("\tError: FEP lambda of LJ must be given for the calculation of SOFT CORE.\n");
			}

			if (controller[0].Command_Exist("soft_core_alpha"))
			{
				this->alpha = atof(controller[0].Command("soft_core_alpha"));
				printf("\tFEP soft core alpha: %f\n", this->alpha);
			}
			else
			{
				printf("\tWarning: FEP alpha of soft core missing for the calculation of SOFT CORE, set to default value 0.0.\n");
				this->alpha = 0.0;
			}

			if (controller[0].Command_Exist("soft_core_power"))
			{
				this->p = atof(controller[0].Command("soft_core_power"));
				printf("\tFEP soft core power: %f\n", this->p);
			}
			else
			{
				printf("\tWarning: FEP p of soft core missing for the calculation of SOFT CORE, set to default value 1.0.\n");
				this->p = 1.0;
			}
			
			if (controller[0].Command_Exist("soft_core_sigma"))
			{
				this->sigma = atof(controller[0].Command("soft_core_sigma"));
				printf("\tFEP soft core sigma: %f\n", this->sigma);
			}
			else
			{
				printf("Warning: FEP sigma of soft core missing for the calculation of SOFT CORE, set to default value 0.0\n");
				this->sigma = 0.0;
			}
			if (controller[0].Command_Exist("soft_core_sigma_min"))
			{
				this->sigma_min = atof(controller[0].Command("soft_core_sigma_min"));
				printf("\tFEP soft core sigma min: %f\n", this->sigma_min);
			}
			else
			{
				printf("Warning: FEP minimal sigma of soft core missing for the calculation of SOFT CORE, set to default value 0.0\n");
				this->sigma_min = 0.0;
			}

			FILE *fp = NULL;
			Open_File_Safely(&fp, controller[0].Command(this->module_name, "in_file"), "r");

			int toscan = fscanf(fp, "%d %d %d", &atom_numbers, &atom_type_numbers_A, &atom_type_numbers_B);
			controller[0].printf("    atom_numbers is %d\n", atom_numbers);
			controller[0].printf("    atom_LJ_type_number_A is %d, atom_LJ_type_number_B is %d\n", atom_type_numbers_A, atom_type_numbers_B);
			pair_type_numbers_A = atom_type_numbers_A * (atom_type_numbers_A + 1) / 2;
			pair_type_numbers_B = atom_type_numbers_B * (atom_type_numbers_B + 1) / 2;
			this->thread_LJ = { 8, 32 };
			LJ_Soft_Core_Malloc();

			for (int i = 0; i < pair_type_numbers_A; i++)
			{
				toscan = fscanf(fp, "%f", h_LJ_AA + i);
				h_LJ_AA[i] *= 12.0f;
			}
			for (int i = 0; i < pair_type_numbers_A; i++)
			{
				toscan = fscanf(fp, "%f", h_LJ_AB + i);
				h_LJ_AB[i] *= 6.0f;
			}
			for (int i = 0; i < pair_type_numbers_B; ++i)
			{
				toscan = fscanf(fp, "%f", h_LJ_BA + i);
				h_LJ_BA[i] *= 12.0f;
			}
			for (int i = 0; i < pair_type_numbers_B; ++i)
			{
				toscan = fscanf(fp, "%f", h_LJ_BB + i);
				h_LJ_BB[i] *= 6.0f;
			}
			for (int i = 0; i < atom_numbers; i++)
			{
				toscan = fscanf(fp, "%d %d", h_atom_LJ_type_A + i, h_atom_LJ_type_B + i);
			}
			fclose(fp);

			if (controller[0].Command_Exist("subsys_division_in_file"))
			{
				FILE * fp = NULL;
				controller->printf("	Start reading subsystem division information:\n");
				Open_File_Safely(&fp, controller[0].Command("subsys_division_in_file"), "r");
				int atom_numbers = 0;
				char lin[CHAR_LENGTH_MAX];
				char * get_ret = fgets(lin, CHAR_LENGTH_MAX, fp);
				toscan = sscanf(lin, "%d", &atom_numbers);
				if (this->atom_numbers > 0 && this->atom_numbers != atom_numbers)
				{
					controller->printf("        Error: atom_numbers is not equal: %d %d\n", this->atom_numbers, atom_numbers);
					getchar();
					exit(1);
				}
				else if (this->atom_numbers == 0)
				{
					this->atom_numbers = atom_numbers;
				}
				for (int i = 0; i < atom_numbers; i++)
				{
					toscan = fscanf(fp, "%d", &h_subsys_division[i]);
				}
				controller->printf("    End reading subsystem division information\n\n");
				fclose(fp);
			}
			else
			{
				controller[0].printf("    subsystem mask is set to 0 as default\n");
				for (int i = 0; i < atom_numbers; i++)
				{
					h_subsys_division[i] = 0;
				}
			}

			Parameter_Host_To_Device();
			is_initialized = 1;
			alpha_lambda_p = alpha * pow(lambda, p);
			alpha_lambda_p_ = alpha * pow(1 - lambda, p);
			sigma_6 = pow(sigma, 6);
			sigma_6_min = pow(sigma_min, 6);
			alpha_lambda_p_1 = alpha * pow(lambda, p-1);
			alpha_lambda_p_1_ = alpha * pow(1.0 - lambda, p-1);

			pme_tolerance = 0.00001;
			if (controller[0].Command_Exist("PME_Direct_Tolerance"))
				pme_tolerance = atof(controller[0].Command("PME_Direct_Tolerance"));

			pme_beta = Get_Beta_LJ(cutoff, pme_tolerance);
		}
		if (is_initialized)
		{
			this->cutoff = cutoff;
			this->uint_dr_to_dr_cof = 1.0f / CONSTANT_UINT_MAX_FLOAT * box_length;
			Cuda_Malloc_Safely((void **)&uint_crd_with_LJ, sizeof(UINT_VECTOR_LJ_FEP_TYPE)* atom_numbers);
			Copy_LJ_Type_And_Mask_To_New_Crd << <ceilf((float)this->atom_numbers / 32), 32 >> >
				(atom_numbers, uint_crd_with_LJ, d_atom_LJ_type_A, d_atom_LJ_type_B, d_subsys_division);

			controller[0].printf("    Start initializing long range LJ correction\n");
			long_range_factor = 0;
			float *d_factor = NULL;
			Cuda_Malloc_Safely((void**)&d_factor, sizeof(float));
			Reset_List(d_factor, 0.0f, 1, 1);
			Total_C6_Get << < {4, 4}, { 32, 32 } >> >(atom_numbers, d_atom_LJ_type_A, d_atom_LJ_type_B,d_LJ_AB, d_LJ_BB, d_factor, this->lambda);
			hipMemcpy(&long_range_factor, d_factor, sizeof(float), hipMemcpyDeviceToHost);
			hipFree(d_factor);

			long_range_factor *= -2.0f / 3.0f * CONSTANT_Pi / cutoff / cutoff / cutoff / 6.0f;
			this->volume = box_length.x * box_length.y * box_length.z;
			controller[0].printf("        long range correction factor is: %e\n", long_range_factor);
			controller[0].printf("    End initializing long range LJ correction\n");
		}
		if (is_initialized && !is_controller_printf_initialized)
		{
			controller[0].Step_Print_Initial("LJ(sc.)", "%.2f");
			controller[0].Step_Print_Initial("LR_corr(sc.)", "%.2f");
			is_controller_printf_initialized = 1;
			controller[0].printf("    structure last modify date is %d\n", last_modify_date);
		}
		controller[0].printf("END INITIALIZING LENNADR JONES INFORMATION\n\n");
}

void LJ_SOFT_CORE::LJ_Soft_Core_Malloc()
{
	Malloc_Safely((void**)&h_LJ_energy_atom, sizeof(float)*atom_numbers);
	Malloc_Safely((void**)&h_atom_LJ_type_A, sizeof(int)*atom_numbers);
	Malloc_Safely((void**)&h_atom_LJ_type_B, sizeof(int)*atom_numbers);
	Malloc_Safely((void**)&h_LJ_AA, sizeof(float)*pair_type_numbers_A);
	Malloc_Safely((void**)&h_LJ_AB, sizeof(float)*pair_type_numbers_A);
	Malloc_Safely((void**)&h_LJ_BA, sizeof(float)*pair_type_numbers_B);
	Malloc_Safely((void**)&h_LJ_BB, sizeof(float)*pair_type_numbers_B);
	Malloc_Safely((void**)&h_subsys_division, sizeof(int)*atom_numbers);
	
	Cuda_Malloc_Safely((void**)&d_LJ_energy_sum, sizeof(float));
	Cuda_Malloc_Safely((void**)&d_LJ_energy_atom, sizeof(float)*atom_numbers);
	Cuda_Malloc_Safely((void**)&d_atom_LJ_type_A, sizeof(int)*atom_numbers);
	Cuda_Malloc_Safely((void**)&d_atom_LJ_type_B, sizeof(int)*atom_numbers);
	Cuda_Malloc_Safely((void**)&d_LJ_AA, sizeof(float)*pair_type_numbers_A);
	Cuda_Malloc_Safely((void**)&d_LJ_AB, sizeof(float)*pair_type_numbers_A);
	Cuda_Malloc_Safely((void**)&d_LJ_BA, sizeof(float)*pair_type_numbers_B);
	Cuda_Malloc_Safely((void**)&d_LJ_BB, sizeof(float)*pair_type_numbers_B);
	Cuda_Malloc_Safely((void**)&d_subsys_division, sizeof(int)*atom_numbers);

	Malloc_Safely((void**)&h_LJ_energy_atom_intersys, sizeof(float)*atom_numbers);
	Malloc_Safely((void**)&h_LJ_energy_atom_intrasys, sizeof(float)*atom_numbers);

	Cuda_Malloc_Safely((void**)&d_direct_ene_sum_intersys, sizeof(float));
	Cuda_Malloc_Safely((void**)&d_direct_ene_sum_intrasys, sizeof(float));
	Cuda_Malloc_Safely((void**)&d_LJ_energy_sum_intersys, sizeof(float));
	Cuda_Malloc_Safely((void**)&d_LJ_energy_sum_intrasys, sizeof(float));
	Cuda_Malloc_Safely((void**)&d_LJ_energy_atom_intersys, sizeof(float)*atom_numbers);
	Cuda_Malloc_Safely((void**)&d_LJ_energy_atom_intrasys, sizeof(float)*atom_numbers);

	Malloc_Safely((void**)&h_sigma_of_dH_dlambda_lj, sizeof(float));
	Malloc_Safely((void**)&h_sigma_of_dH_dlambda_direct, sizeof(float));

	Cuda_Malloc_Safely((void**)&d_sigma_of_dH_dlambda_lj, sizeof(float));
	Cuda_Malloc_Safely((void**)&d_sigma_of_dH_dlambda_direct, sizeof(float));
	Cuda_Malloc_Safely((void**)&d_long_range_correction, sizeof(float));
}

void LJ_SOFT_CORE::Parameter_Host_To_Device()
{
	hipMemcpy(d_LJ_AB, h_LJ_AB, sizeof(float)*pair_type_numbers_A, hipMemcpyHostToDevice);
	hipMemcpy(d_LJ_AA, h_LJ_AA, sizeof(float)*pair_type_numbers_A, hipMemcpyHostToDevice);

	hipMemcpy(d_LJ_BA, h_LJ_BA, sizeof(float)*pair_type_numbers_B, hipMemcpyHostToDevice);
	hipMemcpy(d_LJ_BB, h_LJ_BB, sizeof(float)*pair_type_numbers_B, hipMemcpyHostToDevice);

	hipMemcpy(d_atom_LJ_type_A, h_atom_LJ_type_A, sizeof(int)*atom_numbers, hipMemcpyHostToDevice);
	hipMemcpy(d_atom_LJ_type_B, h_atom_LJ_type_B, sizeof(int)*atom_numbers, hipMemcpyHostToDevice);
	hipMemcpy(d_subsys_division, h_subsys_division, sizeof(int)*atom_numbers, hipMemcpyHostToDevice);
}

void LJ_SOFT_CORE::Clear()
{
	if (is_initialized)
	{
		is_initialized = 0;

		free(h_atom_LJ_type_A);
		free(h_atom_LJ_type_B);
		hipFree(d_atom_LJ_type_A);
		hipFree(d_atom_LJ_type_B);

		free(h_LJ_AA);
		free(h_LJ_AB);
		free(h_LJ_BA);
		free(h_LJ_BB);
		hipFree(d_LJ_AA);
		hipFree(d_LJ_AB);
		hipFree(d_LJ_BA);
		hipFree(d_LJ_BB);

		free(h_LJ_energy_atom);
		hipFree(d_LJ_energy_atom);
		hipFree(d_LJ_energy_sum);

		hipFree(uint_crd_with_LJ);

		free(h_subsys_division);
		hipFree(d_subsys_division);

		h_atom_LJ_type_A = NULL;      
		d_atom_LJ_type_A = NULL;
		h_atom_LJ_type_B = NULL;
		d_atom_LJ_type_B = NULL;       

		h_LJ_AA = NULL;              
		h_LJ_AB = NULL;            
		d_LJ_AA = NULL;          
		d_LJ_AB = NULL;
		
		h_LJ_BA = NULL;
		h_LJ_BB = NULL;
		d_LJ_BA = NULL;
		d_LJ_BB = NULL;

		h_LJ_energy_atom = NULL;    
		d_LJ_energy_atom = NULL;
		d_LJ_energy_sum = NULL;     

		uint_crd_with_LJ = NULL;
		h_subsys_division = NULL;
		d_subsys_division = NULL;
	}
}

void LJ_SOFT_CORE::LJ_Soft_Core_Force_With_PME_Direct_Force(const int atom_numbers, const UINT_VECTOR_LJ_FEP_TYPE *uint_crd, const VECTOR scaler, VECTOR *frc,
	const ATOM_GROUP *nl, const float cutoff, const float pme_beta)
{
	if (is_initialized)
		LJ_Soft_Core_Force_With_Direct_CF_CUDA << <(unsigned int)ceilf((float)atom_numbers / thread_LJ.x), thread_LJ >> >
			(atom_numbers, nl,
			uint_crd, scaler,
			d_LJ_AA, d_LJ_AB, d_LJ_BA, d_LJ_BB, cutoff,
		frc, pme_beta, TWO_DIVIDED_BY_SQRT_PI, lambda, alpha_lambda_p, alpha_lambda_p_, sigma_6, sigma_6_min);
}

void LJ_SOFT_CORE::LJ_Soft_Core_PME_Direct_Force_With_Atom_Energy(const int atom_numbers, const UINT_VECTOR_LJ_FEP_TYPE *uint_crd, const VECTOR scaler, VECTOR *frc,
	const ATOM_GROUP *nl, const float cutoff, const float pme_beta,float *atom_energy)
{
	if (is_initialized)
		LJ_Soft_Core_Direct_CF_Force_With_Atom_Energy_CUDA << <(unsigned int)ceilf((float)atom_numbers / thread_LJ.x), thread_LJ >> >
			(atom_numbers, nl,
			uint_crd, scaler,
			d_LJ_AA, d_LJ_AB, d_LJ_BA, d_LJ_BB, cutoff,
			frc, pme_beta, TWO_DIVIDED_BY_SQRT_PI,atom_energy, lambda, alpha_lambda_p, alpha_lambda_p_, sigma_6, sigma_6_min);
}

void LJ_SOFT_CORE::LJ_Soft_Core_PME_Direct_Force_With_Atom_Energy_And_Virial(const int atom_numbers, const UNSIGNED_INT_VECTOR *uint_crd, const float *charge, VECTOR *frc,
	const ATOM_GROUP *nl, const float pme_beta, const int need_atom_energy, float *atom_energy,
	const int need_virial, float *atom_lj_virial, float *atom_direct_pme_energy)
{
	if (is_initialized)
	{
		Copy_Crd_And_Charge_To_New_Crd << <(unsigned int)ceilf((float)atom_numbers / 32), 32 >> >(atom_numbers, uint_crd, uint_crd_with_LJ, charge);
		if (!need_atom_energy > 0 && !need_virial > 0)
		{
			LJ_Soft_Core_Force_With_PME_Direct_Force(atom_numbers, uint_crd_with_LJ, uint_dr_to_dr_cof, frc, nl, cutoff, pme_beta);
		}
		else if (need_atom_energy > 0 && !need_virial> 0)
		{
			LJ_Soft_Core_PME_Direct_Force_With_Atom_Energy(atom_numbers, uint_crd_with_LJ, uint_dr_to_dr_cof, frc, nl, cutoff, pme_beta, atom_energy);
		}
		else if (!need_atom_energy > 0 && need_virial> 0)
		{
			Reset_List(atom_direct_pme_energy, 0.0f, atom_numbers, 1024);
			LJ_Soft_Core_Direct_CF_Force_With_LJ_Virial_Direct_CF_Energy_CUDA << <(unsigned int)ceilf((float)atom_numbers / thread_LJ.x), thread_LJ >> >
				(atom_numbers, nl,
				uint_crd_with_LJ, uint_dr_to_dr_cof,
				d_LJ_AA, d_LJ_AB, d_LJ_BA, d_LJ_BB,cutoff,
				frc, pme_beta, TWO_DIVIDED_BY_SQRT_PI, atom_lj_virial, atom_direct_pme_energy, lambda, alpha_lambda_p, alpha_lambda_p_, sigma_6, sigma_6_min);
		}
		else
		{
			Reset_List(atom_direct_pme_energy, 0.0f, atom_numbers, 1024);
			LJ_Soft_Core_Direct_CF_Force_With_Atom_Energy_And_LJ_Virial_Direct_CF_Energy_CUDA << <(unsigned int)ceilf((float)atom_numbers / thread_LJ.x), thread_LJ >> >
				(atom_numbers, nl,
				uint_crd_with_LJ, uint_dr_to_dr_cof,
				d_LJ_AA, d_LJ_AB, d_LJ_BA, d_LJ_BB, cutoff,
				frc, pme_beta, TWO_DIVIDED_BY_SQRT_PI, atom_energy, atom_lj_virial, atom_direct_pme_energy, lambda, alpha_lambda_p, alpha_lambda_p, sigma_6, sigma_6_min);
		}
	}
}


float LJ_SOFT_CORE::Get_Energy(const UNSIGNED_INT_VECTOR *uint_crd, const ATOM_GROUP *nl, const float pme_beta, const float * charge,float * direct_ene_sum, int is_download)
{
	if (is_initialized)
	{
		//printf("This func! is_download = %d\n", is_download);
		//printf("thread_LJ.x: %u %u\n", thread_LJ.x, thread_LJ.y);
		Copy_Crd_And_Charge_To_New_Crd << <(unsigned int)ceilf((float)atom_numbers / 32), 32 >> >(atom_numbers, uint_crd, uint_crd_with_LJ, charge);

		Reset_List(d_LJ_energy_atom, 0., atom_numbers, 1024);
		LJ_Soft_Core_Energy_CUDA << <(unsigned int)ceilf((float)atom_numbers / thread_LJ.x), thread_LJ >> >
			(atom_numbers, nl,
			uint_crd_with_LJ, uint_dr_to_dr_cof,
			d_LJ_AA, d_LJ_AB, d_LJ_BA, d_LJ_BB, cutoff, pme_beta,
			d_LJ_energy_atom, direct_ene_sum, lambda, alpha_lambda_p, alpha_lambda_p_, sigma_6, sigma_6_min);

		Sum_Of_List(d_LJ_energy_atom, d_LJ_energy_sum, atom_numbers);

		long_range_correction = long_range_factor / this->volume;

		if (is_download)
		{
			hipMemcpy(&h_LJ_energy_sum, this->d_LJ_energy_sum, sizeof(float), hipMemcpyDeviceToHost);
			return h_LJ_energy_sum;
		}
		else
		{
			return 0;
		}
	}
        return NAN;
}

float LJ_SOFT_CORE::Get_Energy_With_Coulomb_Direct(
	const UNSIGNED_INT_VECTOR * uint_crd, const ATOM_GROUP * nl, const float * charge, int is_download)
{
	if (is_initialized)
	{
		Copy_Crd_And_Charge_To_New_Crd << <(unsigned int)ceilf((float)atom_numbers / 32), 32 >> >(atom_numbers, uint_crd, uint_crd_with_LJ, charge);

		Reset_List(d_LJ_energy_atom_intersys, 0., atom_numbers, 1024);
		Reset_List(d_LJ_energy_atom_intrasys, 0., atom_numbers, 1024);
		hipMemset(d_direct_ene_sum_intersys, 0, sizeof(float));
		hipMemset(d_direct_ene_sum_intrasys, 0, sizeof(float));

		LJ_Soft_Core_Energy_With_Coulumb_Direct_CUDA << <(unsigned int)ceilf((float)atom_numbers / thread_LJ.x), thread_LJ >> >
			(atom_numbers, nl, uint_crd_with_LJ, uint_dr_to_dr_cof,
			d_LJ_AA, d_LJ_AB, d_LJ_BA, d_LJ_BB, cutoff, pme_beta,
			d_LJ_energy_atom_intersys, d_LJ_energy_atom_intrasys, d_direct_ene_sum_intersys, d_direct_ene_sum_intrasys, lambda, alpha_lambda_p, alpha_lambda_p_, sigma_6, sigma_6_min);
		//hipDeviceSynchronize();
		//getchar();

		Sum_Of_List(d_LJ_energy_atom_intersys, d_LJ_energy_sum_intersys, atom_numbers);
		Sum_Of_List(d_LJ_energy_atom_intrasys, d_LJ_energy_sum_intrasys, atom_numbers);
		

		if (is_download)
		{
			hipMemcpy(&h_LJ_energy_sum_intersys, this->d_LJ_energy_sum_intersys, sizeof(float), hipMemcpyDeviceToHost);
			hipMemcpy(&h_LJ_energy_sum_intrasys, this->d_LJ_energy_sum_intrasys, sizeof(float), hipMemcpyDeviceToHost);
			hipMemcpy(&h_direct_ene_sum_intersys, this->d_direct_ene_sum_intersys, sizeof(float), hipMemcpyDeviceToHost);
			hipMemcpy(&h_direct_ene_sum_intrasys, this->d_direct_ene_sum_intrasys, sizeof(float), hipMemcpyDeviceToHost);

			h_LJ_energy_sum = h_LJ_energy_sum_intersys + h_LJ_energy_sum_intrasys;
			return h_LJ_energy_sum;
		}
		else
		{
			return 0.0;
		}
	}
	else
	{
		return NAN;
	}
}


float LJ_SOFT_CORE::Get_Partial_H_Partial_Lambda_With_Columb_Direct(const UNSIGNED_INT_VECTOR * uint_crd, const float * charge, const ATOM_GROUP * nl, const float * charge_B_A, const int charge_pertubated, int is_download)
{
	if (is_initialized)
	{
		Copy_Crd_And_Charge_To_New_Crd << <(unsigned int)ceilf((float)atom_numbers / 32), 32 >> >(atom_numbers, uint_crd, uint_crd_with_LJ, charge);

		hipMemset(d_sigma_of_dH_dlambda_lj, 0, sizeof(float));
		hipMemset(d_sigma_of_dH_dlambda_direct, 0, sizeof(float));
		if (charge_pertubated > 0)
		{
			LJ_Soft_Core_With_Drect_Columb_dH_dlambda_CUDA << <(unsigned int)ceilf((float)atom_numbers / thread_LJ.x), thread_LJ >> >
				(atom_numbers, nl,
				uint_crd_with_LJ, uint_dr_to_dr_cof,
				d_LJ_AA, d_LJ_AB, d_LJ_BA, d_LJ_BB, charge_B_A, cutoff,
				d_sigma_of_dH_dlambda_lj, d_sigma_of_dH_dlambda_direct, pme_beta, ONE_DIVIDED_BY_3_SQRT_PI, lambda, alpha_lambda_p, alpha_lambda_p_, alpha_lambda_p_1, alpha_lambda_p_1_, sigma_6, sigma_6_min, p);
		}
		else
		{
			LJ_Soft_Core_With_Drect_Columb_dH_dlambda_Charge_Unpertubated_CUDA << <(unsigned int)ceilf((float)atom_numbers / thread_LJ.x), thread_LJ >> >
				(atom_numbers, nl,
				uint_crd_with_LJ, uint_dr_to_dr_cof,
				d_LJ_AA, d_LJ_AB, d_LJ_BA, d_LJ_BB, cutoff,
				d_sigma_of_dH_dlambda_lj, d_sigma_of_dH_dlambda_direct, pme_beta, ONE_DIVIDED_BY_3_SQRT_PI, lambda, alpha_lambda_p, alpha_lambda_p_, alpha_lambda_p_1, alpha_lambda_p_1_, sigma_6, sigma_6_min, p);
		}

		if (is_download)
		{
			hipMemcpy(h_sigma_of_dH_dlambda_lj, d_sigma_of_dH_dlambda_lj, sizeof(float), hipMemcpyDeviceToHost);
			hipMemcpy(h_sigma_of_dH_dlambda_direct, d_sigma_of_dH_dlambda_direct, sizeof(float), hipMemcpyDeviceToHost);
			return *h_sigma_of_dH_dlambda_lj;
		}
		else
		{
			return 0;
		}
	}
	else
	{
		return NAN;
	}
}

void LJ_SOFT_CORE::Update_Volume(VECTOR box_length)
{
	if (!is_initialized)
		return;
	this->uint_dr_to_dr_cof = 1.0f / CONSTANT_UINT_MAX_FLOAT * box_length;
	this->volume = box_length.x * box_length.y * box_length.z;
}

void LJ_SOFT_CORE::Long_Range_Correction(float volume)
{
	if (is_initialized)
	{
		device_add << <1, 1 >> >(d_LJ_energy_sum, long_range_factor / volume);
	}
}

void LJ_SOFT_CORE::Long_Range_Correction(int need_pressure, float *d_virial, int need_potential, float *d_potential)
{
	if (is_initialized)
	{	
		if (need_pressure > 0)
		{
			device_add << <1, 1 >> >(d_virial, long_range_factor * 6.0f / volume);
		}
		if (need_potential > 0)
		{
			device_add << <1, 1 >> >(d_potential, long_range_factor / volume);
		}
	}
}

float LJ_SOFT_CORE::Long_Range_Correction()
{
	if (is_initialized)
	{
		return long_range_factor / this->volume;
	}
	else
	{
		return NAN;
	}
}

float LJ_SOFT_CORE::Partial_H_Partial_Lambda_Long_Range_Correction()
{
	if (is_initialized)
	{
		hipMemset(d_long_range_correction, 0, sizeof(float));
		device_add << <1, 1 >> >(d_long_range_correction, long_range_factor / volume);
		hipMemcpy(&long_range_correction, d_long_range_correction, sizeof(float), hipMemcpyDeviceToHost);
		//printf("long range correction: %f\n", *long_range_correction);
		return long_range_correction;
	}
	else
	{
		return NAN;
	}
}
